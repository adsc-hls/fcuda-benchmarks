#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 **************************************************************************
 * \file dct8x8_kernel1.cu
 * \brief Contains 1st CUDA implementations of DCT, IDCT and quantization routines, 
 *        used in JPEG internal data processing. Device code.
 *
 * This code implements first CUDA versions of forward and inverse Discrete Cosine 
 * Transform to blocks of image pixels (of 8x8 size), as in JPEG standard. The data 
 * processing is done using DATATYPEing point representation.
 * The routine that performs quantization of coefficients can be found in 
 * dct8x8_kernel_quantization.cu file.
 */
#include <stdio.h>
#include <fcuda.h>

//#include "Common.h"
/**
 *  The dimension of pixels block
 */
#define BLOCK_SIZE			8

/**
 *  Square of dimension of pixels block
 */
#define BLOCK_SIZE2			64

/**
 *  log_2{BLOCK_SIZE), used for quick multiplication or division by the 
 *  pixels block dimension via shifting
 */
#define BLOCK_SIZE_LOG2		3

/**
 *  log_2{BLOCK_SIZE*BLOCK_SIZE), used for quick multiplication or division by the 
 *  square of pixels block via shifting
 */
#define BLOCK_SIZE2_LOG2	6

/**
 *  Wrapper to the fastest integer multiplication function on CUDA
 */
#ifdef __MUL24_FASTER_THAN_ASTERIX
#define FMUL(x,y)	(__mul24(x,y))
#else
#define FMUL(x,y)	((x)*(y))
#endif


/**
 *  This unitary matrix performs discrete cosine transform of rows of the matrix to the left
 */
/*
   __constant__ DATATYPE DCTv8matrix[] = {
   0.3535533905932738f,  0.4903926402016152f,  0.4619397662556434f,  0.4157348061512726f,  0.3535533905932738f,  0.2777851165098011f,  0.1913417161825449f,  0.0975451610080642f, 
   0.3535533905932738f,  0.4157348061512726f,  0.1913417161825449f, -0.0975451610080641f, -0.3535533905932737f, -0.4903926402016152f, -0.4619397662556434f, -0.2777851165098011f, 
   0.3535533905932738f,  0.2777851165098011f, -0.1913417161825449f, -0.4903926402016152f, -0.3535533905932738f,  0.0975451610080642f,  0.4619397662556433f,  0.4157348061512727f, 
   0.3535533905932738f,  0.0975451610080642f, -0.4619397662556434f, -0.2777851165098011f,  0.3535533905932737f,  0.4157348061512727f, -0.1913417161825450f, -0.4903926402016153f, 
   0.3535533905932738f, -0.0975451610080641f, -0.4619397662556434f,  0.2777851165098009f,  0.3535533905932738f, -0.4157348061512726f, -0.1913417161825453f,  0.4903926402016152f, 
   0.3535533905932738f, -0.2777851165098010f, -0.1913417161825452f,  0.4903926402016153f, -0.3535533905932733f, -0.0975451610080649f,  0.4619397662556437f, -0.4157348061512720f, 
   0.3535533905932738f, -0.4157348061512727f,  0.1913417161825450f,  0.0975451610080640f, -0.3535533905932736f,  0.4903926402016152f, -0.4619397662556435f,  0.2777851165098022f, 
   0.3535533905932738f, -0.4903926402016152f,  0.4619397662556433f, -0.4157348061512721f,  0.3535533905932733f, -0.2777851165098008f,  0.1913417161825431f, -0.0975451610080625f
   };
 */



/**
 **************************************************************************
 *  Performs 1st implementation of 8x8 block-wise Forward Discrete Cosine Transform of the given 
 *  image plane and outputs result to the array of coefficients.
 *
 * \param Dst			[OUT] - Coefficients plane
 * \param ImgWidth		[IN] - Stride of Dst
 * \param OffsetXBlocks	[IN] - Offset along X in blocks from which to perform processing
 * \param OffsetYBlocks	[IN] - Offset along Y in blocks from which to perform processing
 *  
 * \return None
 */
#define GRID_X 2
#define GRID_Y 2
#define SIZE_SRC (GRID_X * GRID_Y * 8 * 8)
#define SIZE_DST (512 * GRID_Y + 8 * GRID_X - 64)
#define SIZE (64 + SIZE_SRC + SIZE_DST)

typedef int DATATYPE;


#define ImgWidth 64
//int OffsetXBlocks
//int OffsetYBlocks

#pragma FCUDA GRID x_dim=8 y_dim=8  
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=dct_const port_id=0
#pragma FCUDA PORTMERGE remove_port_name=TexSrc port_id=0
#pragma FCUDA PORTMERGE remove_port_name=Dst port_id=0
__global__ void InvDCT(DATATYPE *Dst, DATATYPE *TexSrc, DATATYPE *dct_const)
{

  // Temporary blocks
  __shared__ DATATYPE CurBlockLocal1[BLOCK_SIZE2];
  __shared__ DATATYPE CurBlockLocal2[BLOCK_SIZE2];
  //#pragma FCUDA PORTMERGE remove_port_name=DCTv8matrix
  __shared__ DATATYPE DCTv8matrix[BLOCK_SIZE2];
#pragma FCUDA COMPUTE begin name=compute
  // Block index
  int bx;
  bx = blockIdx.x;
  int by;
  by= blockIdx.y;

  // Thread index (current coefficient)
  int tx;
  tx = threadIdx.x;
  int ty;
  ty = threadIdx.y;

  int i;
  i = BLOCK_SIZE * ty + tx;

  DCTv8matrix[i] = dct_const[i];
  __syncthreads();

  // Texture coordinates
  int tex_x;
  tex_x = ( (bx << BLOCK_SIZE_LOG2) + tx );// + 0.5f;
  int tex_y;
  tex_y = ( (by << BLOCK_SIZE_LOG2) + ty );// + 0.5f;

  int offset;
  int j;
  offset = ((by*(gridDim.x*BLOCK_SIZE2))+(bx*BLOCK_SIZE2));
  j = BLOCK_SIZE*ty + tx;
  //copy current image pixel to the first block
  CurBlockLocal1[ (ty << BLOCK_SIZE_LOG2) + tx ] = TexSrc[offset + j]; //tex2D(TexSrc, tex_x, tex_y);

  //synchronize threads to make sure the block is copied
  __syncthreads();

  //calculate the multiplication of DCTv8matrixT * A and place it in the second block
  DATATYPE curelem;
  curelem = 0.0;
  int DCTv8matrixIndex;
  DCTv8matrixIndex = (ty << BLOCK_SIZE_LOG2) + 0;
  int CurBlockLocal1Index;
  CurBlockLocal1Index = 0 * BLOCK_SIZE + tx;

  for (i=0; i<BLOCK_SIZE; i++)
  {
    curelem = curelem + DCTv8matrix[DCTv8matrixIndex] * CurBlockLocal1[CurBlockLocal1Index];
    DCTv8matrixIndex += 1;
    CurBlockLocal1Index += BLOCK_SIZE;
  }
  CurBlockLocal2[ (ty << BLOCK_SIZE_LOG2) + tx ] = curelem;

  //synchronize threads to make sure the first 2 matrices are multiplied and the result is stored in the second block
  __syncthreads();


  //calculate the multiplication of (DCTv8matrixT * A) * DCTv8matrix and place it in the first block
  curelem = 0;
  int CurBlockLocal2Index;
  CurBlockLocal2Index = (ty << BLOCK_SIZE_LOG2) + 0;
  DCTv8matrixIndex = (tx << BLOCK_SIZE_LOG2) + 0;

  for (i=0; i<BLOCK_SIZE; i++)
  {
    curelem += CurBlockLocal2[CurBlockLocal2Index] * DCTv8matrix[DCTv8matrixIndex];
    CurBlockLocal2Index += 1;
    DCTv8matrixIndex += 1;
  }
  CurBlockLocal1[ (ty << BLOCK_SIZE_LOG2) + tx ] = curelem;

  //synchronize threads to make sure the matrices are multiplied and the result is stored back in the first block
  __syncthreads();

  //copy current coefficient to its place in the result array
  Dst[ FMUL(((by << BLOCK_SIZE_LOG2) + ty), ImgWidth) + ((bx << BLOCK_SIZE_LOG2) + tx) ] = CurBlockLocal1[ (ty << BLOCK_SIZE_LOG2) + tx ];
#pragma FCUDA COMPUTE end name=compute
}
