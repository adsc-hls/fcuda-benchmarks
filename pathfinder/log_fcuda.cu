#include "hip/hip_runtime.h"
[LinkSymbol] 103 updates in 0.00 seconds
[AnnotParser] begin
PreAnnotation: #pragma startinclude #include <fcuda.h>

Parent: TranslationUnit 

PreAnnotation: #pragma endinclude

Parent: TranslationUnit 

PreAnnotation: #pragma startinclude #include <string.h>

Parent: TranslationUnit 

PreAnnotation: #pragma endinclude

Parent: TranslationUnit 

PreAnnotation: #pragma startinclude #include "pathfinder.h"

Parent: TranslationUnit 

PreAnnotation: #pragma endinclude

Parent: TranslationUnit 

PreAnnotation: #pragma FCUDA GRID x_dim=256

Token #
Token pragma
Token FCUDA
Token GRID
Token x_dim=256
#pragma fcuda grid x_dim=256 
attached=1

PreAnnotation: #pragma FCUDA COREINFO num_cores=1 pipeline=yes

Token #
Token pragma
Token FCUDA
Token COREINFO
Token num_cores=1
Token pipeline=yes
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
attached=1

PreAnnotation: #pragma FCUDA PORTMERGE remove_port_name=gpuWall port_id=0

Token #
Token pragma
Token FCUDA
Token PORTMERGE
Token remove_port_name=gpuWall
Token port_id=0
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
attached=1

PreAnnotation: #pragma FCUDA PORTMERGE remove_port_name=gpuSrc port_id=0

Token #
Token pragma
Token FCUDA
Token PORTMERGE
Token remove_port_name=gpuSrc
Token port_id=0
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
attached=1

PreAnnotation: #pragma FCUDA PORTMERGE remove_port_name=gpuResults port_id=0

Token #
Token pragma
Token FCUDA
Token PORTMERGE
Token remove_port_name=gpuResults
Token port_id=0
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
attached=1

DeclarationStatement: #pragma FCUDA COMPUTE begin name=fetch cores=[1] unroll=1 mpart=1 array_split=[result] ;

PreAnnotation: #pragma FCUDA COMPUTE begin name=fetch cores=[1] unroll=1 mpart=1 array_split=[result]

Token #
Token pragma
Token FCUDA
Token COMPUTE
Token begin
Token name=fetch
Token cores=[1]
Token unroll=1
Token mpart=1
Token array_split=[result]
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
attached=0

Parent: DeclarationStatement 

DeclarationStatement: #pragma FCUDA COMPUTE end name=fetch cores=[1] unroll=1 mpart=1 array_split=[result] ;

PreAnnotation: #pragma FCUDA COMPUTE end name=fetch cores=[1] unroll=1 mpart=1 array_split=[result]

Token #
Token pragma
Token FCUDA
Token COMPUTE
Token end
Token name=fetch
Token cores=[1]
Token unroll=1
Token mpart=1
Token array_split=[result]
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
attached=0

Parent: DeclarationStatement 

DeclarationStatement: #pragma FCUDA COMPUTE cores=[1] begin name=compute unroll=1 mpart=1 array_split=[result] ;

PreAnnotation: #pragma FCUDA COMPUTE cores=[1] begin name=compute unroll=1 mpart=1 array_split=[result]

Token #
Token pragma
Token FCUDA
Token COMPUTE
Token cores=[1]
Token begin
Token name=compute
Token unroll=1
Token mpart=1
Token array_split=[result]
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
attached=0

Parent: DeclarationStatement 

DeclarationStatement: #pragma FCUDA COMPUTE cores=[1] end name=compute unroll=1 mpart=1 array_split=[result] ;

PreAnnotation: #pragma FCUDA COMPUTE cores=[1] end name=compute unroll=1 mpart=1 array_split=[result]

Token #
Token pragma
Token FCUDA
Token COMPUTE
Token cores=[1]
Token end
Token name=compute
Token unroll=1
Token mpart=1
Token array_split=[result]
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
attached=0

Parent: DeclarationStatement 

DeclarationStatement: #pragma FCUDA COMPUTE cores=[1] begin name=write unroll=1 mpart=1 array_split=[result] ;

PreAnnotation: #pragma FCUDA COMPUTE cores=[1] begin name=write unroll=1 mpart=1 array_split=[result]

Token #
Token pragma
Token FCUDA
Token COMPUTE
Token cores=[1]
Token begin
Token name=write
Token unroll=1
Token mpart=1
Token array_split=[result]
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
attached=0

Parent: DeclarationStatement 

DeclarationStatement: #pragma FCUDA COMPUTE cores=[1] end name=write unroll=1 mpart=1 array_split=[result] ;

PreAnnotation: #pragma FCUDA COMPUTE cores=[1] end name=write unroll=1 mpart=1 array_split=[result]

Token #
Token pragma
Token FCUDA
Token COMPUTE
Token cores=[1]
Token end
Token name=write
Token unroll=1
Token mpart=1
Token array_split=[result]
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
attached=0

Parent: DeclarationStatement 

[AnnotParser] end in 0.03 seconds

*** Before Any Passes  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int bx = blockIdx.x;
int tx = threadIdx.x;
int small_block_cols = (256-((iteration*1)*2));
int blkX = ((small_block_cols*bx)-border);
int blkXmax = ((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
int validXmin;
int validXmax;
int i, computed;
if ((((blkX+((int)tx))>=0)&&((blkX+((int)tx))<=(cols-1))))
{
prev[tx]=gpuSrc[(blkX+tx)];
}
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
for (i=0; i<iteration; i ++ )
{
int xidx = (blkX+tx);
int isValid = ((((int)tx)>=validXmin)&&(((int)tx)<=validXmax));
int W = (tx-1);
int E = (tx+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed=0;
if ((((((int)tx)>=(i+1))&&(((int)tx)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed=1;
left=prev[W];
up=prev[tx];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[tx]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed)
{
prev[tx]=result[tx];
}
__syncthreads();
}
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
if (computed)
{
gpuResults[(blkX+tx)]=result[tx];
}
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[LinkSymbol] 103 updates in 0.00 seconds
[AnnotParser] begin
[AnnotParser] end in 0.00 seconds

*** After AnnotationParser  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int bx = blockIdx.x;
int tx = threadIdx.x;
int small_block_cols = (256-((iteration*1)*2));
int blkX = ((small_block_cols*bx)-border);
int blkXmax = ((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
int validXmin;
int validXmax;
int i, computed;
if ((((blkX+((int)tx))>=0)&&((blkX+((int)tx))<=(cols-1))))
{
prev[tx]=gpuSrc[(blkX+tx)];
}
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
for (i=0; i<iteration; i ++ )
{
int xidx = (blkX+tx);
int isValid = ((((int)tx)>=validXmin)&&(((int)tx)<=validXmax));
int W = (tx-1);
int E = (tx+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed=0;
if ((((((int)tx)>=(i+1))&&(((int)tx)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed=1;
left=prev[W];
up=prev[tx];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[tx]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed)
{
prev[tx]=result[tx];
}
__syncthreads();
}
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
if (computed)
{
gpuResults[(blkX+tx)]=result[tx];
}
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[SingleDeclarator] begin
[SingleDeclarator] end in 0.00 seconds
[LinkSymbol] 103 updates in 0.00 seconds

*** After SingleDeclarator  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int bx = blockIdx.x;
int tx = threadIdx.x;
int small_block_cols = (256-((iteration*1)*2));
int blkX = ((small_block_cols*bx)-border);
int blkXmax = ((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
int validXmin;
int validXmax;
int i;
int computed;
if ((((blkX+((int)tx))>=0)&&((blkX+((int)tx))<=(cols-1))))
{
prev[tx]=gpuSrc[(blkX+tx)];
}
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
for (i=0; i<iteration; i ++ )
{
int xidx = (blkX+tx);
int isValid = ((((int)tx)>=validXmin)&&(((int)tx)<=validXmax));
int W = (tx-1);
int E = (tx+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed=0;
if ((((((int)tx)>=(i+1))&&(((int)tx)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed=1;
left=prev[W];
up=prev[tx];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[tx]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed)
{
prev[tx]=result[tx];
}
__syncthreads();
}
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
if (computed)
{
gpuResults[(blkX+tx)]=result[tx];
}
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================

*** After InlineDeviceFunctions  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int bx = blockIdx.x;
int tx = threadIdx.x;
int small_block_cols = (256-((iteration*1)*2));
int blkX = ((small_block_cols*bx)-border);
int blkXmax = ((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
int validXmin;
int validXmax;
int i;
int computed;
if ((((blkX+((int)tx))>=0)&&((blkX+((int)tx))<=(cols-1))))
{
prev[tx]=gpuSrc[(blkX+tx)];
}
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
for (i=0; i<iteration; i ++ )
{
int xidx = (blkX+tx);
int isValid = ((((int)tx)>=validXmin)&&(((int)tx)<=validXmax));
int W = (tx-1);
int E = (tx+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed=0;
if ((((((int)tx)>=(i+1))&&(((int)tx)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed=1;
left=prev[W];
up=prev[tx];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[tx]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed)
{
prev[tx]=result[tx];
}
__syncthreads();
}
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
if (computed)
{
gpuResults[(blkX+tx)]=result[tx];
}
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[SeparateInitializers] begin
[SeparateInitializers] examining procedure dynproc_kernel
[SeparateInitializers] end in 0.01 seconds
[LinkSymbol] 103 updates in 0.00 seconds

*** After SeparateInitializers  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int bx;
bx=blockIdx.x;
int tx;
tx=threadIdx.x;
int small_block_cols;
small_block_cols=(256-((iteration*1)*2));
int blkX;
blkX=((small_block_cols*bx)-border);
int blkXmax;
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
int validXmin;
int validXmax;
int i;
int computed;
if ((((blkX+((int)tx))>=0)&&((blkX+((int)tx))<=(cols-1))))
{
prev[tx]=gpuSrc[(blkX+tx)];
}
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
for (i=0; i<iteration; i ++ )
{
int xidx;
xidx=(blkX+tx);
int isValid;
isValid=((((int)tx)>=validXmin)&&(((int)tx)<=validXmax));
int W;
W=(tx-1);
int E;
E=(tx+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed=0;
if ((((((int)tx)>=(i+1))&&(((int)tx)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed=1;
left=prev[W];
up=prev[tx];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[tx]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed)
{
prev[tx]=result[tx];
}
__syncthreads();
}
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
if (computed)
{
gpuResults[(blkX+tx)]=result[tx];
}
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[AnsiDeclarations] begin
[AnsiDeclarations] end in 0.00 seconds
[LinkSymbol] 103 updates in 0.00 seconds

*** After AnsiDeclarations  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int bx;
int tx;
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
int i;
int computed;
bx=blockIdx.x;
tx=threadIdx.x;
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*bx)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
if ((((blkX+((int)tx))>=0)&&((blkX+((int)tx))<=(cols-1))))
{
prev[tx]=gpuSrc[(blkX+tx)];
}
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
for (i=0; i<iteration; i ++ )
{
int xidx;
int isValid;
int W;
int E;
xidx=(blkX+tx);
isValid=((((int)tx)>=validXmin)&&(((int)tx)<=validXmax));
W=(tx-1);
E=(tx+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed=0;
if ((((((int)tx)>=(i+1))&&(((int)tx)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed=1;
left=prev[W];
up=prev[tx];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[tx]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed)
{
prev[tx]=result[tx];
}
__syncthreads();
}
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
if (computed)
{
gpuResults[(blkX+tx)]=result[tx];
}
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[StreamInsertion-FCUDA] begin
[StreamInsertion-FCUDA] examining procedure dynproc_kernel
[StreamInsertion-FCUDA] end in 0.01 seconds
[LinkSymbol] 103 updates in 0.00 seconds

*** After StreamInsertion  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int bx;
int tx;
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
int i;
int computed;
bx=blockIdx.x;
tx=threadIdx.x;
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*bx)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
if ((((blkX+((int)tx))>=0)&&((blkX+((int)tx))<=(cols-1))))
{
prev[tx]=gpuSrc[(blkX+tx)];
}
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
for (i=0; i<iteration; i ++ )
{
int xidx;
int isValid;
int W;
int E;
xidx=(blkX+tx);
isValid=((((int)tx)>=validXmin)&&(((int)tx)<=validXmax));
W=(tx-1);
E=(tx+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed=0;
if ((((((int)tx)>=(i+1))&&(((int)tx)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed=1;
left=prev[W];
up=prev[tx];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[tx]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed)
{
prev[tx]=result[tx];
}
__syncthreads();
}
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
if (computed)
{
gpuResults[(blkX+tx)]=result[tx];
}
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[RemoveThrDepLoops-FCUDA] begin
[RemoveThrDepLoops-FCUDA] examining procedure dynproc_kernel
mVar2Var:
{E=[E, cols, blkX, threadIdx, blkXmax, border, validXmax, small_block_cols, blockIdx, iteration], result=[E, index, cols, shortest, xidx, small_block_cols, validXmin, i, up, iteration, startStep, W, blkX, threadIdx, left, border, blkXmax, right, validXmax, blockIdx], cols=[], shortest=[E, cols, shortest, small_block_cols, validXmin, up, iteration, W, blkX, threadIdx, left, border, blkXmax, right, validXmax, blockIdx], gpuWall=[], xidx=[blkX, threadIdx, border, small_block_cols, blockIdx, iteration], validXmin=[blkX, border, small_block_cols, blockIdx, iteration], iteration=[], gpuSrc=[], W=[W, blkX, threadIdx, border, small_block_cols, validXmin, blockIdx, iteration], threadIdx=[], computed=[], left=[W, blkX, threadIdx, border, small_block_cols, validXmin, blockIdx, iteration], border=[], blkXmax=[blkX, border, small_block_cols, blockIdx, iteration], right=[E, cols, blkX, threadIdx, blkXmax, border, validXmax, small_block_cols, blockIdx, iteration], gpuResults=[blkX, threadIdx, border, small_block_cols, blockIdx, iteration], index=[startStep, cols, blkX, threadIdx, xidx, border, small_block_cols, blockIdx, iteration, i], small_block_cols=[iteration], i=[i], prev=[blkX, threadIdx, border, small_block_cols, blockIdx, iteration], up=[threadIdx], startStep=[], blkX=[border, small_block_cols, blockIdx, iteration], isValid=[cols, blkX, threadIdx, border, blkXmax, validXmax, small_block_cols, validXmin, blockIdx, iteration], validXmax=[cols, blkX, border, blkXmax, small_block_cols, blockIdx, iteration], blockIdx=[]}
[RemoveThrDepLoops-FCUDA] end in 0.02 seconds
[LinkSymbol] 117 updates in 0.00 seconds

*** After RemoveThrDepLoops  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
int i;
int computed;
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
for (i=0; i<iteration; i ++ )
{
int xidx;
int isValid;
int W;
int E;
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed)
{
prev[threadIdx.x]=result[threadIdx.x];
}
__syncthreads();
}
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
if (computed)
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[MakeArraysInCompute-FCUDA] begin
[MakeArraysInCompute-FCUDA] examining procedure dynproc_kernel
Statement: {
__shared__ int prev[256];
__shared__ int result[256];
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
int i;
int computed;
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
for (i=0; i<iteration; i ++ )
{
int xidx;
int isValid;
int W;
int E;
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed)
{
prev[threadIdx.x]=result[threadIdx.x];
}
__syncthreads();
}
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
if (computed)
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}
Statement: __shared__ int prev[256];
Statement: __shared__ int result[256];
Statement: int small_block_cols;
Statement: int blkX;
Statement: int blkXmax;
Statement: int validXmin;
Statement: int validXmax;
Statement: int i;
Statement: int computed;
Statement: small_block_cols=(256-((iteration*1)*2));
Statement: blkX=((small_block_cols*blockIdx.x)-border);
Statement: blkXmax=((blkX+256)-1);
Statement: #pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
Statement: if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
Inside compute: if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
Statement: #pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
Statement: __syncthreads();
Statement: validXmin=((blkX<0) ? ( - blkX) : 0);
Statement: validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
Statement: #pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
Statement: for (i=0; i<iteration; i ++ )
{
int xidx;
int isValid;
int W;
int E;
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed)
{
prev[threadIdx.x]=result[threadIdx.x];
}
__syncthreads();
}
Inside compute: for (i=0; i<iteration; i ++ )
{
int xidx;
int isValid;
int W;
int E;
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed)
{
prev[threadIdx.x]=result[threadIdx.x];
}
__syncthreads();
}
Statement: #pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
IDEXPR E
[MakeArrays]: E is scalar:true used:false
IDEXPR index
[MakeArrays]: index is scalar:true used:false
IDEXPR W
[MakeArrays]: W is scalar:true used:false
IDEXPR shortest
[MakeArrays]: shortest is scalar:true used:false
IDEXPR computed
Variable: computed used outside compute core: if (computed)
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
[MakeArrays]: computed is scalar:true used:true
IDEXPR xidx
[MakeArrays]: xidx is scalar:true used:false
IDEXPR left
[MakeArrays]: left is scalar:true used:false
IDEXPR right
[MakeArrays]: right is scalar:true used:false
IDEXPR isValid
[MakeArrays]: isValid is scalar:true used:false
IDEXPR up
[MakeArrays]: up is scalar:true used:false
IDEXPR i
[MakeArrays]: i is scalar:true used:false
Variable to convert: computed
Statement: #pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
Statement: if (computed)
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
Inside compute: if (computed)
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
Statement: #pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
Variable to convert: computed
Statement: {
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
Statement: i=0;
Statement: {
int xidx;
int isValid;
int W;
int E;
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed)
{
prev[threadIdx.x]=result[threadIdx.x];
}
__syncthreads();
}
Statement: {
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
Statement: prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
Statement: int xidx;
Statement: int isValid;
Statement: int W;
Statement: int E;
Statement: xidx=(blkX+threadIdx.x);
Statement: isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
Statement: W=(threadIdx.x-1);
Statement: E=(threadIdx.x+1);
Statement: W=((W<validXmin) ? validXmin : W);
Statement: E=((E>validXmax) ? validXmax : E);
Statement: computed=0;
Statement: if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
Statement: __syncthreads();
Statement: if ((i==(iteration-1)))
{
break;
}
Statement: if (computed)
{
prev[threadIdx.x]=result[threadIdx.x];
}
Statement: __syncthreads();
Statement: gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
Statement: {
int left;
int up;
int right;
int shortest;
int index;
computed=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
Statement: {
break;
}
Statement: {
prev[threadIdx.x]=result[threadIdx.x];
}
Statement: int left;
Statement: int up;
Statement: int right;
Statement: int shortest;
Statement: int index;
Statement: computed=1;
Statement: left=prev[W];
Statement: up=prev[threadIdx.x];
Statement: right=prev[E];
Statement: shortest=((left<=up) ? left : up);
Statement: shortest=((shortest<=right) ? shortest : right);
Statement: index=((cols*(startStep+i))+xidx);
Statement: result[threadIdx.x]=(shortest+gpuWall[index]);
Statement: break;
Statement: prev[threadIdx.x]=result[threadIdx.x];
[MakeArraysInCompute-FCUDA] end in 0.02 seconds
[LinkSymbol] 117 updates in 0.00 seconds

*** After MakeArraysInCompute  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
int i;
__shared__ int computed_block[BLOCKDIM_X_dynproc_kernel];
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
for (i=0; i<iteration; i ++ )
{
int xidx;
int isValid;
int W;
int E;
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
__syncthreads();
}
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[SplitFcudaTasks-FCUDA] begin
[SplitFcudaTasks-FCUDA] examining procedure dynproc_kernel
BRAM:prev  specs: [[256]] size:1
BRAM:result  specs: [[256]] size:1
BRAM:computed_block  specs: [[BLOCKDIM_X_dynproc_kernel]] size:1

 ... Preprocessing pragma: 
	#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
Creating new FcudaCoreData for core: dynproc_kernel_fetch()

 ... Preprocessing pragma: 
	#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 

 ... Preprocessing pragma: 
	#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
Creating new FcudaCoreData for core: dynproc_kernel_compute()

 ... Preprocessing pragma: 
	#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 

 ... Preprocessing pragma: 
	#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
Creating new FcudaCoreData for core: dynproc_kernel_write()

 ... Preprocessing pragma: 
	#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
fcudaCores (splitTasks-start):
[dynproc_kernel_fetch(), dynproc_kernel_compute(), dynproc_kernel_write()]
coreNames: 
[dynproc_kernel_compute(), dynproc_kernel_fetch(), dynproc_kernel_write()]
Checking Annotation Statement: [#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true ]
FCUDA compute begin=true level=1
Task stmt: if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
of type: class cetus.hir.IfStatement
Task stmt: #pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
of type: class cetus.hir.AnnotationStatement
Checking Annotation Statement: [#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false ]
FCUDA compute begin=false level=1
Task use set: [blkX, cols, gpuSrc[(blkX+threadIdx.x)], threadIdx.x]
Task def set: [prev[threadIdx.x]]
Task maydef set: []
tmpExpr: blkX
  of class: class cetus.hir.Identifier
decl: int blkX
defStmt: none
tmpExpr: cols
  of class: class cetus.hir.Identifier
decl: int cols
defStmt: none
tmpExpr: gpuSrc[(blkX+threadIdx.x)]
  of class: class cetus.hir.ArrayAccess
decl: int * gpuSrc
defStmt: none
tmpExpr: prev[threadIdx.x]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ int prev[256]
defStmt: none
tmpExpr: threadIdx.x
  of class: class cetus.hir.AccessExpression
WARNING: currently do not handle RHS elements of AccessExpression threadIdx.x in buildTaskVars()
taskArgs: [enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev]
taskDecls: [int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256]]
defStmts: []
Checking Annotation Statement: [#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true ]
FCUDA compute begin=true level=1
Task stmt: for (i=0; i<iteration; i ++ )
{
int xidx;
int isValid;
int W;
int E;
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
__syncthreads();
}
of type: class cetus.hir.ForLoop
Task stmt: #pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
of type: class cetus.hir.AnnotationStatement
Checking Annotation Statement: [#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false ]
FCUDA compute begin=false level=1
Task use set: [E, W, blkX, cols, computed_block[threadIdx.x], gpuWall[index], i, index, isValid, iteration, left, prev[E], prev[W], prev[threadIdx.x], result[threadIdx.x], right, shortest, startStep, threadIdx.x, up, validXmax, validXmin, xidx]
Task def set: [E, W, computed_block[threadIdx.x], i, index, isValid, left, prev[threadIdx.x], result[threadIdx.x], right, shortest, up, xidx]
Task maydef set: []
tmpExpr: E
  of class: class cetus.hir.Identifier
tmpExpr: W
  of class: class cetus.hir.Identifier
tmpExpr: blkX
  of class: class cetus.hir.Identifier
decl: int blkX
defStmt: none
tmpExpr: cols
  of class: class cetus.hir.Identifier
decl: int cols
defStmt: none
tmpExpr: computed_block[threadIdx.x]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel]
defStmt: none
tmpExpr: gpuWall[index]
  of class: class cetus.hir.ArrayAccess
decl: int * gpuWall
defStmt: none
tmpExpr: i
  of class: class cetus.hir.Identifier
decl: int i
defStmt: none
tmpExpr: index
  of class: class cetus.hir.Identifier
tmpExpr: isValid
  of class: class cetus.hir.Identifier
tmpExpr: iteration
  of class: class cetus.hir.Identifier
decl: int iteration
defStmt: none
tmpExpr: left
  of class: class cetus.hir.Identifier
tmpExpr: prev[E]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ int prev[256]
defStmt: none
tmpExpr: prev[W]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ int prev[256]
defStmt: none
tmpExpr: prev[threadIdx.x]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ int prev[256]
defStmt: none
tmpExpr: result[threadIdx.x]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ int result[256]
defStmt: none
tmpExpr: right
  of class: class cetus.hir.Identifier
tmpExpr: shortest
  of class: class cetus.hir.Identifier
tmpExpr: startStep
  of class: class cetus.hir.Identifier
decl: int startStep
defStmt: none
tmpExpr: threadIdx.x
  of class: class cetus.hir.AccessExpression
WARNING: currently do not handle RHS elements of AccessExpression threadIdx.x in buildTaskVars()
tmpExpr: up
  of class: class cetus.hir.Identifier
tmpExpr: validXmax
  of class: class cetus.hir.Identifier
decl: int validXmax
defStmt: none
tmpExpr: validXmin
  of class: class cetus.hir.Identifier
decl: int validXmin
defStmt: none
tmpExpr: xidx
  of class: class cetus.hir.Identifier
taskArgs: [enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, i, iteration, prev, result, startStep, validXmax, validXmin]
taskDecls: [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int i, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin]
defStmts: []
Checking Annotation Statement: [#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true ]
FCUDA compute begin=true level=1
Task stmt: if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
of type: class cetus.hir.IfStatement
Task stmt: #pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
of type: class cetus.hir.AnnotationStatement
Checking Annotation Statement: [#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false ]
FCUDA compute begin=false level=1
Task use set: [blkX, computed_block[threadIdx.x], result[threadIdx.x], threadIdx.x]
Task def set: [gpuResults[(blkX+threadIdx.x)]]
Task maydef set: []
tmpExpr: blkX
  of class: class cetus.hir.Identifier
decl: int blkX
defStmt: none
tmpExpr: computed_block[threadIdx.x]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel]
defStmt: none
tmpExpr: gpuResults[(blkX+threadIdx.x)]
  of class: class cetus.hir.ArrayAccess
decl: int * gpuResults
defStmt: none
tmpExpr: result[threadIdx.x]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ int result[256]
defStmt: none
tmpExpr: threadIdx.x
  of class: class cetus.hir.AccessExpression
WARNING: currently do not handle RHS elements of AccessExpression threadIdx.x in buildTaskVars()
taskArgs: [enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result]
taskDecls: [int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256]]
defStmts: []
[SplitFcudaTasks-FCUDA] end in 0.02 seconds
[LinkSymbol] 145 updates in 0.00 seconds

*** After SplitFcudaTasks  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int i, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
if (enableSignal_compute)
{
for (i=0; i<iteration; i ++ )
{
int xidx;
int isValid;
int W;
int E;
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
__syncthreads();
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
int i;
__shared__ int computed_block[BLOCKDIM_X_dynproc_kernel];
int enableSignal_fetch;
int enableSignal_compute;
int enableSignal_write;
dim3 blockIdx;
enableSignal_write=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_fetch=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, i, iteration, prev, result, startStep, validXmax, validXmin);
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result);
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[CleanKernelDecls-FCUDA] begin
[CleanKernelDecls-FCUDA] examining procedure dynproc_kernel
cur_level:0
Defs+Uses:[blockIdx, blockIdx.x, blockIdx.y, enableSignal_write, gridDim, gridDim.x, gridDim.y]
Defs+Uses:[blockIdx, blockIdx.x, blockIdx.y, enableSignal_compute, gridDim, gridDim.x, gridDim.y]
Defs+Uses:[blockIdx, blockIdx.x, blockIdx.y, enableSignal_fetch, gridDim, gridDim.x, gridDim.y]
Defs+Uses:[iteration, small_block_cols]
Defs+Uses:[blkX, blockIdx, blockIdx.x, border, small_block_cols]
Defs+Uses:[blkX, blkXmax]
Defs+Uses:[]
Defs+Uses:[blkX, blockDim, blockIdx, cols, dynproc_kernel_fetch, enableSignal_fetch, gpuSrc, gridDim, prev]
Defs+Uses:[]
Defs+Uses:[__syncthreads]
Defs+Uses:[blkX, validXmin]
Defs+Uses:[blkXmax, cols, validXmax]
Defs+Uses:[]
Defs+Uses:[blkX, blockDim, blockIdx, cols, computed_block, dynproc_kernel_compute, enableSignal_compute, gpuWall, gridDim, i, iteration, prev, result, startStep, validXmax, validXmin]
Defs+Uses:[]
Defs+Uses:[]
Defs+Uses:[blkX, blockDim, blockIdx, computed_block, dynproc_kernel_write, enableSignal_write, gpuResults, gridDim, result]
Defs+Uses:[]
cur_level:0
var2freqMap{blkX=6, blkXmax=2, computed_block=2, enableSignal_compute=2, enableSignal_fetch=2, enableSignal_write=2, i=1, prev=2, result=2, small_block_cols=2, validXmax=2, validXmin=2}
funcCallParams[blkX, blockDim, blockIdx, cols, computed_block, enableSignal_compute, enableSignal_fetch, enableSignal_write, gpuResults, gpuSrc, gpuWall, gridDim, i, iteration, prev, result, startStep, validXmax, validXmin]
fcall:dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev)
fcall:__syncthreads()
fcall:dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, i, iteration, prev, result, startStep, validXmax, validXmin)
-arg:i contains i
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int i, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin]
[CleanKernelDecls-FCUDA] end in 0.01 seconds
[LinkSymbol] 144 updates in 0.00 seconds

*** After CleanKernelDecls  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
int i;
if (enableSignal_compute)
{
for (i=0; i<iteration; i ++ )
{
int xidx;
int isValid;
int W;
int E;
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
__syncthreads();
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
__shared__ int computed_block[BLOCKDIM_X_dynproc_kernel];
int enableSignal_fetch;
int enableSignal_compute;
int enableSignal_write;
dim3 blockIdx;
enableSignal_write=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_fetch=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin);
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result);
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[SerializeThreads-MCUDA] begin
[SerializeThreads-MCUDA] examining procedure dynproc_kernel
[SerializeThreads-MCUDA] end in 0.00 seconds
[LinkSymbol] 144 updates in 0.00 seconds

*** After SerializeThreads  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
int i;
if (enableSignal_compute)
{
for (i=0; i<iteration; i ++ )
{
int xidx;
int isValid;
int W;
int E;
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
__syncthreads();
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
__syncthreads();
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
__shared__ int computed_block[BLOCKDIM_X_dynproc_kernel];
int enableSignal_fetch;
int enableSignal_compute;
int enableSignal_write;
dim3 blockIdx;
enableSignal_write=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_fetch=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin);
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result);
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[EnforceSyncs-MCUDA] begin
[EnforceSyncs-MCUDA] examining procedure dynproc_kernel
[EnforceSyncs-MCUDA] end in 0.00 seconds
[LinkSymbol] 144 updates in 0.00 seconds

*** After EnforceSyncs  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i=0;
}
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i ++ ;
}
__syncthreads();
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
__shared__ int computed_block[BLOCKDIM_X_dynproc_kernel];
int enableSignal_fetch;
int enableSignal_compute;
int enableSignal_write;
dim3 blockIdx;
enableSignal_write=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_fetch=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin);
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result);
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[PrivatizeScalarsInThreadLoops - FCUDA] begin
[PrivatizeScalarsInThreadLoops - FCUDA] examining procedure dynproc_kernel
THREADLOOP: {
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
THREADLOOP: {
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
THREADLOOP: {
i ++ ;
}
THREADLOOP: {
i=0;
}
THREADLOOP: {
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
DEFSET: [xidx, isValid, W, E, left, up, right, shortest, index]
Find UseSet of: {
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
USESET: [iteration, result[256], computed_block[BLOCKDIM_X_dynproc_kernel], <threadIdx.x>, i]
DEFSET WITHIN LOOP: []
REMAINSET: []
Find UseSet of: {
i ++ ;
}
USESET: [i]
DEFSET WITHIN LOOP: []
REMAINSET: []
Find UseSet of: {
i=0;
}
USESET: []
DEFSET WITHIN LOOP: []
REMAINSET: []
THREADLOOP: {
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
Scalars to be privatized: []
[PrivatizeScalarsInThreadLoops - FCUDA] end in 0.01 seconds
[LinkSymbol] 144 updates in 0.00 seconds

*** After PrivatizeScalarsInThreadLoop  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i=0;
}
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i ++ ;
}
__syncthreads();
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
__shared__ int computed_block[BLOCKDIM_X_dynproc_kernel];
int enableSignal_fetch;
int enableSignal_compute;
int enableSignal_write;
dim3 blockIdx;
enableSignal_write=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_fetch=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin);
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result);
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[UnrollThreadLoops-MCUDA] begin
[UnrollThreadLoops-MCUDA] examining procedure dynproc_kernel

[Unrolling] : dynproc_kernel_fetch
[Proc]: #pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}


[unrollFactor] 1

[Unrolling] : dynproc_kernel_compute
[Proc]: #pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i=0;
}
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i ++ ;
}
__syncthreads();
}
}
}


[unrollFactor] 1

[Unrolling] : dynproc_kernel_write
[Proc]: #pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}


[unrollFactor] 1
mUnrolledIDs: 
{}
[UnrollThreadLoops-MCUDA] end in 0.00 seconds
[LinkSymbol] 144 updates in 0.00 seconds

*** After UnrollThreadLoops  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i=0;
}
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i ++ ;
}
__syncthreads();
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
__shared__ int computed_block[BLOCKDIM_X_dynproc_kernel];
int enableSignal_fetch;
int enableSignal_compute;
int enableSignal_write;
dim3 blockIdx;
enableSignal_write=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_fetch=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin);
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result);
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[PartitionArrays-MCUDA] begin
[PartitionArrays-MCUDA] examining procedure dynproc_kernel
[numDims]1
[Memory partition] : dynproc_kernel_fetch

[Proc]: #pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}


[mempartFactor]1
[numDims]1
[Memory partition] : dynproc_kernel_compute

[Proc]: #pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i=0;
}
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i ++ ;
}
__syncthreads();
}
}
}


[mempartFactor]1
[numDims]1
[Memory partition] : dynproc_kernel_write

[Proc]: #pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}


[mempartFactor]1
[Memory partition] : dynproc_kernel

HAA 1 {
__shared__ int prev[256];
__shared__ int result[256];
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
__shared__ int computed_block[BLOCKDIM_X_dynproc_kernel];
int enableSignal_fetch;
int enableSignal_compute;
int enableSignal_write;
dim3 blockIdx;
enableSignal_write=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_fetch=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin);
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result);
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}
[PartitionArrays-MCUDA] end in 0.00 seconds
[LinkSymbol] 144 updates in 0.00 seconds

*** After PartitionArrays  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i=0;
}
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i ++ ;
}
__syncthreads();
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
__shared__ int computed_block[BLOCKDIM_X_dynproc_kernel];
int enableSignal_fetch;
int enableSignal_compute;
int enableSignal_write;
dim3 blockIdx;
enableSignal_write=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_fetch=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin);
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result);
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[IfSplitPass-FCUDA] begin
[IfSplitPass-FCUDA] examining procedure dynproc_kernel
fcudaCores:
[dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev), dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin), dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result)]
coreNames: 
[dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin), dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev), dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result)]
Handling control flow for dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev)
mCurrEnableSignal: enableSignal_fetch
Handling control flow for dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin)
mCurrEnableSignal: enableSignal_compute
Handling control flow for dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result)
mCurrEnableSignal: enableSignal_write
[IfSplitPass-FCUDA] end in 0.00 seconds
[LinkSymbol] 144 updates in 0.00 seconds

*** After IfSplitPass  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i=0;
}
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i ++ ;
}
__syncthreads();
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border)
{
__shared__ int prev[256];
__shared__ int result[256];
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
__shared__ int computed_block[BLOCKDIM_X_dynproc_kernel];
int enableSignal_fetch;
int enableSignal_compute;
int enableSignal_write;
dim3 blockIdx;
enableSignal_write=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_fetch=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin);
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result);
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
}



===========================================
[WrapBlockIdxLoop-FCUDA] begin
[WrapBlockIdxLoop-FCUDA] examining procedure dynproc_kernel
[WrapBlockIdxLoop-FCUDA] end in 0.00 seconds
[LinkSymbol] 144 updates in 0.00 seconds

*** After WrapBlockIdxLoop  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i=0;
}
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i ++ ;
}
__syncthreads();
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
__shared__ int prev[256];
__shared__ int result[256];
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
__shared__ int computed_block[BLOCKDIM_X_dynproc_kernel];
int enableSignal_fetch;
int enableSignal_compute;
int enableSignal_write;
dim3 blockIdx;
blockIdx.y=0;
blockIdx.x=core_id;
while (1)
{
while (blockIdx.x>=gridDim.x)
{
blockIdx.x=(blockIdx.x-gridDim.x);
blockIdx.y=(blockIdx.y+1);
}
if ((blockIdx.y>=gridDim.y))
{
break;
}
enableSignal_write=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_fetch=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin);
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result);
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
blockIdx.x=(blockIdx.x+num_cores);
}
}



===========================================
[PipelineFCUDACores-FCUDA] begin
[PipelineFCUDACores-FCUDA] examining procedure dynproc_kernel
Found only one Core in the Loop. Hence proceed to pipeline the next loop.
There is no WhileLoop which can be pipelined. Hence proceed to pipeline the BlockIDX loop.
Here is the set of functions for pipelining: 

dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev);
dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin);
dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result);
Before handleBRAMs coreNames: 
[dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin), dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev), dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result)]
 *** coreCall == origCall
 *** coreCall == origCall
 *** coreCall == origCall
 *** coreCall == origCall
 *** coreCall == origCall
 *** coreCall == origCall
After handleBRAMs coreNames: 
[dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin), dynproc_kernel_compute(enableSignal_compute_pipe_1, blockDim, gridDim, blockIdx_pipe_1, blkX_pipe_1, cols, computed_block_ping, gpuWall, iteration, prev_pong, result_ping, startStep, validXmax_pipe_1, validXmin_pipe_1), dynproc_kernel_compute(enableSignal_compute_pipe_1, blockDim, gridDim, blockIdx_pipe_1, blkX_pipe_1, cols, computed_block_pong, gpuWall, iteration, prev_ping, result_pong, startStep, validXmax_pipe_1, validXmin_pipe_1), dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev), dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev_ping), dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev_pong), dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result), dynproc_kernel_write(enableSignal_write_pipe_2, blockDim, gridDim, blockIdx_pipe_2, blkX_pipe_2, computed_block_ping, gpuResults, result_ping), dynproc_kernel_write(enableSignal_write_pipe_2, blockDim, gridDim, blockIdx_pipe_2, blkX_pipe_2, computed_block_pong, gpuResults, result_pong)]
After update coreNames: 
[dynproc_kernel_compute(enableSignal_compute_pipe_1, blockDim, gridDim, blockIdx_pipe_1, blkX_pipe_1, cols, computed_block_ping, gpuWall, iteration, prev_pong, result_ping, startStep, validXmax_pipe_1, validXmin_pipe_1), dynproc_kernel_compute(enableSignal_compute_pipe_1, blockDim, gridDim, blockIdx_pipe_1, blkX_pipe_1, cols, computed_block_pong, gpuWall, iteration, prev_ping, result_pong, startStep, validXmax_pipe_1, validXmin_pipe_1), dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev_ping), dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev_pong), dynproc_kernel_write(enableSignal_write_pipe_2, blockDim, gridDim, blockIdx_pipe_2, blkX_pipe_2, computed_block_ping, gpuResults, result_ping), dynproc_kernel_write(enableSignal_write_pipe_2, blockDim, gridDim, blockIdx_pipe_2, blkX_pipe_2, computed_block_pong, gpuResults, result_pong)]
Scalars to duplicate : 
{blkX=dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result), enableSignal_compute=dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin), enableSignal_write=dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result), validXmin=dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin), validXmax=dynproc_kernel_compute(enableSignal_compute, blockDim, gridDim, blockIdx, blkX, cols, computed_block, gpuWall, iteration, prev, result, startStep, validXmax, validXmin), blockIdx=dynproc_kernel_write(enableSignal_write, blockDim, gridDim, blockIdx, blkX, computed_block, gpuResults, result)}
cData getNumCores#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
int enableSignal_fetch;
int enableSignal_compute;
int enableSignal_write;
dim3 blockIdx;
int pingpong_0;
__shared__ int prev_ping[256];
__shared__ int computed_block_ping[BLOCKDIM_X_dynproc_kernel];
__shared__ int prev_pong[256];
__shared__ int result_ping[256];
__shared__ int computed_block_pong[BLOCKDIM_X_dynproc_kernel];
__shared__ int result_pong[256];
int blkX_pipe_1;
int blkX_pipe_2;
int enableSignal_compute_pipe_1;
int enableSignal_write_pipe_1;
int enableSignal_write_pipe_2;
int validXmin_pipe_1;
int validXmax_pipe_1;
dim3 blockIdx_pipe_1;
dim3 blockIdx_pipe_2;
blockIdx.y=0;
blockIdx.x=core_id;
pingpong_0=0;
enableSignal_compute_pipe_1=0;
enableSignal_write_pipe_1=0;
enableSignal_write_pipe_2=0;
blockIdx_pipe_2.x=-1;
while (1)
{
while (blockIdx.x>=gridDim.x)
{
blockIdx.x=(blockIdx.x-gridDim.x);
blockIdx.y=(blockIdx.y+1);
}
if ((blockIdx.y>=gridDim.y))
{
break;
}
enableSignal_write=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_fetch=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
{
if ((pingpong_0==0))
{
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev_ping);
dynproc_kernel_compute(enableSignal_compute_pipe_1, blockDim, gridDim, blockIdx_pipe_1, blkX_pipe_1, cols, computed_block_ping, gpuWall, iteration, prev_pong, result_ping, startStep, validXmax_pipe_1, validXmin_pipe_1);
dynproc_kernel_write(enableSignal_write_pipe_2, blockDim, gridDim, blockIdx_pipe_2, blkX_pipe_2, computed_block_pong, gpuResults, result_pong);
pingpong_0=1;
}
else
{
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev_pong);
dynproc_kernel_compute(enableSignal_compute_pipe_1, blockDim, gridDim, blockIdx_pipe_1, blkX_pipe_1, cols, computed_block_pong, gpuWall, iteration, prev_ping, result_pong, startStep, validXmax_pipe_1, validXmin_pipe_1);
dynproc_kernel_write(enableSignal_write_pipe_2, blockDim, gridDim, blockIdx_pipe_2, blkX_pipe_2, computed_block_ping, gpuResults, result_ping);
pingpong_0=0;
}
}
blkX_pipe_2=blkX_pipe_1;
blkX_pipe_1=blkX;
enableSignal_compute_pipe_1=enableSignal_compute;
enableSignal_write_pipe_2=enableSignal_write_pipe_1;
enableSignal_write_pipe_1=enableSignal_write;
validXmin_pipe_1=validXmin;
validXmax_pipe_1=validXmax;
blockIdx_pipe_2=blockIdx_pipe_1;
blockIdx_pipe_1=blockIdx;
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
blockIdx.x=(blockIdx.x+num_cores);
}
}
1
[PipelineFCUDACores-FCUDA] end in 0.01 seconds
[LinkSymbol] 194 updates in 0.00 seconds

*** After PipelineFCUDACores  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i=0;
}
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i ++ ;
}
__syncthreads();
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
int enableSignal_fetch;
int enableSignal_compute;
int enableSignal_write;
dim3 blockIdx;
int pingpong_0;
__shared__ int prev_ping[256];
__shared__ int computed_block_ping[BLOCKDIM_X_dynproc_kernel];
__shared__ int prev_pong[256];
__shared__ int result_ping[256];
__shared__ int computed_block_pong[BLOCKDIM_X_dynproc_kernel];
__shared__ int result_pong[256];
int blkX_pipe_1;
int blkX_pipe_2;
int enableSignal_compute_pipe_1;
int enableSignal_write_pipe_1;
int enableSignal_write_pipe_2;
int validXmin_pipe_1;
int validXmax_pipe_1;
dim3 blockIdx_pipe_1;
dim3 blockIdx_pipe_2;
blockIdx.y=0;
blockIdx.x=core_id;
pingpong_0=0;
enableSignal_compute_pipe_1=0;
enableSignal_write_pipe_1=0;
enableSignal_write_pipe_2=0;
blockIdx_pipe_2.x=-1;
while (1)
{
while (blockIdx.x>=gridDim.x)
{
blockIdx.x=(blockIdx.x-gridDim.x);
blockIdx.y=(blockIdx.y+1);
}
if ((blockIdx.y>=(gridDim.y+((2*num_cores)+core_id))))
{
break;
}
enableSignal_write=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_fetch=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
{
if ((pingpong_0==0))
{
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev_ping);
dynproc_kernel_compute(enableSignal_compute_pipe_1, blockDim, gridDim, blockIdx_pipe_1, blkX_pipe_1, cols, computed_block_ping, gpuWall, iteration, prev_pong, result_ping, startStep, validXmax_pipe_1, validXmin_pipe_1);
dynproc_kernel_write(enableSignal_write_pipe_2, blockDim, gridDim, blockIdx_pipe_2, blkX_pipe_2, computed_block_pong, gpuResults, result_pong);
pingpong_0=1;
}
else
{
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev_pong);
dynproc_kernel_compute(enableSignal_compute_pipe_1, blockDim, gridDim, blockIdx_pipe_1, blkX_pipe_1, cols, computed_block_pong, gpuWall, iteration, prev_ping, result_pong, startStep, validXmax_pipe_1, validXmin_pipe_1);
dynproc_kernel_write(enableSignal_write_pipe_2, blockDim, gridDim, blockIdx_pipe_2, blkX_pipe_2, computed_block_ping, gpuResults, result_ping);
pingpong_0=0;
}
}
blkX_pipe_2=blkX_pipe_1;
blkX_pipe_1=blkX;
enableSignal_compute_pipe_1=enableSignal_compute;
enableSignal_write_pipe_2=enableSignal_write_pipe_1;
enableSignal_write_pipe_1=enableSignal_write;
validXmin_pipe_1=validXmin;
validXmax_pipe_1=validXmax;
blockIdx_pipe_2=blockIdx_pipe_1;
blockIdx_pipe_1=blockIdx;
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
blockIdx.x=(blockIdx.x+num_cores);
}
}



===========================================
[DuplicateForFCUDA-FCUDA] begin
[DuplicateForFCUDA-FCUDA] examining procedure dynproc_kernel
cData getNumCores#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int small_block_cols;
int blkX;
int blkXmax;
int validXmin;
int validXmax;
int enableSignal_fetch;
int enableSignal_compute;
int enableSignal_write;
dim3 blockIdx;
int pingpong_0;
__shared__ int prev_ping[256];
__shared__ int computed_block_ping[BLOCKDIM_X_dynproc_kernel];
__shared__ int prev_pong[256];
__shared__ int result_ping[256];
__shared__ int computed_block_pong[BLOCKDIM_X_dynproc_kernel];
__shared__ int result_pong[256];
int blkX_pipe_1;
int blkX_pipe_2;
int enableSignal_compute_pipe_1;
int enableSignal_write_pipe_1;
int enableSignal_write_pipe_2;
int validXmin_pipe_1;
int validXmax_pipe_1;
dim3 blockIdx_pipe_1;
dim3 blockIdx_pipe_2;
blockIdx.y=0;
blockIdx.x=core_id;
pingpong_0=0;
enableSignal_compute_pipe_1=0;
enableSignal_write_pipe_1=0;
enableSignal_write_pipe_2=0;
blockIdx_pipe_2.x=-1;
while (1)
{
while (blockIdx.x>=gridDim.x)
{
blockIdx.x=(blockIdx.x-gridDim.x);
blockIdx.y=(blockIdx.y+1);
}
if ((blockIdx.y>=(gridDim.y+((2*num_cores)+core_id))))
{
break;
}
enableSignal_write=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
enableSignal_fetch=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
small_block_cols=(256-((iteration*1)*2));
blkX=((small_block_cols*blockIdx.x)-border);
blkXmax=((blkX+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
__syncthreads();
validXmin=((blkX<0) ? ( - blkX) : 0);
validXmax=((blkXmax>(cols-1)) ? ((256-1)-((blkXmax-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=true cores=[1] unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
{
if ((pingpong_0==0))
{
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev_ping);
dynproc_kernel_compute(enableSignal_compute_pipe_1, blockDim, gridDim, blockIdx_pipe_1, blkX_pipe_1, cols, computed_block_ping, gpuWall, iteration, prev_pong, result_ping, startStep, validXmax_pipe_1, validXmin_pipe_1);
dynproc_kernel_write(enableSignal_write_pipe_2, blockDim, gridDim, blockIdx_pipe_2, blkX_pipe_2, computed_block_pong, gpuResults, result_pong);
pingpong_0=1;
}
else
{
dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev_pong);
dynproc_kernel_compute(enableSignal_compute_pipe_1, blockDim, gridDim, blockIdx_pipe_1, blkX_pipe_1, cols, computed_block_pong, gpuWall, iteration, prev_ping, result_pong, startStep, validXmax_pipe_1, validXmin_pipe_1);
dynproc_kernel_write(enableSignal_write_pipe_2, blockDim, gridDim, blockIdx_pipe_2, blkX_pipe_2, computed_block_ping, gpuResults, result_ping);
pingpong_0=0;
}
}
blkX_pipe_2=blkX_pipe_1;
blkX_pipe_1=blkX;
enableSignal_compute_pipe_1=enableSignal_compute;
enableSignal_write_pipe_2=enableSignal_write_pipe_1;
enableSignal_write_pipe_1=enableSignal_write;
validXmin_pipe_1=validXmin;
validXmax_pipe_1=validXmax;
blockIdx_pipe_2=blockIdx_pipe_1;
blockIdx_pipe_1=blockIdx;
#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=true unroll=1 begin=false 
blockIdx.x=(blockIdx.x+num_cores);
}
}
1
numParallelThreadBlocks:1
dynproc_kernel is inStreamProc: false
fcudaCores:
[dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev_ping), dynproc_kernel_compute(enableSignal_compute_pipe_1, blockDim, gridDim, blockIdx_pipe_1, blkX_pipe_1, cols, computed_block_ping, gpuWall, iteration, prev_pong, result_ping, startStep, validXmax_pipe_1, validXmin_pipe_1), dynproc_kernel_write(enableSignal_write_pipe_2, blockDim, gridDim, blockIdx_pipe_2, blkX_pipe_2, computed_block_pong, gpuResults, result_pong), dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev_pong), dynproc_kernel_compute(enableSignal_compute_pipe_1, blockDim, gridDim, blockIdx_pipe_1, blkX_pipe_1, cols, computed_block_pong, gpuWall, iteration, prev_ping, result_pong, startStep, validXmax_pipe_1, validXmin_pipe_1), dynproc_kernel_write(enableSignal_write_pipe_2, blockDim, gridDim, blockIdx_pipe_2, blkX_pipe_2, computed_block_ping, gpuResults, result_ping)]
coreNames: 
[dynproc_kernel_compute(enableSignal_compute_pipe_1, blockDim, gridDim, blockIdx_pipe_1, blkX_pipe_1, cols, computed_block_ping, gpuWall, iteration, prev_pong, result_ping, startStep, validXmax_pipe_1, validXmin_pipe_1), dynproc_kernel_compute(enableSignal_compute_pipe_1, blockDim, gridDim, blockIdx_pipe_1, blkX_pipe_1, cols, computed_block_pong, gpuWall, iteration, prev_ping, result_pong, startStep, validXmax_pipe_1, validXmin_pipe_1), dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev_ping), dynproc_kernel_fetch(enableSignal_fetch, blockDim, gridDim, blockIdx, blkX, cols, gpuSrc, prev_pong), dynproc_kernel_write(enableSignal_write_pipe_2, blockDim, gridDim, blockIdx_pipe_2, blkX_pipe_2, computed_block_ping, gpuResults, result_ping), dynproc_kernel_write(enableSignal_write_pipe_2, blockDim, gridDim, blockIdx_pipe_2, blkX_pipe_2, computed_block_pong, gpuResults, result_pong)]
------------------------
mVarsToDuplicate: []
mId2sym: {blkX_block0=blkX_block0, blkX_pipe_1_block0=blkX_pipe_1_block0, blkX_pipe_2_block0=blkX_pipe_2_block0, blkXmax_block0=blkXmax_block0, blockIdx_block0=blockIdx_block0, blockIdx_pipe_1_block0=blockIdx_pipe_1_block0, blockIdx_pipe_2_block0=blockIdx_pipe_2_block0, computed_block_ping_block0=computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel], computed_block_pong_block0=computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel], enableSignal_compute_block0=enableSignal_compute_block0, enableSignal_compute_pipe_1_block0=enableSignal_compute_pipe_1_block0, enableSignal_fetch_block0=enableSignal_fetch_block0, enableSignal_write_block0=enableSignal_write_block0, enableSignal_write_pipe_1_block0=enableSignal_write_pipe_1_block0, enableSignal_write_pipe_2_block0=enableSignal_write_pipe_2_block0, prev_ping_block0=prev_ping_block0[256], prev_pong_block0=prev_pong_block0[256], result_ping_block0=result_ping_block0[256], result_pong_block0=result_pong_block0[256], small_block_cols_block0=small_block_cols_block0, validXmax_block0=validXmax_block0, validXmax_pipe_1_block0=validXmax_pipe_1_block0, validXmin_block0=validXmin_block0, validXmin_pipe_1_block0=validXmin_pipe_1_block0}
mId2sym: {blkX_block0=blkX_block0, blkX_pipe_1_block0=blkX_pipe_1_block0, blkX_pipe_2_block0=blkX_pipe_2_block0, blkXmax_block0=blkXmax_block0, blockIdx_block0=blockIdx_block0, blockIdx_pipe_1_block0=blockIdx_pipe_1_block0, blockIdx_pipe_2_block0=blockIdx_pipe_2_block0, computed_block_ping_block0=computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel], computed_block_pong_block0=computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel], enableSignal_compute_block0=enableSignal_compute_block0, enableSignal_compute_pipe_1_block0=enableSignal_compute_pipe_1_block0, enableSignal_fetch_block0=enableSignal_fetch_block0, enableSignal_write_block0=enableSignal_write_block0, enableSignal_write_pipe_1_block0=enableSignal_write_pipe_1_block0, enableSignal_write_pipe_2_block0=enableSignal_write_pipe_2_block0, prev_ping_block0=prev_ping_block0[256], prev_pong_block0=prev_pong_block0[256], result_ping_block0=result_ping_block0[256], result_pong_block0=result_pong_block0[256], small_block_cols_block0=small_block_cols_block0, validXmax_block0=validXmax_block0, validXmax_pipe_1_block0=validXmax_pipe_1_block0, validXmin_block0=validXmin_block0, validXmin_pipe_1_block0=validXmin_pipe_1_block0}
mId2sym: {blkX_block0=blkX_block0, blkX_pipe_1_block0=blkX_pipe_1_block0, blkX_pipe_2_block0=blkX_pipe_2_block0, blkXmax_block0=blkXmax_block0, blockIdx_block0=blockIdx_block0, blockIdx_pipe_1_block0=blockIdx_pipe_1_block0, blockIdx_pipe_2_block0=blockIdx_pipe_2_block0, computed_block_ping_block0=computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel], computed_block_pong_block0=computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel], enableSignal_compute_block0=enableSignal_compute_block0, enableSignal_compute_pipe_1_block0=enableSignal_compute_pipe_1_block0, enableSignal_fetch_block0=enableSignal_fetch_block0, enableSignal_write_block0=enableSignal_write_block0, enableSignal_write_pipe_1_block0=enableSignal_write_pipe_1_block0, enableSignal_write_pipe_2_block0=enableSignal_write_pipe_2_block0, prev_ping_block0=prev_ping_block0[256], prev_pong_block0=prev_pong_block0[256], result_ping_block0=result_ping_block0[256], result_pong_block0=result_pong_block0[256], small_block_cols_block0=small_block_cols_block0, validXmax_block0=validXmax_block0, validXmax_pipe_1_block0=validXmax_pipe_1_block0, validXmin_block0=validXmin_block0, validXmin_pipe_1_block0=validXmin_pipe_1_block0}
mId2sym: {blkX_block0=blkX_block0, blkX_pipe_1_block0=blkX_pipe_1_block0, blkX_pipe_2_block0=blkX_pipe_2_block0, blkXmax_block0=blkXmax_block0, blockIdx_block0=blockIdx_block0, blockIdx_pipe_1_block0=blockIdx_pipe_1_block0, blockIdx_pipe_2_block0=blockIdx_pipe_2_block0, computed_block_ping_block0=computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel], computed_block_pong_block0=computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel], enableSignal_compute_block0=enableSignal_compute_block0, enableSignal_compute_pipe_1_block0=enableSignal_compute_pipe_1_block0, enableSignal_fetch_block0=enableSignal_fetch_block0, enableSignal_write_block0=enableSignal_write_block0, enableSignal_write_pipe_1_block0=enableSignal_write_pipe_1_block0, enableSignal_write_pipe_2_block0=enableSignal_write_pipe_2_block0, prev_ping_block0=prev_ping_block0[256], prev_pong_block0=prev_pong_block0[256], result_ping_block0=result_ping_block0[256], result_pong_block0=result_pong_block0[256], small_block_cols_block0=small_block_cols_block0, validXmax_block0=validXmax_block0, validXmax_pipe_1_block0=validXmax_pipe_1_block0, validXmin_block0=validXmin_block0, validXmin_pipe_1_block0=validXmin_pipe_1_block0}
mId2sym: {blkX_block0=blkX_block0, blkX_pipe_1_block0=blkX_pipe_1_block0, blkX_pipe_2_block0=blkX_pipe_2_block0, blkXmax_block0=blkXmax_block0, blockIdx_block0=blockIdx_block0, blockIdx_pipe_1_block0=blockIdx_pipe_1_block0, blockIdx_pipe_2_block0=blockIdx_pipe_2_block0, computed_block_ping_block0=computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel], computed_block_pong_block0=computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel], enableSignal_compute_block0=enableSignal_compute_block0, enableSignal_compute_pipe_1_block0=enableSignal_compute_pipe_1_block0, enableSignal_fetch_block0=enableSignal_fetch_block0, enableSignal_write_block0=enableSignal_write_block0, enableSignal_write_pipe_1_block0=enableSignal_write_pipe_1_block0, enableSignal_write_pipe_2_block0=enableSignal_write_pipe_2_block0, prev_ping_block0=prev_ping_block0[256], prev_pong_block0=prev_pong_block0[256], result_ping_block0=result_ping_block0[256], result_pong_block0=result_pong_block0[256], small_block_cols_block0=small_block_cols_block0, validXmax_block0=validXmax_block0, validXmax_pipe_1_block0=validXmax_pipe_1_block0, validXmin_block0=validXmin_block0, validXmin_pipe_1_block0=validXmin_pipe_1_block0}
mId2sym: {blkX_block0=blkX_block0, blkX_pipe_1_block0=blkX_pipe_1_block0, blkX_pipe_2_block0=blkX_pipe_2_block0, blkXmax_block0=blkXmax_block0, blockIdx_block0=blockIdx_block0, blockIdx_pipe_1_block0=blockIdx_pipe_1_block0, blockIdx_pipe_2_block0=blockIdx_pipe_2_block0, computed_block_ping_block0=computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel], computed_block_pong_block0=computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel], enableSignal_compute_block0=enableSignal_compute_block0, enableSignal_compute_pipe_1_block0=enableSignal_compute_pipe_1_block0, enableSignal_fetch_block0=enableSignal_fetch_block0, enableSignal_write_block0=enableSignal_write_block0, enableSignal_write_pipe_1_block0=enableSignal_write_pipe_1_block0, enableSignal_write_pipe_2_block0=enableSignal_write_pipe_2_block0, prev_ping_block0=prev_ping_block0[256], prev_pong_block0=prev_pong_block0[256], result_ping_block0=result_ping_block0[256], result_pong_block0=result_pong_block0[256], small_block_cols_block0=small_block_cols_block0, validXmax_block0=validXmax_block0, validXmax_pipe_1_block0=validXmax_pipe_1_block0, validXmin_block0=validXmin_block0, validXmin_pipe_1_block0=validXmin_pipe_1_block0}
mId2sym: {blkX_block0=blkX_block0, blkX_pipe_1_block0=blkX_pipe_1_block0, blkX_pipe_2_block0=blkX_pipe_2_block0, blkXmax_block0=blkXmax_block0, blockIdx_block0=blockIdx_block0, blockIdx_pipe_1_block0=blockIdx_pipe_1_block0, blockIdx_pipe_2_block0=blockIdx_pipe_2_block0, computed_block_ping_block0=computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel], computed_block_pong_block0=computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel], enableSignal_compute_block0=enableSignal_compute_block0, enableSignal_compute_pipe_1_block0=enableSignal_compute_pipe_1_block0, enableSignal_fetch_block0=enableSignal_fetch_block0, enableSignal_write_block0=enableSignal_write_block0, enableSignal_write_pipe_1_block0=enableSignal_write_pipe_1_block0, enableSignal_write_pipe_2_block0=enableSignal_write_pipe_2_block0, prev_ping_block0=prev_ping_block0[256], prev_pong_block0=prev_pong_block0[256], result_ping_block0=result_ping_block0[256], result_pong_block0=result_pong_block0[256], small_block_cols_block0=small_block_cols_block0, validXmax_block0=validXmax_block0, validXmax_pipe_1_block0=validXmax_pipe_1_block0, validXmin_block0=validXmin_block0, validXmin_pipe_1_block0=validXmin_pipe_1_block0}
... handleCompoundStatement finished! 
[DuplicateForFCUDA-FCUDA] end in 0.01 seconds
[LinkSymbol] 216 updates in 0.00 seconds

*** After DuplicateForFCUDA  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i=0;
}
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
i ++ ;
}
__syncthreads();
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int small_block_cols_block0;
int blkX_block0;
int blkXmax_block0;
int validXmin_block0;
int validXmax_block0;
int enableSignal_fetch_block0;
int enableSignal_compute_block0;
int enableSignal_write_block0;
dim3 blockIdx_block0;
int pingpong_0;
__shared__ int prev_ping_block0[256];
__shared__ int computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int prev_pong_block0[256];
__shared__ int result_ping_block0[256];
__shared__ int computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int result_pong_block0[256];
int blkX_pipe_1_block0;
int blkX_pipe_2_block0;
int enableSignal_compute_pipe_1_block0;
int enableSignal_write_pipe_1_block0;
int enableSignal_write_pipe_2_block0;
int validXmin_pipe_1_block0;
int validXmax_pipe_1_block0;
dim3 blockIdx_pipe_1_block0;
dim3 blockIdx_pipe_2_block0;
blockIdx_block0.y=0;
blockIdx_block0.x=((core_id*1)+0);
pingpong_0=0;
enableSignal_compute_pipe_1_block0=0;
enableSignal_write_pipe_1_block0=0;
enableSignal_write_pipe_2_block0=0;
blockIdx_pipe_2_block0.x=-1;
while (1)
{
while (blockIdx_block0.x>=gridDim.x)
{
blockIdx_block0.x=(blockIdx_block0.x-gridDim.x);
blockIdx_block0.y=(blockIdx_block0.y+1);
}
if ((blockIdx_block0.y>=(gridDim.y+((2*num_cores)+core_id))))
{
break;
}
enableSignal_write_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_compute_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_fetch_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
small_block_cols_block0=(256-((iteration*1)*2));
blkX_block0=((small_block_cols_block0*blockIdx_block0.x)-border);
blkXmax_block0=((blkX_block0+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=false begin=true 
__syncthreads();
validXmin_block0=((blkX_block0<0) ? ( - blkX_block0) : 0);
validXmax_block0=((blkXmax_block0>(cols-1)) ? ((256-1)-((blkXmax_block0-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=true begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=false cores=[1] begin=true 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=true cores=[1] begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=false cores=[1] begin=true 
{
if ((pingpong_0==0))
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_ping_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_ping_block0, gpuWall, iteration, prev_pong_block0, result_ping_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_pong_block0, gpuResults, result_pong_block0);
pingpong_0=1;
}
else
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_pong_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_pong_block0, gpuWall, iteration, prev_ping_block0, result_pong_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_ping_block0, gpuResults, result_ping_block0);
pingpong_0=0;
}
}
blkX_pipe_2_block0=blkX_pipe_1_block0;
blkX_pipe_1_block0=blkX_block0;
enableSignal_compute_pipe_1_block0=enableSignal_compute_block0;
enableSignal_write_pipe_2_block0=enableSignal_write_pipe_1_block0;
enableSignal_write_pipe_1_block0=enableSignal_write_block0;
validXmin_pipe_1_block0=validXmin_block0;
validXmax_pipe_1_block0=validXmax_block0;
blockIdx_pipe_2_block0=blockIdx_pipe_1_block0;
blockIdx_pipe_1_block0=blockIdx_block0;
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=true cores=[1] begin=false 
blockIdx_block0.x=(blockIdx_block0.x+(num_cores*1));
}
}



===========================================
[CleanThreadLoops-MCUDA] begin
[CleanThreadLoops-MCUDA] examining procedure dynproc_kernel
mVar2Var:
{result=[E, index, cols, shortest, xidx, validXmin, i, up, startStep, W, blkX, threadIdx, left, right, validXmax], E=[E, threadIdx, validXmax], cols=[], index=[startStep, cols, blkX, threadIdx, xidx, i], gpuWall=[], shortest=[E, W, shortest, threadIdx, left, right, validXmax, validXmin, up], xidx=[blkX, threadIdx], validXmin=[], prev=[threadIdx], up=[threadIdx], i=[i], iteration=[], W=[W, threadIdx, validXmin], startStep=[], blkX=[], blockDim=[], enableSignal_compute=[], threadIdx=[], computed_block=[threadIdx], left=[W, threadIdx, validXmin], validXmax=[], isValid=[threadIdx, validXmax, validXmin], right=[E, threadIdx, validXmax]}
mVar2Var:
{result=[E, index, cols, shortest, xidx, validXmin, i, up, startStep, W, blkX, threadIdx, left, right, validXmax], E=[E, threadIdx, validXmax], cols=[], index=[startStep, cols, blkX, threadIdx, xidx, i], gpuWall=[], shortest=[E, W, shortest, threadIdx, left, right, validXmax, validXmin, up], xidx=[blkX, threadIdx], validXmin=[], prev=[threadIdx], up=[threadIdx], i=[i], iteration=[], W=[W, threadIdx, validXmin], startStep=[], blkX=[], blockDim=[], enableSignal_compute=[], threadIdx=[], computed_block=[threadIdx], left=[W, threadIdx, validXmin], validXmax=[], isValid=[threadIdx, validXmax, validXmin], right=[E, threadIdx, validXmax]}
mVar2Var:
{result=[E, index, cols, shortest, xidx, validXmin, i, up, startStep, W, blkX, threadIdx, left, right, validXmax], E=[E, threadIdx, validXmax], cols=[], index=[startStep, cols, blkX, threadIdx, xidx, i], gpuWall=[], shortest=[E, W, shortest, threadIdx, left, right, validXmax, validXmin, up], xidx=[blkX, threadIdx], validXmin=[], prev=[threadIdx], up=[threadIdx], i=[i], iteration=[], W=[W, threadIdx, validXmin], startStep=[], blkX=[], blockDim=[], enableSignal_compute=[], threadIdx=[], computed_block=[threadIdx], left=[W, threadIdx, validXmin], validXmax=[], isValid=[threadIdx, validXmax, validXmin], right=[E, threadIdx, validXmax]}
mVar2Var:
{result=[E, index, cols, shortest, xidx, validXmin, i, up, startStep, W, blkX, threadIdx, left, right, validXmax], E=[E, threadIdx, validXmax], cols=[], index=[startStep, cols, blkX, threadIdx, xidx, i], gpuWall=[], shortest=[E, W, shortest, threadIdx, left, right, validXmax, validXmin, up], xidx=[blkX, threadIdx], validXmin=[], prev=[threadIdx], up=[threadIdx], i=[i], iteration=[], W=[W, threadIdx, validXmin], startStep=[], blkX=[], blockDim=[], enableSignal_compute=[], threadIdx=[], computed_block=[threadIdx], left=[W, threadIdx, validXmin], validXmax=[], isValid=[threadIdx, validXmax, validXmin], right=[E, threadIdx, validXmax]}
mVar2Var:
{result=[E, index, cols, shortest, xidx, validXmin, i, up, startStep, W, blkX, threadIdx, left, right, validXmax], E=[E, threadIdx, validXmax], cols=[], index=[startStep, cols, blkX, threadIdx, xidx, i], gpuWall=[], shortest=[E, W, shortest, threadIdx, left, right, validXmax, validXmin, up], xidx=[blkX, threadIdx], validXmin=[], prev=[threadIdx], up=[threadIdx], i=[i], iteration=[], W=[W, threadIdx, validXmin], startStep=[], blkX=[], blockDim=[], enableSignal_compute=[], threadIdx=[], computed_block=[threadIdx], left=[W, threadIdx, validXmin], validXmax=[], isValid=[threadIdx, validXmax, validXmin], right=[E, threadIdx, validXmax]}
mVar2Var:
{result=[E, index, cols, shortest, xidx, validXmin, i, up, startStep, W, blkX, threadIdx, left, right, validXmax], E=[E, threadIdx, validXmax], cols=[], index=[startStep, cols, blkX, threadIdx, xidx, i], gpuWall=[], shortest=[E, W, shortest, threadIdx, left, right, validXmax, validXmin, up], xidx=[blkX, threadIdx], validXmin=[], prev=[threadIdx], up=[threadIdx], i=[i], iteration=[], W=[W, threadIdx, validXmin], startStep=[], blkX=[], blockDim=[], enableSignal_compute=[], threadIdx=[], computed_block=[threadIdx], left=[W, threadIdx, validXmin], validXmax=[], isValid=[threadIdx, validXmax, validXmin], right=[E, threadIdx, validXmax]}
mVar2Var:
{result=[E, index, cols, shortest, xidx, validXmin, i, up, startStep, W, blkX, threadIdx, left, right, validXmax], E=[E, threadIdx, validXmax], cols=[], index=[startStep, cols, blkX, threadIdx, xidx, i], gpuWall=[], shortest=[E, W, shortest, threadIdx, left, right, validXmax, validXmin, up], xidx=[blkX, threadIdx], validXmin=[], prev=[threadIdx], up=[threadIdx], i=[i], iteration=[], W=[W, threadIdx, validXmin], startStep=[], blkX=[], blockDim=[], enableSignal_compute=[], threadIdx=[], computed_block=[threadIdx], left=[W, threadIdx, validXmin], validXmax=[], isValid=[threadIdx, validXmax, validXmin], right=[E, threadIdx, validXmax]}
mVar2Var:
{result=[E, index, cols, shortest, xidx, validXmin, i, up, startStep, W, blkX, threadIdx, left, right, validXmax], E=[E, threadIdx, validXmax], cols=[], index=[startStep, cols, blkX, threadIdx, xidx, i], gpuWall=[], shortest=[E, W, shortest, threadIdx, left, right, validXmax, validXmin, up], xidx=[blkX, threadIdx], validXmin=[], prev=[threadIdx], up=[threadIdx], i=[i], iteration=[], W=[W, threadIdx, validXmin], startStep=[], blkX=[], blockDim=[], enableSignal_compute=[], threadIdx=[], computed_block=[threadIdx], left=[W, threadIdx, validXmin], validXmax=[], isValid=[threadIdx, validXmax, validXmin], right=[E, threadIdx, validXmax]}
[CleanThreadLoops-MCUDA] end in 0.05 seconds
[LinkSymbol] 216 updates in 0.00 seconds

*** After CleanThreadLoops  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
i=0;
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
__syncthreads();
i ++ ;
__syncthreads();
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int small_block_cols_block0;
int blkX_block0;
int blkXmax_block0;
int validXmin_block0;
int validXmax_block0;
int enableSignal_fetch_block0;
int enableSignal_compute_block0;
int enableSignal_write_block0;
dim3 blockIdx_block0;
int pingpong_0;
__shared__ int prev_ping_block0[256];
__shared__ int computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int prev_pong_block0[256];
__shared__ int result_ping_block0[256];
__shared__ int computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int result_pong_block0[256];
int blkX_pipe_1_block0;
int blkX_pipe_2_block0;
int enableSignal_compute_pipe_1_block0;
int enableSignal_write_pipe_1_block0;
int enableSignal_write_pipe_2_block0;
int validXmin_pipe_1_block0;
int validXmax_pipe_1_block0;
dim3 blockIdx_pipe_1_block0;
dim3 blockIdx_pipe_2_block0;
blockIdx_block0.y=0;
blockIdx_block0.x=((core_id*1)+0);
pingpong_0=0;
enableSignal_compute_pipe_1_block0=0;
enableSignal_write_pipe_1_block0=0;
enableSignal_write_pipe_2_block0=0;
blockIdx_pipe_2_block0.x=-1;
while (1)
{
while (blockIdx_block0.x>=gridDim.x)
{
blockIdx_block0.x=(blockIdx_block0.x-gridDim.x);
blockIdx_block0.y=(blockIdx_block0.y+1);
}
if ((blockIdx_block0.y>=(gridDim.y+((2*num_cores)+core_id))))
{
break;
}
enableSignal_write_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_compute_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_fetch_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
small_block_cols_block0=(256-((iteration*1)*2));
blkX_block0=((small_block_cols_block0*blockIdx_block0.x)-border);
blkXmax_block0=((blkX_block0+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=false begin=true 
__syncthreads();
validXmin_block0=((blkX_block0<0) ? ( - blkX_block0) : 0);
validXmax_block0=((blkXmax_block0>(cols-1)) ? ((256-1)-((blkXmax_block0-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=true begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=false cores=[1] begin=true 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=true cores=[1] begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=false cores=[1] begin=true 
{
if ((pingpong_0==0))
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_ping_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_ping_block0, gpuWall, iteration, prev_pong_block0, result_ping_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_pong_block0, gpuResults, result_pong_block0);
pingpong_0=1;
}
else
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_pong_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_pong_block0, gpuWall, iteration, prev_ping_block0, result_pong_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_ping_block0, gpuResults, result_ping_block0);
pingpong_0=0;
}
}
blkX_pipe_2_block0=blkX_pipe_1_block0;
blkX_pipe_1_block0=blkX_block0;
enableSignal_compute_pipe_1_block0=enableSignal_compute_block0;
enableSignal_write_pipe_2_block0=enableSignal_write_pipe_1_block0;
enableSignal_write_pipe_1_block0=enableSignal_write_block0;
validXmin_pipe_1_block0=validXmin_block0;
validXmax_pipe_1_block0=validXmax_block0;
blockIdx_pipe_2_block0=blockIdx_pipe_1_block0;
blockIdx_pipe_1_block0=blockIdx_block0;
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=true cores=[1] begin=false 
blockIdx_block0.x=(blockIdx_block0.x+(num_cores*1));
}
}



===========================================
[KernelStateTransform-MCUDA] begin
[KernelStateTransform-MCUDA] examining procedure dynproc_kernel
>>> outside uses:
[blkX_block0, blkX_pipe_1_block0, blkX_pipe_2_block0, blkXmax_block0, blockIdx_block0, blockIdx_block0.x, blockIdx_block0.y, blockIdx_pipe_1_block0, blockIdx_pipe_2_block0, border, cols, computed_block_ping_block0, computed_block_pong_block0, enableSignal_compute_block0, enableSignal_compute_pipe_1_block0, enableSignal_fetch_block0, enableSignal_write_block0, enableSignal_write_pipe_1_block0, enableSignal_write_pipe_2_block0, gpuResults, gpuSrc, gpuWall, gridDim.x, gridDim.y, iteration, pingpong_0, prev_ping_block0, prev_pong_block0, result_ping_block0, result_pong_block0, small_block_cols_block0, startStep, validXmax_block0, validXmax_pipe_1_block0, validXmin_block0, validXmin_pipe_1_block0]
>>> handling: blkX_block0
>>> handling: blkX_pipe_1_block0
>>> handling: blkX_pipe_2_block0
>>> handling: blkXmax_block0
>>> handling: blockIdx_block0
>>> handling: blockIdx_block0
>>> handling: blockIdx_block0
>>> handling: blockIdx_pipe_1_block0
>>> handling: blockIdx_pipe_2_block0
>>> handling: border
>>> handling: cols
>>> handling: computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel]
>>> handling: computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel]
>>> handling: enableSignal_compute_block0
>>> handling: enableSignal_compute_pipe_1_block0
>>> handling: enableSignal_fetch_block0
>>> handling: enableSignal_write_block0
>>> handling: enableSignal_write_pipe_1_block0
>>> handling: enableSignal_write_pipe_2_block0
>>> handling: * gpuResults
>>> handling: * gpuSrc
>>> handling: * gpuWall
>>> handling: gridDim
>>> handling: gridDim
>>> handling: iteration
>>> handling: pingpong_0
>>> handling: prev_ping_block0[256]
>>> handling: prev_pong_block0[256]
>>> handling: result_ping_block0[256]
>>> handling: result_pong_block0[256]
>>> handling: small_block_cols_block0
>>> handling: startStep
>>> handling: validXmax_block0
>>> handling: validXmax_pipe_1_block0
>>> handling: validXmin_block0
>>> handling: validXmin_pipe_1_block0
transforming Decls
[KernelStateTransform-MCUDA] end in 0.01 seconds
[LinkSymbol] 216 updates in 0.00 seconds

*** After KernelStateTransform  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
i=0;
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
__syncthreads();
i ++ ;
__syncthreads();
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int small_block_cols_block0;
int blkX_block0;
int blkXmax_block0;
int validXmin_block0;
int validXmax_block0;
int enableSignal_fetch_block0;
int enableSignal_compute_block0;
int enableSignal_write_block0;
dim3 blockIdx_block0;
int pingpong_0;
__shared__ int prev_ping_block0[256];
__shared__ int computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int prev_pong_block0[256];
__shared__ int result_ping_block0[256];
__shared__ int computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int result_pong_block0[256];
int blkX_pipe_1_block0;
int blkX_pipe_2_block0;
int enableSignal_compute_pipe_1_block0;
int enableSignal_write_pipe_1_block0;
int enableSignal_write_pipe_2_block0;
int validXmin_pipe_1_block0;
int validXmax_pipe_1_block0;
dim3 blockIdx_pipe_1_block0;
dim3 blockIdx_pipe_2_block0;
blockIdx_block0.y=0;
blockIdx_block0.x=((core_id*1)+0);
pingpong_0=0;
enableSignal_compute_pipe_1_block0=0;
enableSignal_write_pipe_1_block0=0;
enableSignal_write_pipe_2_block0=0;
blockIdx_pipe_2_block0.x=-1;
while (1)
{
while (blockIdx_block0.x>=gridDim.x)
{
blockIdx_block0.x=(blockIdx_block0.x-gridDim.x);
blockIdx_block0.y=(blockIdx_block0.y+1);
}
if ((blockIdx_block0.y>=(gridDim.y+((2*num_cores)+core_id))))
{
break;
}
enableSignal_write_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_compute_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_fetch_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
small_block_cols_block0=(256-((iteration*1)*2));
blkX_block0=((small_block_cols_block0*blockIdx_block0.x)-border);
blkXmax_block0=((blkX_block0+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=false begin=true 
__syncthreads();
validXmin_block0=((blkX_block0<0) ? ( - blkX_block0) : 0);
validXmax_block0=((blkXmax_block0>(cols-1)) ? ((256-1)-((blkXmax_block0-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=true begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=false cores=[1] begin=true 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=true cores=[1] begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=false cores=[1] begin=true 
{
if ((pingpong_0==0))
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_ping_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_ping_block0, gpuWall, iteration, prev_pong_block0, result_ping_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_pong_block0, gpuResults, result_pong_block0);
pingpong_0=1;
}
else
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_pong_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_pong_block0, gpuWall, iteration, prev_ping_block0, result_pong_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_ping_block0, gpuResults, result_ping_block0);
pingpong_0=0;
}
}
blkX_pipe_2_block0=blkX_pipe_1_block0;
blkX_pipe_1_block0=blkX_block0;
enableSignal_compute_pipe_1_block0=enableSignal_compute_block0;
enableSignal_write_pipe_2_block0=enableSignal_write_pipe_1_block0;
enableSignal_write_pipe_1_block0=enableSignal_write_block0;
validXmin_pipe_1_block0=validXmin_block0;
validXmax_pipe_1_block0=validXmax_block0;
blockIdx_pipe_2_block0=blockIdx_pipe_1_block0;
blockIdx_pipe_1_block0=blockIdx_block0;
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=true cores=[1] begin=false 
blockIdx_block0.x=(blockIdx_block0.x+(num_cores*1));
}
}



===========================================
[CleanSyncFunc-MCUDA] begin
[CleanSyncFunc-MCUDA] examining procedure dynproc_kernel
[CleanSyncFunc-MCUDA] end in 0.00 seconds
[LinkSymbol] 213 updates in 0.00 seconds

*** After CleanSyncFunc  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
i=0;
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
i ++ ;
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuWall 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuSrc 
#pragma fcuda portmerge port_id=0 remove_port_name=gpuResults 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int small_block_cols_block0;
int blkX_block0;
int blkXmax_block0;
int validXmin_block0;
int validXmax_block0;
int enableSignal_fetch_block0;
int enableSignal_compute_block0;
int enableSignal_write_block0;
dim3 blockIdx_block0;
int pingpong_0;
__shared__ int prev_ping_block0[256];
__shared__ int computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int prev_pong_block0[256];
__shared__ int result_ping_block0[256];
__shared__ int computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int result_pong_block0[256];
int blkX_pipe_1_block0;
int blkX_pipe_2_block0;
int enableSignal_compute_pipe_1_block0;
int enableSignal_write_pipe_1_block0;
int enableSignal_write_pipe_2_block0;
int validXmin_pipe_1_block0;
int validXmax_pipe_1_block0;
dim3 blockIdx_pipe_1_block0;
dim3 blockIdx_pipe_2_block0;
blockIdx_block0.y=0;
blockIdx_block0.x=((core_id*1)+0);
pingpong_0=0;
enableSignal_compute_pipe_1_block0=0;
enableSignal_write_pipe_1_block0=0;
enableSignal_write_pipe_2_block0=0;
blockIdx_pipe_2_block0.x=-1;
while (1)
{
while (blockIdx_block0.x>=gridDim.x)
{
blockIdx_block0.x=(blockIdx_block0.x-gridDim.x);
blockIdx_block0.y=(blockIdx_block0.y+1);
}
if ((blockIdx_block0.y>=(gridDim.y+((2*num_cores)+core_id))))
{
break;
}
enableSignal_write_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_compute_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_fetch_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
small_block_cols_block0=(256-((iteration*1)*2));
blkX_block0=((small_block_cols_block0*blockIdx_block0.x)-border);
blkXmax_block0=((blkX_block0+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=false begin=true 
validXmin_block0=((blkX_block0<0) ? ( - blkX_block0) : 0);
validXmax_block0=((blkXmax_block0>(cols-1)) ? ((256-1)-((blkXmax_block0-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=true begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=false cores=[1] begin=true 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=true cores=[1] begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=false cores=[1] begin=true 
{
if ((pingpong_0==0))
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_ping_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_ping_block0, gpuWall, iteration, prev_pong_block0, result_ping_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_pong_block0, gpuResults, result_pong_block0);
pingpong_0=1;
}
else
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_pong_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_pong_block0, gpuWall, iteration, prev_ping_block0, result_pong_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_ping_block0, gpuResults, result_ping_block0);
pingpong_0=0;
}
}
blkX_pipe_2_block0=blkX_pipe_1_block0;
blkX_pipe_1_block0=blkX_block0;
enableSignal_compute_pipe_1_block0=enableSignal_compute_block0;
enableSignal_write_pipe_2_block0=enableSignal_write_pipe_1_block0;
enableSignal_write_pipe_1_block0=enableSignal_write_block0;
validXmin_pipe_1_block0=validXmin_block0;
validXmax_pipe_1_block0=validXmax_block0;
blockIdx_pipe_2_block0=blockIdx_pipe_1_block0;
blockIdx_pipe_1_block0=blockIdx_block0;
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=true cores=[1] begin=false 
blockIdx_block0.x=(blockIdx_block0.x+(num_cores*1));
}
}



===========================================
[GenWrapperSingleKernel - FCUDA] begin
[GenWrapperSingleKernel - FCUDA] examining procedure dynproc_kernel
[GenWrapperSingleKernel - FCUDA] end in 0.00 seconds
[LinkSymbol] 213 updates in 0.00 seconds

*** After GenWrapperSingleKernel  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
i=0;
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
i ++ ;
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int small_block_cols_block0;
int blkX_block0;
int blkXmax_block0;
int validXmin_block0;
int validXmax_block0;
int enableSignal_fetch_block0;
int enableSignal_compute_block0;
int enableSignal_write_block0;
dim3 blockIdx_block0;
int pingpong_0;
__shared__ int prev_ping_block0[256];
__shared__ int computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int prev_pong_block0[256];
__shared__ int result_ping_block0[256];
__shared__ int computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int result_pong_block0[256];
int blkX_pipe_1_block0;
int blkX_pipe_2_block0;
int enableSignal_compute_pipe_1_block0;
int enableSignal_write_pipe_1_block0;
int enableSignal_write_pipe_2_block0;
int validXmin_pipe_1_block0;
int validXmax_pipe_1_block0;
dim3 blockIdx_pipe_1_block0;
dim3 blockIdx_pipe_2_block0;
blockIdx_block0.y=0;
blockIdx_block0.x=((core_id*1)+0);
pingpong_0=0;
enableSignal_compute_pipe_1_block0=0;
enableSignal_write_pipe_1_block0=0;
enableSignal_write_pipe_2_block0=0;
blockIdx_pipe_2_block0.x=-1;
while (1)
{
while (blockIdx_block0.x>=gridDim.x)
{
blockIdx_block0.x=(blockIdx_block0.x-gridDim.x);
blockIdx_block0.y=(blockIdx_block0.y+1);
}
if ((blockIdx_block0.y>=(gridDim.y+((2*num_cores)+core_id))))
{
break;
}
enableSignal_write_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_compute_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_fetch_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
small_block_cols_block0=(256-((iteration*1)*2));
blkX_block0=((small_block_cols_block0*blockIdx_block0.x)-border);
blkXmax_block0=((blkX_block0+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=false begin=true 
validXmin_block0=((blkX_block0<0) ? ( - blkX_block0) : 0);
validXmax_block0=((blkXmax_block0>(cols-1)) ? ((256-1)-((blkXmax_block0-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=true begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=false cores=[1] begin=true 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=true cores=[1] begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=false cores=[1] begin=true 
{
if ((pingpong_0==0))
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_ping_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_ping_block0, gpuWall, iteration, prev_pong_block0, result_ping_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_pong_block0, gpuResults, result_pong_block0);
pingpong_0=1;
}
else
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_pong_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_pong_block0, gpuWall, iteration, prev_ping_block0, result_pong_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_ping_block0, gpuResults, result_ping_block0);
pingpong_0=0;
}
}
blkX_pipe_2_block0=blkX_pipe_1_block0;
blkX_pipe_1_block0=blkX_block0;
enableSignal_compute_pipe_1_block0=enableSignal_compute_block0;
enableSignal_write_pipe_2_block0=enableSignal_write_pipe_1_block0;
enableSignal_write_pipe_1_block0=enableSignal_write_block0;
validXmin_pipe_1_block0=validXmin_block0;
validXmax_pipe_1_block0=validXmax_block0;
blockIdx_pipe_2_block0=blockIdx_pipe_1_block0;
blockIdx_pipe_1_block0=blockIdx_block0;
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=true cores=[1] begin=false 
blockIdx_block0.x=(blockIdx_block0.x+(num_cores*1));
}
}

#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuWall_addr remove_port_name=gpuWall_core0 port_core=0 
#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuSrc_addr remove_port_name=gpuSrc_core0 port_core=0 
#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuResults_addr remove_port_name=gpuResults_core0 port_core=0 
__global__ void fcuda1(int iteration, int gpuWall_addr, int * gpuWall_core0, int gpuSrc_addr, int * gpuSrc_core0, int gpuResults_addr, int * gpuResults_core0, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim)
{
dynproc_kernel(iteration, gpuWall_core0, gpuSrc_core0, gpuResults_core0, cols, rows, startStep, border, gridDim, blockDim, 1, 0);
}



===========================================
[FixFCUDAMemParams-FCUDA] begin
[FixFCUDAMemParams-FCUDA] examining procedure dynproc_kernel
Proc: #pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int small_block_cols_block0;
int blkX_block0;
int blkXmax_block0;
int validXmin_block0;
int validXmax_block0;
int enableSignal_fetch_block0;
int enableSignal_compute_block0;
int enableSignal_write_block0;
dim3 blockIdx_block0;
int pingpong_0;
__shared__ int prev_ping_block0[256];
__shared__ int computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int prev_pong_block0[256];
__shared__ int result_ping_block0[256];
__shared__ int computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int result_pong_block0[256];
int blkX_pipe_1_block0;
int blkX_pipe_2_block0;
int enableSignal_compute_pipe_1_block0;
int enableSignal_write_pipe_1_block0;
int enableSignal_write_pipe_2_block0;
int validXmin_pipe_1_block0;
int validXmax_pipe_1_block0;
dim3 blockIdx_pipe_1_block0;
dim3 blockIdx_pipe_2_block0;
blockIdx_block0.y=0;
blockIdx_block0.x=((core_id*1)+0);
pingpong_0=0;
enableSignal_compute_pipe_1_block0=0;
enableSignal_write_pipe_1_block0=0;
enableSignal_write_pipe_2_block0=0;
blockIdx_pipe_2_block0.x=-1;
while (1)
{
while (blockIdx_block0.x>=gridDim.x)
{
blockIdx_block0.x=(blockIdx_block0.x-gridDim.x);
blockIdx_block0.y=(blockIdx_block0.y+1);
}
if ((blockIdx_block0.y>=(gridDim.y+((2*num_cores)+core_id))))
{
break;
}
enableSignal_write_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_compute_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_fetch_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
small_block_cols_block0=(256-((iteration*1)*2));
blkX_block0=((small_block_cols_block0*blockIdx_block0.x)-border);
blkXmax_block0=((blkX_block0+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=false begin=true 
validXmin_block0=((blkX_block0<0) ? ( - blkX_block0) : 0);
validXmax_block0=((blkXmax_block0>(cols-1)) ? ((256-1)-((blkXmax_block0-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=true begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=false cores=[1] begin=true 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=true cores=[1] begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=false cores=[1] begin=true 
{
if ((pingpong_0==0))
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_ping_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_ping_block0, gpuWall, iteration, prev_pong_block0, result_ping_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_pong_block0, gpuResults, result_pong_block0);
pingpong_0=1;
}
else
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_pong_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_pong_block0, gpuWall, iteration, prev_ping_block0, result_pong_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_ping_block0, gpuResults, result_ping_block0);
pingpong_0=0;
}
}
blkX_pipe_2_block0=blkX_pipe_1_block0;
blkX_pipe_1_block0=blkX_block0;
enableSignal_compute_pipe_1_block0=enableSignal_compute_block0;
enableSignal_write_pipe_2_block0=enableSignal_write_pipe_1_block0;
enableSignal_write_pipe_1_block0=enableSignal_write_block0;
validXmin_pipe_1_block0=validXmin_block0;
validXmax_pipe_1_block0=validXmax_block0;
blockIdx_pipe_2_block0=blockIdx_pipe_1_block0;
blockIdx_pipe_1_block0=blockIdx_block0;
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=true cores=[1] begin=false 
blockIdx_block0.x=(blockIdx_block0.x+(num_cores*1));
}
}

This proc does not contain remove ports pragmas. Skip.
[FixFCUDAMemParams-FCUDA] examining procedure fcuda1
Proc: #pragma fcuda portmerge data_pack=no port_id=0 offset=gpuWall_addr remove_port_name=gpuWall_core0 port_core=0 
#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuSrc_addr remove_port_name=gpuSrc_core0 port_core=0 
#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuResults_addr remove_port_name=gpuResults_core0 port_core=0 
__global__ void fcuda1(int iteration, int gpuWall_addr, int * gpuWall_core0, int gpuSrc_addr, int * gpuSrc_core0, int gpuResults_addr, int * gpuResults_core0, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim)
{
dynproc_kernel(iteration, gpuWall_core0, gpuSrc_core0, gpuResults_core0, cols, rows, startStep, border, gridDim, blockDim, 1, 0);
}

fcuda1(int iteration, int gpuWall_addr, int * gpuWall_core0, int gpuSrc_addr, int * gpuSrc_core0, int gpuResults_addr, int * gpuResults_core0, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim)
in getMatchingDecl
[int iteration, int gpuWall_addr, int * gpuWall_core0, int gpuSrc_addr, int * gpuSrc_core0, int gpuResults_addr, int * gpuResults_core0, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim]
[]
Removals: [gpuWall_core0, gpuSrc_core0, gpuResults_core0]
child: int iteration
[iteration]
iteration
[gpuWall_core0, gpuSrc_core0, gpuResults_core0]
child: int gpuWall_addr
[gpuWall_addr]
gpuWall_addr
[gpuWall_core0, gpuSrc_core0, gpuResults_core0]
child: int * gpuWall_core0
[gpuWall_core0]
gpuWall_core0
[gpuWall_core0, gpuSrc_core0, gpuResults_core0]
id: gpuWall_core0
child: int gpuSrc_addr
[gpuSrc_addr]
gpuSrc_addr
[gpuWall_core0, gpuSrc_core0, gpuResults_core0]
child: int * gpuSrc_core0
[gpuSrc_core0]
gpuSrc_core0
[gpuWall_core0, gpuSrc_core0, gpuResults_core0]
id: gpuSrc_core0
child: int gpuResults_addr
[gpuResults_addr]
gpuResults_addr
[gpuWall_core0, gpuSrc_core0, gpuResults_core0]
child: int * gpuResults_core0
[gpuResults_core0]
gpuResults_core0
[gpuWall_core0, gpuSrc_core0, gpuResults_core0]
id: gpuResults_core0
child: int cols
[cols]
cols
[gpuWall_core0, gpuSrc_core0, gpuResults_core0]
child: int rows
[rows]
rows
[gpuWall_core0, gpuSrc_core0, gpuResults_core0]
child: int startStep
[startStep]
startStep
[gpuWall_core0, gpuSrc_core0, gpuResults_core0]
child: int border
[border]
border
[gpuWall_core0, gpuSrc_core0, gpuResults_core0]
child: dim3 gridDim
[gridDim]
gridDim
[gpuWall_core0, gpuSrc_core0, gpuResults_core0]
child: dim3 blockDim
[blockDim]
blockDim
[gpuWall_core0, gpuSrc_core0, gpuResults_core0]
before exit
[int * gpuWall_core0, int * gpuSrc_core0, int * gpuResults_core0]
[int * gpuWall_core0, int * gpuSrc_core0, int * gpuResults_core0]
VariableDeclaration: [* ]
int * gpuSrc_core0
[int]
int * gpuResults_core0
[int]
[int]
int * memport_core0_p0
gpuWall_core0
gpuWall_core0 [#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuWall_addr remove_port_name=gpuWall_core0 port_core=0 , #pragma fcuda portmerge data_pack=no port_id=0 offset=gpuSrc_addr remove_port_name=gpuSrc_core0 port_core=0 , #pragma fcuda portmerge data_pack=no port_id=0 offset=gpuResults_addr remove_port_name=gpuResults_core0 port_core=0 ]
gpuSrc_core0
gpuSrc_core0 [#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuWall_addr remove_port_name=gpuWall_core0 port_core=0 , #pragma fcuda portmerge data_pack=no port_id=0 offset=gpuSrc_addr remove_port_name=gpuSrc_core0 port_core=0 , #pragma fcuda portmerge data_pack=no port_id=0 offset=gpuResults_addr remove_port_name=gpuResults_core0 port_core=0 ]
gpuResults_core0
gpuResults_core0 [#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuWall_addr remove_port_name=gpuWall_core0 port_core=0 , #pragma fcuda portmerge data_pack=no port_id=0 offset=gpuSrc_addr remove_port_name=gpuSrc_core0 port_core=0 , #pragma fcuda portmerge data_pack=no port_id=0 offset=gpuResults_addr remove_port_name=gpuResults_core0 port_core=0 ]
Annot is: #pragma HLS interface ap_bus port=memport_core0_p0 
[FixFCUDAMemParams-FCUDA] end in 0.00 seconds
[LinkSymbol] 219 updates in 0.00 seconds

*** After FixFCUDAMemParams ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
i=0;
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
i ++ ;
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int small_block_cols_block0;
int blkX_block0;
int blkXmax_block0;
int validXmin_block0;
int validXmax_block0;
int enableSignal_fetch_block0;
int enableSignal_compute_block0;
int enableSignal_write_block0;
dim3 blockIdx_block0;
int pingpong_0;
__shared__ int prev_ping_block0[256];
__shared__ int computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int prev_pong_block0[256];
__shared__ int result_ping_block0[256];
__shared__ int computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int result_pong_block0[256];
int blkX_pipe_1_block0;
int blkX_pipe_2_block0;
int enableSignal_compute_pipe_1_block0;
int enableSignal_write_pipe_1_block0;
int enableSignal_write_pipe_2_block0;
int validXmin_pipe_1_block0;
int validXmax_pipe_1_block0;
dim3 blockIdx_pipe_1_block0;
dim3 blockIdx_pipe_2_block0;
blockIdx_block0.y=0;
blockIdx_block0.x=((core_id*1)+0);
pingpong_0=0;
enableSignal_compute_pipe_1_block0=0;
enableSignal_write_pipe_1_block0=0;
enableSignal_write_pipe_2_block0=0;
blockIdx_pipe_2_block0.x=-1;
while (1)
{
while (blockIdx_block0.x>=gridDim.x)
{
blockIdx_block0.x=(blockIdx_block0.x-gridDim.x);
blockIdx_block0.y=(blockIdx_block0.y+1);
}
if ((blockIdx_block0.y>=(gridDim.y+((2*num_cores)+core_id))))
{
break;
}
enableSignal_write_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_compute_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_fetch_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
small_block_cols_block0=(256-((iteration*1)*2));
blkX_block0=((small_block_cols_block0*blockIdx_block0.x)-border);
blkXmax_block0=((blkX_block0+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=false begin=true 
validXmin_block0=((blkX_block0<0) ? ( - blkX_block0) : 0);
validXmax_block0=((blkXmax_block0>(cols-1)) ? ((256-1)-((blkXmax_block0-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=true begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=false cores=[1] begin=true 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=true cores=[1] begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=false cores=[1] begin=true 
{
if ((pingpong_0==0))
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_ping_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_ping_block0, gpuWall, iteration, prev_pong_block0, result_ping_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_pong_block0, gpuResults, result_pong_block0);
pingpong_0=1;
}
else
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_pong_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_pong_block0, gpuWall, iteration, prev_ping_block0, result_pong_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_ping_block0, gpuResults, result_ping_block0);
pingpong_0=0;
}
}
blkX_pipe_2_block0=blkX_pipe_1_block0;
blkX_pipe_1_block0=blkX_block0;
enableSignal_compute_pipe_1_block0=enableSignal_compute_block0;
enableSignal_write_pipe_2_block0=enableSignal_write_pipe_1_block0;
enableSignal_write_pipe_1_block0=enableSignal_write_block0;
validXmin_pipe_1_block0=validXmin_block0;
validXmax_pipe_1_block0=validXmax_block0;
blockIdx_pipe_2_block0=blockIdx_pipe_1_block0;
blockIdx_pipe_1_block0=blockIdx_block0;
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=true cores=[1] begin=false 
blockIdx_block0.x=(blockIdx_block0.x+(num_cores*1));
}
}

#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuWall_addr remove_port_name=gpuWall_core0 port_core=0 
#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuSrc_addr remove_port_name=gpuSrc_core0 port_core=0 
#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuResults_addr remove_port_name=gpuResults_core0 port_core=0 
__global__ void fcuda1(int iteration, int gpuWall_addr, int gpuSrc_addr, int gpuResults_addr, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int * memport_core0_p0)
{
int * gpuSrc_core0;
int * gpuResults_core0;
int * gpuWall_core0;
gpuResults_core0=( & memport_core0_p0[gpuResults_addr]);
gpuSrc_core0=( & memport_core0_p0[gpuSrc_addr]);
gpuWall_core0=( & memport_core0_p0[gpuWall_addr]);
dynproc_kernel(iteration, gpuWall_core0, gpuSrc_core0, gpuResults_core0, cols, rows, startStep, border, gridDim, blockDim, 1, 0);
}



===========================================
[GenWrapperMultiKernels - FCUDA] begin
[GenWrapperMultiKernels - FCUDA] examining procedure dynproc_kernel
[GenWrapperMultiKernels - FCUDA] examining procedure fcuda1
[GenWrapperMultiKernels - FCUDA] end in 0.00 seconds
[LinkSymbol] 219 updates in 0.00 seconds

*** After GenWrapperMultiKernels  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
i=0;
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
i ++ ;
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int small_block_cols_block0;
int blkX_block0;
int blkXmax_block0;
int validXmin_block0;
int validXmax_block0;
int enableSignal_fetch_block0;
int enableSignal_compute_block0;
int enableSignal_write_block0;
dim3 blockIdx_block0;
int pingpong_0;
__shared__ int prev_ping_block0[256];
__shared__ int computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int prev_pong_block0[256];
__shared__ int result_ping_block0[256];
__shared__ int computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int result_pong_block0[256];
int blkX_pipe_1_block0;
int blkX_pipe_2_block0;
int enableSignal_compute_pipe_1_block0;
int enableSignal_write_pipe_1_block0;
int enableSignal_write_pipe_2_block0;
int validXmin_pipe_1_block0;
int validXmax_pipe_1_block0;
dim3 blockIdx_pipe_1_block0;
dim3 blockIdx_pipe_2_block0;
blockIdx_block0.y=0;
blockIdx_block0.x=((core_id*1)+0);
pingpong_0=0;
enableSignal_compute_pipe_1_block0=0;
enableSignal_write_pipe_1_block0=0;
enableSignal_write_pipe_2_block0=0;
blockIdx_pipe_2_block0.x=-1;
while (1)
{
while (blockIdx_block0.x>=gridDim.x)
{
blockIdx_block0.x=(blockIdx_block0.x-gridDim.x);
blockIdx_block0.y=(blockIdx_block0.y+1);
}
if ((blockIdx_block0.y>=(gridDim.y+((2*num_cores)+core_id))))
{
break;
}
enableSignal_write_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_compute_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_fetch_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
small_block_cols_block0=(256-((iteration*1)*2));
blkX_block0=((small_block_cols_block0*blockIdx_block0.x)-border);
blkXmax_block0=((blkX_block0+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=false begin=true 
validXmin_block0=((blkX_block0<0) ? ( - blkX_block0) : 0);
validXmax_block0=((blkXmax_block0>(cols-1)) ? ((256-1)-((blkXmax_block0-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=true begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=false cores=[1] begin=true 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=true cores=[1] begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=false cores=[1] begin=true 
{
if ((pingpong_0==0))
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_ping_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_ping_block0, gpuWall, iteration, prev_pong_block0, result_ping_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_pong_block0, gpuResults, result_pong_block0);
pingpong_0=1;
}
else
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_pong_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_pong_block0, gpuWall, iteration, prev_ping_block0, result_pong_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_ping_block0, gpuResults, result_ping_block0);
pingpong_0=0;
}
}
blkX_pipe_2_block0=blkX_pipe_1_block0;
blkX_pipe_1_block0=blkX_block0;
enableSignal_compute_pipe_1_block0=enableSignal_compute_block0;
enableSignal_write_pipe_2_block0=enableSignal_write_pipe_1_block0;
enableSignal_write_pipe_1_block0=enableSignal_write_block0;
validXmin_pipe_1_block0=validXmin_block0;
validXmax_pipe_1_block0=validXmax_block0;
blockIdx_pipe_2_block0=blockIdx_pipe_1_block0;
blockIdx_pipe_1_block0=blockIdx_block0;
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=true cores=[1] begin=false 
blockIdx_block0.x=(blockIdx_block0.x+(num_cores*1));
}
}

#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuWall_addr remove_port_name=gpuWall_core0 port_core=0 
#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuSrc_addr remove_port_name=gpuSrc_core0 port_core=0 
#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuResults_addr remove_port_name=gpuResults_core0 port_core=0 
__global__ void fcuda1(int iteration, int gpuWall_addr, int gpuSrc_addr, int gpuResults_addr, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int * memport_core0_p0)
{
int * gpuSrc_core0;
int * gpuResults_core0;
int * gpuWall_core0;
gpuResults_core0=( & memport_core0_p0[gpuResults_addr]);
gpuSrc_core0=( & memport_core0_p0[gpuSrc_addr]);
gpuWall_core0=( & memport_core0_p0[gpuWall_addr]);
dynproc_kernel(iteration, gpuWall_core0, gpuSrc_core0, gpuResults_core0, cols, rows, startStep, border, gridDim, blockDim, 1, 0);
}

__global__ void fcuda(int iteration, int gpuWall_addr, int gpuSrc_addr, int gpuResults_addr, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int * memport_core0_p0_int, int en_fcuda1)
{
if ((en_fcuda1==1))
{
fcuda1(iteration, gpuWall_addr, gpuSrc_addr, gpuResults_addr, cols, rows, startStep, border, gridDim, blockDim, memport_core0_p0_int);
}
}



===========================================
[AddHLSPragmas-FCUDA] begin
[AddHLSPragmas-FCUDA] examining procedure dynproc_kernel
[AddHLSPragmas-FCUDA] examining procedure fcuda1
[AddHLSPragmas-FCUDA] examining procedure fcuda
[AddHLSPragmas-FCUDA] end in 0.00 seconds
[LinkSymbol] 219 updates in 0.00 seconds

*** After AddHLSPragmas  ***
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"
const int BLOCKDIM_X_dynproc_kernel = 256;
#pragma fcuda compute array_split=[result] mpart=1 name=fetch end=false cores=[1] unroll=1 begin=true 
void dynproc_kernel_fetch(int enableSignal_fetch, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, int * gpuSrc, __shared__ int prev[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_fetch)
{
if ((((blkX+((int)threadIdx.x))>=0)&&((blkX+((int)threadIdx.x))<=(cols-1))))
{
prev[threadIdx.x]=gpuSrc[(blkX+threadIdx.x)];
}
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, int cols, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuWall, int iteration, __shared__ int prev[256], __shared__ int result[256], int startStep, int validXmax, int validXmin)
{
dim3 __shared__ threadIdx;
int i;
if (enableSignal_compute)
{
i=0;
while (i<iteration)
{
int xidx;
int isValid;
int W;
int E;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
xidx=(blkX+threadIdx.x);
isValid=((((int)threadIdx.x)>=validXmin)&&(((int)threadIdx.x)<=validXmax));
W=(threadIdx.x-1);
E=(threadIdx.x+1);
W=((W<validXmin) ? validXmin : W);
E=((E>validXmax) ? validXmax : E);
computed_block[threadIdx.x]=0;
if ((((((int)threadIdx.x)>=(i+1))&&(((int)threadIdx.x)<=((256-i)-2)))&&isValid))
{
int left;
int up;
int right;
int shortest;
int index;
computed_block[threadIdx.x]=1;
left=prev[W];
up=prev[threadIdx.x];
right=prev[E];
shortest=((left<=up) ? left : up);
shortest=((shortest<=right) ? shortest : right);
index=((cols*(startStep+i))+xidx);
result[threadIdx.x]=(shortest+gpuWall[index]);
}
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((i==(iteration-1)))
{
break;
}
if (computed_block[threadIdx.x])
{
prev[threadIdx.x]=result[threadIdx.x];
}
}
i ++ ;
}
}
}

#pragma fcuda compute array_split=[result] mpart=1 name=write cores=[1] end=false unroll=1 begin=true 
void dynproc_kernel_write(int enableSignal_write, dim3 blockDim, dim3 gridDim, dim3 blockIdx, int blkX, __shared__ int computed_block[BLOCKDIM_X_dynproc_kernel], int * gpuResults, __shared__ int result[256])
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if (enableSignal_write)
{
if (computed_block[threadIdx.x])
{
gpuResults[(blkX+threadIdx.x)]=result[threadIdx.x];
}
}
}
}

#pragma fcuda grid x_dim=256 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
__global__ void dynproc_kernel(int iteration, int * gpuWall, int * gpuSrc, int * gpuResults, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int small_block_cols_block0;
int blkX_block0;
int blkXmax_block0;
int validXmin_block0;
int validXmax_block0;
int enableSignal_fetch_block0;
int enableSignal_compute_block0;
int enableSignal_write_block0;
dim3 blockIdx_block0;
int pingpong_0;
__shared__ int prev_ping_block0[256];
__shared__ int computed_block_ping_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int prev_pong_block0[256];
__shared__ int result_ping_block0[256];
__shared__ int computed_block_pong_block0[BLOCKDIM_X_dynproc_kernel];
__shared__ int result_pong_block0[256];
int blkX_pipe_1_block0;
int blkX_pipe_2_block0;
int enableSignal_compute_pipe_1_block0;
int enableSignal_write_pipe_1_block0;
int enableSignal_write_pipe_2_block0;
int validXmin_pipe_1_block0;
int validXmax_pipe_1_block0;
dim3 blockIdx_pipe_1_block0;
dim3 blockIdx_pipe_2_block0;
blockIdx_block0.y=0;
blockIdx_block0.x=((core_id*1)+0);
pingpong_0=0;
enableSignal_compute_pipe_1_block0=0;
enableSignal_write_pipe_1_block0=0;
enableSignal_write_pipe_2_block0=0;
blockIdx_pipe_2_block0.x=-1;
while (1)
{
while (blockIdx_block0.x>=gridDim.x)
{
blockIdx_block0.x=(blockIdx_block0.x-gridDim.x);
blockIdx_block0.y=(blockIdx_block0.y+1);
}
if ((blockIdx_block0.y>=(gridDim.y+((2*num_cores)+core_id))))
{
break;
}
enableSignal_write_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_compute_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
enableSignal_fetch_block0=((blockIdx_block0.x<gridDim.x)&&(blockIdx_block0.y<gridDim.y));
small_block_cols_block0=(256-((iteration*1)*2));
blkX_block0=((small_block_cols_block0*blockIdx_block0.x)-border);
blkXmax_block0=((blkX_block0+256)-1);
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=false begin=true 
validXmin_block0=((blkX_block0<0) ? ( - blkX_block0) : 0);
validXmax_block0=((blkXmax_block0>(cols-1)) ? ((256-1)-((blkXmax_block0-cols)+1)) : (256-1));
#pragma fcuda compute array_split=[result] mpart=1 name=fetch unroll=1 cores=[1] end=true begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=false cores=[1] begin=true 
#pragma fcuda compute array_split=[result] mpart=1 name=compute unroll=1 end=true cores=[1] begin=false 
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=false cores=[1] begin=true 
{
if ((pingpong_0==0))
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_ping_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_ping_block0, gpuWall, iteration, prev_pong_block0, result_ping_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_pong_block0, gpuResults, result_pong_block0);
pingpong_0=1;
}
else
{
dynproc_kernel_fetch(enableSignal_fetch_block0, blockDim, gridDim, blockIdx_block0, blkX_block0, cols, gpuSrc, prev_pong_block0);
dynproc_kernel_compute(enableSignal_compute_pipe_1_block0, blockDim, gridDim, blockIdx_pipe_1_block0, blkX_pipe_1_block0, cols, computed_block_pong_block0, gpuWall, iteration, prev_ping_block0, result_pong_block0, startStep, validXmax_pipe_1_block0, validXmin_pipe_1_block0);
dynproc_kernel_write(enableSignal_write_pipe_2_block0, blockDim, gridDim, blockIdx_pipe_2_block0, blkX_pipe_2_block0, computed_block_ping_block0, gpuResults, result_ping_block0);
pingpong_0=0;
}
}
blkX_pipe_2_block0=blkX_pipe_1_block0;
blkX_pipe_1_block0=blkX_block0;
enableSignal_compute_pipe_1_block0=enableSignal_compute_block0;
enableSignal_write_pipe_2_block0=enableSignal_write_pipe_1_block0;
enableSignal_write_pipe_1_block0=enableSignal_write_block0;
validXmin_pipe_1_block0=validXmin_block0;
validXmax_pipe_1_block0=validXmax_block0;
blockIdx_pipe_2_block0=blockIdx_pipe_1_block0;
blockIdx_pipe_1_block0=blockIdx_block0;
#pragma fcuda compute array_split=[result] mpart=1 name=write unroll=1 end=true cores=[1] begin=false 
blockIdx_block0.x=(blockIdx_block0.x+(num_cores*1));
}
}

#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuWall_addr remove_port_name=gpuWall_core0 port_core=0 
#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuSrc_addr remove_port_name=gpuSrc_core0 port_core=0 
#pragma fcuda portmerge data_pack=no port_id=0 offset=gpuResults_addr remove_port_name=gpuResults_core0 port_core=0 
__global__ void fcuda1(int iteration, int gpuWall_addr, int gpuSrc_addr, int gpuResults_addr, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int * memport_core0_p0)
{
#pragma HLS INTERFACE ap_none register port=iteration 
#pragma HLS RESOURCE core=AXI4LiteS variable=iteration 
#pragma HLS INTERFACE ap_none register port=gpuWall_addr 
#pragma HLS RESOURCE core=AXI4LiteS variable=gpuWall_addr 
#pragma HLS INTERFACE ap_none register port=gpuSrc_addr 
#pragma HLS RESOURCE core=AXI4LiteS variable=gpuSrc_addr 
#pragma HLS INTERFACE ap_none register port=gpuResults_addr 
#pragma HLS RESOURCE core=AXI4LiteS variable=gpuResults_addr 
#pragma HLS INTERFACE ap_none register port=cols 
#pragma HLS RESOURCE core=AXI4LiteS variable=cols 
#pragma HLS INTERFACE ap_none register port=rows 
#pragma HLS RESOURCE core=AXI4LiteS variable=rows 
#pragma HLS INTERFACE ap_none register port=startStep 
#pragma HLS RESOURCE core=AXI4LiteS variable=startStep 
#pragma HLS INTERFACE ap_none register port=border 
#pragma HLS RESOURCE core=AXI4LiteS variable=border 
#pragma HLS INTERFACE ap_none register port=gridDim 
#pragma HLS RESOURCE core=AXI4LiteS variable=gridDim 
#pragma HLS INTERFACE ap_none register port=blockDim 
#pragma HLS RESOURCE core=AXI4LiteS variable=blockDim 
#pragma HLS RESOURCE core=AXI4LiteS variable=return 
#pragma HLS interface ap_bus port=memport_core0_p0 
#pragma HLS RESOURCE variable=memport_core0_p0 core=AXI4M 
int * gpuSrc_core0;
int * gpuResults_core0;
int * gpuWall_core0;
gpuResults_core0=( & memport_core0_p0[gpuResults_addr]);
gpuSrc_core0=( & memport_core0_p0[gpuSrc_addr]);
gpuWall_core0=( & memport_core0_p0[gpuWall_addr]);
dynproc_kernel(iteration, gpuWall_core0, gpuSrc_core0, gpuResults_core0, cols, rows, startStep, border, gridDim, blockDim, 1, 0);
}

__global__ void fcuda(int iteration, int gpuWall_addr, int gpuSrc_addr, int gpuResults_addr, int cols, int rows, int startStep, int border, dim3 gridDim, dim3 blockDim, int * memport_core0_p0_int, int en_fcuda1)
{
#pragma HLS INTERFACE ap_none register port=en_fcuda1 
#pragma HLS RESOURCE core=AXI4LiteS variable=en_fcuda1 
#pragma HLS RESOURCE core=AXI4LiteS variable=return 
if ((en_fcuda1==1))
{
fcuda1(iteration, gpuWall_addr, gpuSrc_addr, gpuResults_addr, cols, rows, startStep, border, gridDim, blockDim, memport_core0_p0_int);
}
}



===========================================
