#include "hip/hip_runtime.h"
#include <fcuda.h>
#include <string.h>
#include "pathfinder.h"

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))
#define BLOCK_SIZE 256
#define HALO 1

#pragma FCUDA GRID x_dim=256
#pragma FCUDA COREINFO num_cores=1 pipeline=yes
#pragma FCUDA PORTMERGE remove_port_name=gpuWall port_id=0
#pragma FCUDA PORTMERGE remove_port_name=gpuSrc port_id=0
#pragma FCUDA PORTMERGE remove_port_name=gpuResults port_id=0
__global__ void dynproc_kernel(
    int iteration,
    DATATYPE *gpuWall,
    DATATYPE *gpuSrc,
    DATATYPE *gpuResults,
    int cols,
    int rows,
    int startStep,
    int border)
{
  __shared__ DATATYPE prev[BLOCK_SIZE];
  __shared__ DATATYPE result[BLOCK_SIZE];
  int bx = blockIdx.x;
  int tx=threadIdx.x;

  // each block finally computes result for a small block
  // after N iterations.
  // it is the non-overlapping small blocks that cover
  // all the input data

  // calculate the small block size
  int small_block_cols = BLOCK_SIZE-iteration*HALO*2;

  // calculate the boundary for the block according to
  // the boundary of its small block
  int blkX = small_block_cols*bx-border;
  int blkXmax = blkX+BLOCK_SIZE-1;

  // calculate the global thread coordination
  //int xidx = blkX+tx;

  // effective range within this block that falls within
  // the valid range of the input data
  // used to rule out computation outside the boundary.
  /*
     int validXmin = (blkX < 0) ? -blkX : 0;
     int validXmax = (blkXmax > cols-1) ? BLOCK_SIZE-1-(blkXmax-cols+1) : BLOCK_SIZE-1;

     int W = tx-1;
     int E = tx+1;

     W = (W < validXmin) ? validXmin : W;
     E = (E > validXmax) ? validXmax : E;

     bool isValid = IN_RANGE(tx, validXmin, validXmax);
   */
  // if(IN_RANGE(xidx, 0, cols-1)) {
#pragma FCUDA COMPUTE begin name=fetch cores=[1] unroll=1 mpart=1 array_split=[result]
  if(IN_RANGE(blkX + (int)tx, 0, cols-1)){
    prev[tx] = gpuSrc[blkX + tx];
  }
#pragma FCUDA COMPUTE end name=fetch cores=[1] unroll=1 mpart=1 array_split=[result]
  __syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012

  int validXmin = (blkX < 0) ? -blkX : 0;
  int validXmax = (blkXmax > cols-1) ? BLOCK_SIZE-1-(blkXmax-cols+1) : BLOCK_SIZE-1;

#pragma FCUDA COMPUTE cores=[1] begin name=compute unroll=1 mpart=1 array_split=[result]
  int i, computed;
  for (i=0; i<iteration ; i++)
  {

    int xidx = blkX+tx;
    int isValid = IN_RANGE((int)tx, validXmin, validXmax);
    int W = tx-1;
    int E = tx+1;
    W = (W < validXmin) ? validXmin : W;
    E = (E > validXmax) ? validXmax : E;
    computed = 0;

    if(IN_RANGE((int)tx, i+1, BLOCK_SIZE-i-2) && isValid) {
      computed = 1;
      DATATYPE left = prev[W];
      DATATYPE up = prev[tx];
      DATATYPE right = prev[E];
      DATATYPE shortest = MIN(left, up);
      shortest = MIN(shortest, right);
      int index = cols*(startStep+i)+xidx;
      result[tx] = shortest + gpuWall[index];
    }
    __syncthreads();

    if(i==iteration-1)
      break;

    if(computed)     //Assign the computation range
      prev[tx]= result[tx];
    __syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
  }
#pragma FCUDA COMPUTE cores=[1] end name=compute unroll=1 mpart=1 array_split=[result]

  // update the global memory
  // after the last iteration, only threads coordinated within the
  // small block perform the calculation and switch on ``computed''
#pragma FCUDA COMPUTE cores=[1] begin name=write unroll=1 mpart=1 array_split=[result]
  if (computed) {
    gpuResults[blkX + tx]=result[tx];
  }
#pragma FCUDA COMPUTE cores=[1] end name=write unroll=1 mpart=1 array_split=[result]
}

