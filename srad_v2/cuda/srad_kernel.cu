#include "hip/hip_runtime.h"
#include "srad.h"
#include <stdio.h>

__global__ void
srad_cuda_1(
		  double *E_C, 
		  double *W_C, 
		  double *N_C, 
		  double *S_C,
		  double * J_cuda, 
		  double * C_cuda, 
		  int cols, 
		  int rows, 
		  double q0sqr
) 
{

  //block id
  int bx = blockIdx.x;
  int by = blockIdx.y;

  //thread id
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  
  //indices
  int index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
  int index_n = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + tx - cols;
  int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
  int index_w = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty - 1;
  int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;

  double n, w, e, s, jc, g2, l, num, den, qsqr, c;

  //shared memory allocation
  __shared__ double temp[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double temp_result[BLOCK_SIZE][BLOCK_SIZE];

  __shared__ double north[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double south[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double  east[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double  west[BLOCK_SIZE][BLOCK_SIZE];

  //load data to shared memory
  north[ty][tx] = J_cuda[index_n]; 
  south[ty][tx] = J_cuda[index_s];
  if ( by == 0 ){
  north[ty][tx] = J_cuda[BLOCK_SIZE * bx + tx]; 
  }
  else if ( by == gridDim.y - 1 ){
  south[ty][tx] = J_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * ( BLOCK_SIZE - 1 ) + tx];
  }
   __syncthreads();
 
  west[ty][tx] = J_cuda[index_w];
  east[ty][tx] = J_cuda[index_e];

  if ( bx == 0 ){
  west[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + cols * ty]; 
  }
  else if ( bx == gridDim.x - 1 ){
  east[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * ( gridDim.x - 1) + cols * ty + BLOCK_SIZE-1];
  }
 
  __syncthreads();
  
 

  temp[ty][tx]      = J_cuda[index];

  __syncthreads();

   jc = temp[ty][tx];

   if ( ty == 0 && tx == 0 ){ //nw
	n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = west[ty][tx]  - jc; 
    e  = temp[ty][tx+1] - jc;
   }	    
   else if ( ty == 0 && tx == BLOCK_SIZE-1 ){ //ne
	n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = east[ty][tx] - jc;
   }
   else if ( ty == BLOCK_SIZE -1 && tx == BLOCK_SIZE - 1){ //se
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = east[ty][tx]  - jc;
   }
   else if ( ty == BLOCK_SIZE -1 && tx == 0 ){//sw
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = west[ty][tx]  - jc; 
    e  = temp[ty][tx+1] - jc;
   }

   else if ( ty == 0 ){ //n
	n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = temp[ty][tx+1] - jc;
   }
   else if ( tx == BLOCK_SIZE -1 ){ //e
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = east[ty][tx] - jc;
   }
   else if ( ty == BLOCK_SIZE -1){ //s
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = temp[ty][tx+1] - jc;
   }
   else if ( tx == 0 ){ //w
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = west[ty][tx] - jc; 
    e  = temp[ty][tx+1] - jc;
   }
   else{  //the data elements which are not on the borders 
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = temp[ty][tx+1] - jc;
   }


    g2 = ( n * n + s * s + w * w + e * e ) / (jc * jc);

    l = ( n + s + w + e ) / (jc);

	num  = (0.5*g2) - ((1.0/16.0)*(l*l)) ;
	den  = 1.0 + (.25*l);
	qsqr = num/(den*den);

	// diffusion coefficent (equ 33)
	den = (qsqr-q0sqr) / (q0sqr * (1.0+q0sqr)) ;
	c = 1.0 / (1.0+den) ;

    // saturate diffusion coefficent
	if (c < 0){temp_result[ty][tx] = 0;}
	else if (c > 1) {temp_result[ty][tx] = 1;}
	else {temp_result[ty][tx] = c;}

    __syncthreads();

    C_cuda[index] = temp_result[ty][tx];
	E_C[index] = e;
	W_C[index] = w;
	S_C[index] = s;
	N_C[index] = n;

}

__global__ void
srad_cuda_2(
		  double *E_C, 
		  double *W_C, 
		  double *N_C, 
		  double *S_C,	
		  double * J_cuda, 
		  double * C_cuda, 
		  int cols, 
		  int rows, 
		  double lambda,
		  double q0sqr
) 
{
	//block id
	int bx = blockIdx.x;
    int by = blockIdx.y;

	//thread id
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	//indices
    int index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
	int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
    int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;
	double cc, cn, cs, ce, cw, d_sum;

	//shared memory allocation
	__shared__ double south_c[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double  east_c[BLOCK_SIZE][BLOCK_SIZE];

    __shared__ double c_cuda_temp[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double c_cuda_result[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double temp[BLOCK_SIZE][BLOCK_SIZE];

    //load data to shared memory
	temp[ty][tx]      = J_cuda[index];

    __syncthreads();
	 
	south_c[ty][tx] = C_cuda[index_s];

	if ( by == gridDim.y - 1 ){
	south_c[ty][tx] = C_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * ( BLOCK_SIZE - 1 ) + tx];
	}
	__syncthreads();
	 
	 
	east_c[ty][tx] = C_cuda[index_e];
	
	if ( bx == gridDim.x - 1 ){
	east_c[ty][tx] = C_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * ( gridDim.x - 1) + cols * ty + BLOCK_SIZE-1];
	}
	 
    __syncthreads();
  
    c_cuda_temp[ty][tx]      = C_cuda[index];

    __syncthreads();

	cc = c_cuda_temp[ty][tx];

   if ( ty == BLOCK_SIZE -1 && tx == BLOCK_SIZE - 1){ //se
	cn  = cc;
    cs  = south_c[ty][tx];
    cw  = cc; 
    ce  = east_c[ty][tx];
   } 
   else if ( tx == BLOCK_SIZE -1 ){ //e
	cn  = cc;
    cs  = c_cuda_temp[ty+1][tx];
    cw  = cc; 
    ce  = east_c[ty][tx];
   }
   else if ( ty == BLOCK_SIZE -1){ //s
	cn  = cc;
    cs  = south_c[ty][tx];
    cw  = cc; 
    ce  = c_cuda_temp[ty][tx+1];
   }
   else{ //the data elements which are not on the borders 
	cn  = cc;
    cs  = c_cuda_temp[ty+1][tx];
    cw  = cc; 
    ce  = c_cuda_temp[ty][tx+1];
   }

   // divergence (equ 58)
   d_sum = cn * N_C[index] + cs * S_C[index] + cw * W_C[index] + ce * E_C[index];

   // image update (equ 61)
   c_cuda_result[ty][tx] = temp[ty][tx] + 0.25 * lambda * d_sum;

   __syncthreads();
              
   J_cuda[index] = c_cuda_result[ty][tx];
    
}
