#include "hip/hip_runtime.h"
#include <fcuda.h>
#include "srad.h"
#include <string.h>


#pragma FCUDA GRID x_dim=16 y_dim=16
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=E_C port_id=0
#pragma FCUDA PORTMERGE remove_port_name=W_C port_id=0
#pragma FCUDA PORTMERGE remove_port_name=N_C port_id=0
#pragma FCUDA PORTMERGE remove_port_name=S_C port_id=0
#pragma FCUDA PORTMERGE remove_port_name=J_cuda port_id=0
#pragma FCUDA PORTMERGE remove_port_name=C_cuda port_id=0
__global__ void
srad_cuda_1(double *E_C,
	double *W_C,
	double *N_C,
	double *S_C,
	double * J_cuda,
	double * C_cuda,
	int cols,
	int rows,
	double q0sqr
)
{

  //block id
  int bx = blockIdx.x;
  int by = blockIdx.y;

  //thread id
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  //indices
  //int index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
  int index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx;

  //int index_n = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + tx - cols;
  int index_n = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx - cols;

  //int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
  int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE;

  //int index_w = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty - 1;
  //int index_w = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx - 1;

  //int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;
  //int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + BLOCK_SIZE;

  double n, w, e, s, jc, g2, l, num, den, qsqr, c;

  //shared memory allocation
  __shared__ double temp[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double temp_result[BLOCK_SIZE][BLOCK_SIZE];

  __shared__ double north[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double south[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double east[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double west[BLOCK_SIZE][BLOCK_SIZE];

  //load data to shared memory

  if ( by == 0 ){
#pragma FCUDA TRANSFER begin name=fetch_n type=burst dir=[0|0] pointer=[J_cuda|J_cuda] size=[16|16] cores=[1] unroll=1 mpart=1 array_split=[temp_result]
  	north[ty][tx] = J_cuda[BLOCK_SIZE * bx + tx];
  	south[ty][tx] = J_cuda[index_s + tx];
#pragma FCUDA TRANSFER end name=fetch_n type=burst dir=[0|0] pointer=[J_cuda|J_cuda] size=[16|16] cores=[1] unroll=1 mpart=1 array_split=[temp_result]
  }
  else if ( by == gridDim.y - 1 ){
#pragma FCUDA TRANSFER begin name=fetch_s type=burst dir=[0|0] pointer=[J_cuda|J_cuda] size=[16|16] cores=[1] unroll=1 mpart=1 array_split=[temp_result]
  	south[ty][tx] = J_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * ( BLOCK_SIZE - 1 ) + tx];
  	north[ty][tx] = J_cuda[index_n + tx];
#pragma FCUDA TRANSFER end name=fetch_s type=burst dir=[0|0] pointer=[J_cuda|J_cuda] size=[16|16] cores=[1] unroll=1 mpart=1 array_split=[temp_result]
  } else {
#pragma FCUDA TRANSFER begin name=fetch_n_s type=burst dir=[0|0] pointer=[J_cuda|J_cuda] size=[16|16] cores=[1] unroll=1 mpart=1 array_split=[temp_result]
  	north[ty][tx] = J_cuda[index_n + tx];
  	south[ty][tx] = J_cuda[index_s + tx];
#pragma FCUDA TRANSFER end name=fetch_n_s type=burst dir=[0|0] pointer=[J_cuda|J_cuda] size=[16|16] cores=[1] unroll=1 mpart=1 array_split=[temp_result]
  }
   __syncthreads();

#pragma FCUDA COMPUTE begin name=compute_w_s cores=[1] unroll=1 mpart=1 array_split=[temp_result]
  int index_w = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty - 1;
  int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;

  if ( bx == 0 ){
  	west[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + cols * ty];
	east[ty][tx] = J_cuda[index_e];
  }
  else if ( bx == gridDim.x - 1 ){
  	east[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * ( gridDim.x - 1) + cols * ty + BLOCK_SIZE-1];
	west[ty][tx] = J_cuda[index_w];
  }
  else {
	west[ty][tx] = J_cuda[index_w];
  	east[ty][tx] = J_cuda[index_e];
  }
#pragma FCUDA COMPUTE end name=compute_w_s cores=[1] unroll=1 mpart=1 array_split=[temp_result]
  __syncthreads();

#pragma FCUDA TRANSFER begin name=fetch_temp type=burst dir=[0] pointer=[J_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[temp_result]
  temp[ty][tx] = J_cuda[index + cols * ty + tx];
#pragma FCUDA TRANSFER end name=fetch_temp type=burst dir=[0] pointer=[J_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[temp_result]

  __syncthreads();

#pragma FCUDA COMPUTE begin name=compute cores=[1] unroll=1 mpart=1 array_split=[temp_result]
   jc = temp[ty][tx];

   if ( ty == 0 && tx == 0 ){ //nw
	n  = north[ty][tx] - jc;
    	s  = temp[ty+1][tx] - jc;
    	w  = west[ty][tx]  - jc;
    	e  = temp[ty][tx+1] - jc;
   }
   else if ( ty == 0 && tx == BLOCK_SIZE-1 ){ //ne
	n  = north[ty][tx] - jc;
    	s  = temp[ty+1][tx] - jc;
    	w  = temp[ty][tx-1] - jc;
    	e  = east[ty][tx] - jc;
   }
   else if ( ty == BLOCK_SIZE -1 && tx == BLOCK_SIZE - 1){ //se
	n  = temp[ty-1][tx] - jc;
    	s  = south[ty][tx] - jc;
    	w  = temp[ty][tx-1] - jc;
    	e  = east[ty][tx]  - jc;
   }
   else if ( ty == BLOCK_SIZE -1 && tx == 0 ){//sw
	n  = temp[ty-1][tx] - jc;
    	s  = south[ty][tx] - jc;
    	w  = west[ty][tx]  - jc;
    	e  = temp[ty][tx+1] - jc;
   }

   else if ( ty == 0 ){ //n
	n  = north[ty][tx] - jc;
    	s  = temp[ty+1][tx] - jc;
    	w  = temp[ty][tx-1] - jc;
    	e  = temp[ty][tx+1] - jc;
   }
   else if ( tx == BLOCK_SIZE -1 ){ //e
	n  = temp[ty-1][tx] - jc;
    	s  = temp[ty+1][tx] - jc;
    	w  = temp[ty][tx-1] - jc;
    	e  = east[ty][tx] - jc;
   }
   else if ( ty == BLOCK_SIZE -1){ //s
	n  = temp[ty-1][tx] - jc;
    	s  = south[ty][tx] - jc;
    	w  = temp[ty][tx-1] - jc;
    	e  = temp[ty][tx+1] - jc;
   }
   else if ( tx == 0 ){ //w
	n  = temp[ty-1][tx] - jc;
    	s  = temp[ty+1][tx] - jc;
    	w  = west[ty][tx] - jc;
    	e  = temp[ty][tx+1] - jc;
   }
   else{  //the data elements which are not on the borders
	n  = temp[ty-1][tx] - jc;
    	s  = temp[ty+1][tx] - jc;
    	w  = temp[ty][tx-1] - jc;
    	e  = temp[ty][tx+1] - jc;
   }


    g2 = ( n * n + s * s + w * w + e * e ) / (jc * jc);

    l = ( n + s + w + e ) / jc;

	num  = (0.5*g2) - ((1.0/16.0)*(l*l)) ;
	den  = 1 + (.25*l);
	qsqr = num/(den*den);

	// diffusion coefficent (equ 33)
	den = (qsqr-q0sqr) / (q0sqr * (1+q0sqr)) ;
	c = 1.0 / (1.0+den) ;

    // saturate diffusion coefficent
	if (c < 0){temp_result[ty][tx] = 0;}
	else if (c > 1) {temp_result[ty][tx] = 1;}
	else {temp_result[ty][tx] = c;}

    __syncthreads();

    E_C[index + cols * ty + tx] = e;
    W_C[index + cols * ty + tx] = w;
    S_C[index + cols * ty + tx] = s;
    N_C[index + cols * ty + tx] = n;
#pragma FCUDA COMPUTE end name=compute cores=[1] unroll=1 mpart=1 array_split=[temp_result]

#pragma FCUDA TRANSFER begin name=write type=burst dir=[1] pointer=[C_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[temp_result]
    C_cuda[index + cols * ty + tx] = temp_result[ty][tx];
#pragma FCUDA TRANSFER end name=write type=burst dir=[1] pointer=[C_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[temp_result]

	//E_C[index] = e;
	//W_C[index] = w;
	//S_C[index] = s;
	//N_C[index] = n;
}


#pragma FCUDA GRID x_dim=16 y_dim=16
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=E_C port_id=0
#pragma FCUDA PORTMERGE remove_port_name=W_C port_id=0
#pragma FCUDA PORTMERGE remove_port_name=N_C port_id=0
#pragma FCUDA PORTMERGE remove_port_name=S_C port_id=0
#pragma FCUDA PORTMERGE remove_port_name=J_cuda port_id=0
#pragma FCUDA PORTMERGE remove_port_name=C_cuda port_id=0
__global__ void
srad_cuda_2(double *E_C,
	double *W_C,
	double *N_C,
	double *S_C,
	double * J_cuda,
	double * C_cuda,
	int cols,
	int rows,
	double lambda,
	double q0sqr
) {
    	//block id
    	int bx = blockIdx.x;
    	int by = blockIdx.y;

	//thread id
    	int tx = threadIdx.x;
    	int ty = threadIdx.y;

	//indices
    	//int index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
	int index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx;

    	//int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
	int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE;

    	//int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;
	double cc, cn, cs, ce, cw, d_sum;

	//shared memory allocation
	__shared__ double south_c[BLOCK_SIZE][BLOCK_SIZE];
    	__shared__ double east_c[BLOCK_SIZE][BLOCK_SIZE];

    	__shared__ double c_cuda_temp[BLOCK_SIZE][BLOCK_SIZE];
    	__shared__ double c_cuda_result[BLOCK_SIZE][BLOCK_SIZE];
    	__shared__ double temp[BLOCK_SIZE][BLOCK_SIZE];

    	//load data to shared memory
#pragma FCUDA TRANSFER begin name=fetch_temp type=burst dir=[0] pointer=[J_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[c_cuda_result]
	temp[ty][tx]      = J_cuda[index + cols * ty + tx];
#pragma FCUDA TRANSFER end name=fetch_temp type=burst dir=[0] pointer=[J_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[c_cuda_result]

    	__syncthreads();

	if ( by == gridDim.y - 1 ){
#pragma FCUDA TRANSFER begin name=fetch_south_c_2 type=burst dir=[0] pointer=[C_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[c_cuda_result]
		south_c[ty][tx] = C_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * ( BLOCK_SIZE - 1 ) + tx];
#pragma FCUDA TRANSFER end name=fetch_south_c_2 type=burst dir=[0] pointer=[C_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[c_cuda_result]
	} else {
#pragma FCUDA TRANSFER begin name=fetch_south_c_1 type=burst dir=[0] pointer=[C_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[c_cuda_result]
		south_c[ty][tx] = C_cuda[index_s + tx];
#pragma FCUDA TRANSFER end name=fetch_south_c_1 type=burst dir=[0] pointer=[C_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[c_cuda_result]
	}
	__syncthreads();

#pragma FCUDA COMPUTE begin name=compute1 cores=[1] unroll=1 mpart=1 array_split=[c_cuda_result]
	int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;
	
	if ( bx == gridDim.x - 1 ){
		east_c[ty][tx] = C_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * ( gridDim.x - 1) + cols * ty + BLOCK_SIZE-1];
	} else {
		east_c[ty][tx] = C_cuda[index_e];
	}
#pragma FCUDA COMPUTE end name=compute1 cores=[1] unroll=1 mpart=1 array_split=[c_cuda_result]
    	__syncthreads();

#pragma FCUDA TRANSFER begin name=fetch_c_cuda_temp type=burst dir=[0] pointer=[C_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[c_cuda_result]
    	c_cuda_temp[ty][tx] = C_cuda[index + cols * ty + tx];
#pragma FCUDA TRANSFER end name=fetch_c_cuda_temp type=burst dir=[0] pointer=[C_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[c_cuda_result]
    	__syncthreads();

#pragma FCUDA COMPUTE begin name=compute2 cores=[1] unroll=1 mpart=1 array_split=[c_cuda_result]
	cc = c_cuda_temp[ty][tx];

   	if ( ty == BLOCK_SIZE -1 && tx == BLOCK_SIZE - 1){ //se
		cn  = cc;
    		cs  = south_c[ty][tx];
    		cw  = cc;
    		ce  = east_c[ty][tx];
   	}
   	else if ( tx == BLOCK_SIZE -1 ){ //e
		cn  = cc;
    		cs  = c_cuda_temp[ty+1][tx];
    		cw  = cc;
    		ce  = east_c[ty][tx];
   	}
   	else if ( ty == BLOCK_SIZE -1){ //s
		cn  = cc;
    		cs  = south_c[ty][tx];
    		cw  = cc;
    		ce  = c_cuda_temp[ty][tx+1];
   	}
   	else{ //the data elements which are not on the borders
		cn  = cc;
    		cs  = c_cuda_temp[ty+1][tx];
    		cw  = cc;
    		ce  = c_cuda_temp[ty][tx+1];
   	}
	__syncthreads();
   	// divergence (equ 58)
   	d_sum = cn * N_C[index + cols * ty + tx] + cs * S_C[index + cols * ty + tx] + cw * W_C[index + cols * ty + tx] + ce * E_C[index + cols * ty + tx];

   	// image update (equ 61)
   	c_cuda_result[ty][tx] = temp[ty][tx] + 0.25 * lambda * d_sum;
#pragma FCUDA COMPUTE end name=compute2 cores=[1] unroll=1 mpart=1 array_split=[c_cuda_result]
   	__syncthreads();

#pragma FCUDA TRANSFER begin name=write type=burst dir=[1] pointer=[J_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[c_cuda_result]
   	J_cuda[index + cols * ty + tx] = c_cuda_result[ty][tx];
#pragma FCUDA TRANSFER end name=write type=burst dir=[1] pointer=[J_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[c_cuda_result]

}

