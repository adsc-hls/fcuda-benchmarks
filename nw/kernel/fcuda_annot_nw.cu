#include "hip/hip_runtime.h"
#include "needle.h"
#include <stdio.h>
#include "fcuda.h"
#include <string.h>

#define SDATA( index) CUT_BANK_CHECKER(sdata, index)

/*
   __device__ int
   int maximum(int a, int b, int c) {

   int k;
   if( a <= b )
   k = b;
   else
   k = a;

   if (k <=c )
   return(c);
   else
   return(k);
   }
 */

#pragma FCUDA GRID x_dim=16
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=referrence port_id=0
#pragma FCUDA PORTMERGE remove_port_name=matrix_cuda port_id=0
__global__ void
needle_cuda_shared_1(  int* referrence,
    int* matrix_cuda,
    //int* matrix_cuda_out,
    int cols,
    int penalty,
    int i,
    int block_width)
{
  int bx = blockIdx.x;
  int tx = threadIdx.x;

  int b_index_x = bx;
  int b_index_y = i - 1 - bx;

  //int index   = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + tx + ( cols + 1 );
  int index   = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + ( cols + 1 );
  //int index_n   = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + tx + ( 1 );
  int index_n   = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + ( 1 );
  int index_w   = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + ( cols );
  int index_nw =  cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x;

  __shared__  int temp[BLOCK_SIZE+1][BLOCK_SIZE+1];
  __shared__  int ref[BLOCK_SIZE][BLOCK_SIZE];

#pragma FCUDA COMPUTE begin name=init_temp cores=[1]
  if (tx == 0)
    temp[tx][0] = matrix_cuda[index_nw];
#pragma FCUDA COMPUTE end name=init_temp cores=[1]

  int ty;
  //#pragma FCUDA COMPUTE begin name=fetch_ref cores=[1] unroll=1 mpart=1 array_split=[]
  for (ty = 0 ; ty < BLOCK_SIZE; ty++) {
#pragma FCUDA TRANSFER begin name=fetch_ref type=burst dir=[0] pointer=[referrence] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    ref[ty][tx] = referrence[index + tx + cols * ty];
#pragma FCUDA TRANSFER end name=fetch_ref type=burst dir=[0] pointer=[referrence] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
  }
  //#pragma FCUDA COMPUTE end name=fetch_ref cores=[1] unroll=1 mpart=1 array_split=[]

  __syncthreads();

#pragma FCUDA COMPUTE begin name=init_temp2 cores=[1]
  temp[tx + 1][0] = matrix_cuda[index_w + cols * tx];
#pragma FCUDA COMPUTE end name=init_temp2 cores=[1]

  __syncthreads();

#pragma FCUDA TRANSFER begin name=fetch_temp type=burst dir=[0] pointer=[matrix_cuda] size=[16] cores=[1] array_split=[]

  temp[0][tx + 1] = matrix_cuda[index_n + tx];
#pragma FCUDA TRANSFER end name=fetch_temp type=burst dir=[0] pointer=[matrix_cuda] size=[16] cores=[1] array_split=[]
  __syncthreads();

#pragma FCUDA COMPUTE begin name=compute cores=[1] unroll=1 mpart=1 array_split=[]
  int m;
  for (m = 0 ; m < BLOCK_SIZE ; m++) {

    if (tx <= m) {

      int t_index_x =  tx + 1;
      int t_index_y =  m - tx + 1;

      temp[t_index_y][t_index_x] = maximum(temp[t_index_y-1][t_index_x-1] + ref[t_index_y-1][t_index_x-1],
          temp[t_index_y][t_index_x-1]  - penalty,
          temp[t_index_y-1][t_index_x]  - penalty);
    }
    __syncthreads();
  }

  for (m = BLOCK_SIZE - 2 ; m >=0 ; m--) {

    if (tx <= m) {

      int t_index_x =  tx + BLOCK_SIZE - m ;
      int t_index_y =  BLOCK_SIZE - tx;

      temp[t_index_y][t_index_x] = maximum(temp[t_index_y-1][t_index_x-1] + ref[t_index_y-1][t_index_x-1],
          temp[t_index_y][t_index_x-1]  - penalty,
          temp[t_index_y-1][t_index_x]  - penalty);
    }

    __syncthreads();
  }
#pragma FCUDA COMPUTE end name=compute cores=[1] unroll=1 mpart=1 array_split=[]

  for (ty = 0 ; ty < BLOCK_SIZE ; ty++) {
#pragma FCUDA TRANSFER begin name=write type=burst dir=[1] pointer=[matrix_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]

    matrix_cuda[index + tx + ty * cols] = temp[ty+1][tx+1];
#pragma FCUDA TRANSFER end name=write type=burst dir=[1] pointer=[matrix_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
  }
}


#pragma FCUDA GRID x_dim=16
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=referrence port_id=0
#pragma FCUDA PORTMERGE remove_port_name=matrix_cuda port_id=0
__global__ void
needle_cuda_shared_2(  int* referrence,
    int* matrix_cuda,
    //int* matrix_cuda_out,

    int cols,
    int penalty,
    int i,
    int block_width)
{
  int bx = blockIdx.x;
  int tx = threadIdx.x;

  int b_index_x = bx + block_width - i  ;
  int b_index_y = block_width - bx -1;

  //int index   = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + tx + ( cols + 1 );
  int index   = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + ( cols + 1 );
  //int index_n   = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + tx + ( 1 );
  int index_n   = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + ( 1 );
  int index_w   = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + ( cols );
  int index_nw =  cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x;

  __shared__  int temp[BLOCK_SIZE+1][BLOCK_SIZE+1];
  __shared__  int ref[BLOCK_SIZE][BLOCK_SIZE];
  int ty;
  for (ty = 0 ; ty < BLOCK_SIZE ; ty++) {
#pragma FCUDA TRANSFER begin name=fetch_ref type=burst dir=[0] pointer=[referrence] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    ref[ty][tx] = referrence[index + cols * ty + tx];
#pragma FCUDA TRANSFER end name=fetch_ref type=burst dir=[0] pointer=[referrence] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
  }
  __syncthreads();

#pragma FCUDA COMPUTE begin name=init_temp cores=[1]
  if (tx == 0)
    temp[tx][0] = matrix_cuda[index_nw];
#pragma FCUDA COMPUTE end name=init_temp cores=[1]

#pragma FCUDA COMPUTE begin name=init_temp2 cores=[1]
  temp[tx + 1][0] = matrix_cuda[index_w + cols * tx];
#pragma FCUDA COMPUTE end name=init_temp2 cores=[1]

  __syncthreads();

#pragma FCUDA TRANSFER begin name=fetch_temp type=burst dir=[0] pointer=[matrix_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
  temp[0][tx + 1] = matrix_cuda[index_n + tx];
#pragma FCUDA TRANSFER end name=fetch_temp type=burst dir=[0] pointer=[matrix_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
  __syncthreads();

#pragma FCUDA COMPUTE begin name=compute cores=[1] unroll=1 mpart=1 array_split=[]
  int m;
  for (m = 0 ; m < BLOCK_SIZE ; m++) {

    if (tx <= m) {

      int t_index_x =  tx + 1;
      int t_index_y =  m - tx + 1;

      temp[t_index_y][t_index_x] = maximum(temp[t_index_y-1][t_index_x-1] + ref[t_index_y-1][t_index_x-1],
          temp[t_index_y][t_index_x-1]  - penalty,
          temp[t_index_y-1][t_index_x]  - penalty);

    }
    __syncthreads();
  }


  for(m = BLOCK_SIZE - 2 ; m >=0 ; m--){

    if (tx <= m) {

      int t_index_x =  tx + BLOCK_SIZE - m ;
      int t_index_y =  BLOCK_SIZE - tx;

      temp[t_index_y][t_index_x] = maximum(temp[t_index_y-1][t_index_x-1] + ref[t_index_y-1][t_index_x-1],
          temp[t_index_y][t_index_x-1]  - penalty,
          temp[t_index_y-1][t_index_x]  - penalty);


    }

    __syncthreads();
  }
#pragma FCUDA COMPUTE end name=compute cores=[1] unroll=1 mpart=1 array_split=[]

  for (ty = 0 ; ty < BLOCK_SIZE ; ty++) {
#pragma FCUDA TRANSFER begin name=write type=burst dir=[1] pointer=[matrix_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    matrix_cuda[index + tx + ty * cols] = temp[ty+1][tx+1];
#pragma FCUDA TRANSFER end name=write type=burst dir=[1] pointer=[matrix_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
  }
}

