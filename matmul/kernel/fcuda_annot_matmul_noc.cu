#include "hip/hip_runtime.h"
/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <fcuda.h>
//#include "matrixMul.h"
#include <string.h>

#define BLOCK_SIZE 16
typedef int DATATYPE;

#define CHECK_BANK_CONFLICTS 0
#if CHECK_BANK_CONFLICTS
#define AS(i, j) cutilBankChecker(((DATATYPE*)&As[0][0]), (BLOCK_SIZE * i + j))
#define BS(i, j) cutilBankChecker(((DATATYPE*)&Bs[0][0]), (BLOCK_SIZE * i + j))
#else
#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]
#endif

////////////////////////////////////////////////////////////////////////////////
//! Matrix multiplication on the device: C = A * B
//! wA is A's width and wB is B's width
////////////////////////////////////////////////////////////////////////////////
#pragma FCUDA GRID x_dim=16 y_dim=16
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=A port_id=0
#pragma FCUDA PORTMERGE remove_port_name=B port_id=0
#pragma FCUDA PORTMERGE remove_port_name=C port_id=0
__global__ void matrixMul( DATATYPE *C, DATATYPE *A, DATATYPE *B, int wA, int wB)
{
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd   = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep  = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep  = BLOCK_SIZE * wB;

#pragma FCUDA COMPUTE cores=[1] begin name=vec_init unroll=1 mpart=1 array_split=[Csub_block|As] inline=no
  DATATYPE Csub = 0;
#pragma FCUDA COMPUTE cores=[1] end name=vec_init unroll=1 mpart=1 array_split=[Csub_block|As] inline=no

  int a = 0, b = 0, k = 0;
  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (a = aBegin, b = bBegin;
      a <= aEnd;
      a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
#pragma FCUDA PORTMERGE remove_port_name=As
    __shared__ DATATYPE As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
#pragma FCUDA PORTMERGE remove_port_name=Bs
    __shared__ DATATYPE Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0|0] pointer=[A|B] size=[16|16] begin name=fetch unroll=1 mpart=1 array_split=[Csub_block|As] inline=no
    AS(ty, tx) = A[a + wA * ty + tx];
    BS(ty, tx) = B[b + wB * ty + tx];
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0|0] pointer=[A|B] size=[16|16] end name=fetch unroll=1 mpart=1 array_split=[Csub_block|As] inline=no

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma FCUDA COMPUTE cores=[1] begin name=vec_blk array_split=[Csub_block|As] unroll=1 mpart=1 inline=no
lp1: for (k = 0; k < BLOCK_SIZE; ++k)
       Csub += AS(ty, k) * BS(k, tx);
#pragma FCUDA COMPUTE cores=[1] end name=vec_blk array_split=[Csub_block|As] unroll=1 mpart=1 inline=no

     // Synchronize to make sure that the preceding
     // computation is done before loading two new
     // sub-matrices of A and B in the next iteration
     __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[1] pointer=[C] size=[16] begin name=write unroll=1 mpart=1 array_split=[Csub_block|As] inline=yes
  C[c + wB * ty + tx] = Csub;
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[1] pointer=[C] size=[16] end name=write unroll=1 mpart=1 array_split=[Csub_block|As] inline=yes
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
