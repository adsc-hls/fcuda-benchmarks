#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include <stdio.h>
//#include "cuenergy.h"
#include <fcuda.h>
#include <math.h>
#include <string.h>
/* Size of a thread block */
#define BLOCKSIZEX 16
#define BLOCKSIZEY 16

/* Number of grid points processed by a thread */
#define UNROLLX 1

/* Number of atoms processed by a kernel */
#define MAXATOMS 128

/* Size of the benchmark problem.  The GPU can run larger problems in a
 * reasonable time.
 *
 * For VOLSIZEX, VOLSIZEY, size 1024 is suitable for a few seconds of
 * GPU computation and size 128 is suitable for a few seconds of
 * CPU computation.
 *
 * For ATOMCOUNT, 100000 is suitable for GPU computation and 10000 is
 * suitable for CPU computation.
 */
#define VOLSIZEX 128
#define VOLSIZEY 128
#define ATOMCOUNT 1024
//typedef float DATATYPE;
typedef int DATATYPE;

__constant__ DATATYPE atominfo[4*MAXATOMS];

/* Size of the benchmark problem.  The GPU can run larger problems in a
 * reasonable time.
 *
 * For VOLSIZEX, VOLSIZEY, size 256 is suitable for a few seconds of
 * GPU computation and size 128 is suitable for a few seconds of
 * CPU computation.
 *
 * For ATOMCOUNT, 100000 is suitable for GPU computation and 10000 is
 * suitable for CPU computation.
 */

/* This is a reference version of the kernel.  It is simpler and slower
 * than the optimzed version. */

#pragma FCUDA GRID x_dim=16 y_dim=16
#pragma FCUDA PORTMERGE remove_port_name=atomdata port_id=0
#pragma FCUDA PORTMERGE remove_port_name=energygrid port_id=0
__global__ void cenergy(int numatoms, DATATYPE gridspacing, DATATYPE *energygrid) {

	int atombytes = numatoms*4;
#pragma FCUDA TRANSFER cores=[1] type=stream dir=[0] pointer=[atomdata] local=[atominfo] size=[atombytes] rsize=[4096] begin name=atomfetch unroll=1 mpart=1 array_split=[energyval]
	__shared__ DATATYPE energyval[16][16], energyval1[16][16];
	int offset = gridDim.x*blockDim.x*blockIdx.y*blockDim.y + blockIdx.x*blockDim.x;
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0] pointer=[energygrid] begin name=energyfetch size=[16] unroll=1 mpart=1 array_split=[energyval]
	energyval[threadIdx.y][threadIdx.x] = energygrid[offset+gridDim.x*blockDim.x*threadIdx.y + threadIdx.x];
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0] pointer=[energygrid] end name=energyfetch size=[16] unroll=1 mpart=1 array_split=[energyval]

#pragma FCUDA COMPUTE cores=[1] begin name=blk_compute unroll=1 mpart=1 array_split=[energyval] //shared=[atominfo] non_shared=[energyval] bram_core=[132|24|20|256]
	unsigned int xindex  = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	unsigned int yindex  = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	DATATYPE coorx = gridspacing * xindex;
	DATATYPE coory = gridspacing * yindex;

	int atomid;
  	/* For each atom, compute and accumulate its contribution to energyval
  	 * for this thread's grid point */
	lp1:for (atomid=0; atomid<atombytes; atomid+=4) {
    		DATATYPE dx = coorx - atominfo[atomid];
    		DATATYPE dy = coory - atominfo[atomid+1];
    		//DATATYPE r_1 = 1.0f / sqrtf(dx*dx + dy*dy + atominfo[atomid+2]);
                DATATYPE r_1 = dx*dx + dy*dy + atominfo[atomid+2];
    		energyval[threadIdx.y][threadIdx.x] += atominfo[atomid+3] * r_1;
  	}
        energyval1[threadIdx.y][threadIdx.x] = energyval[threadIdx.y][threadIdx.x];
#pragma FCUDA COMPUTE cores=[1] end name=blk_compute unroll=1 mpart=1 array_split=[energyval] //shared=[atominfo] non_shared=[energyval] bram_core=[132|84|80|256]

#pragma FCUDA TRANSFER cores=[1] type=burst dir=[1] pointer=[energygrid] begin name=write size=[16] unroll=1 mpart=1 array_split=[energyval]
	energygrid[offset+gridDim.x*blockDim.x*threadIdx.y + threadIdx.x] = energyval1[threadIdx.y][threadIdx.x];
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[1] pointer=[energygrid] end name=write size=[16] unroll=1 mpart=1 array_split=[energyval]


}
