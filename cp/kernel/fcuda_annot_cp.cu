#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include <stdio.h>
#include "cuenergy.h"
#include <fcuda.h>
#include <math.h>
#include <string.h>
__constant__ DATATYPE atominfo[4*MAXATOMS];

/* Size of the benchmark problem.  The GPU can run larger problems in a
 * reasonable time.
 *
 * For VOLSIZEX, VOLSIZEY, size 256 is suitable for a few seconds of
 * GPU computation and size 128 is suitable for a few seconds of
 * CPU computation.
 *
 * For ATOMCOUNT, 100000 is suitable for GPU computation and 10000 is
 * suitable for CPU computation.
 */

/* This is a reference version of the kernel.  It is simpler and slower
 * than the optimzed version. */

#pragma FCUDA GRID x_dim=16 y_dim=16
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=atomdata port_id=0
#pragma FCUDA PORTMERGE remove_port_name=energygrid port_id=0
__global__ void cenergy(int numatoms, DATATYPE gridspacing, DATATYPE *energygrid) {

	int atombytes = numatoms*4;
#pragma FCUDA TRANSFER cores=[1] type=stream dir=[0] pointer=[atomdata] local=[atominfo] size=[atombytes] rsize=[4096] begin name=atomfetch
	__shared__ DATATYPE energyval[16][16], energyval1[16][16];
	int offset = gridDim.x*blockDim.x*blockIdx.y*blockDim.y + blockIdx.x*blockDim.x;
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0] pointer=[energygrid] begin name=energyfetch size=[16] unroll=1 mpart=1 array_split=[energyval]
	energyval[threadIdx.y][threadIdx.x] = energygrid[offset+gridDim.x*blockDim.x*threadIdx.y + threadIdx.x];
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0] pointer=[energygrid] end name=energyfetch size=[16] unroll=1 mpart=1 array_split=[energyval]

#pragma FCUDA COMPUTE cores=[1] begin name=blk_compute unroll=1 mpart=1 array_split=[energyval]
	unsigned int xindex  = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	unsigned int yindex  = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	DATATYPE coorx = gridspacing * xindex;
	DATATYPE coory = gridspacing * yindex;

	int atomid;
  	/* For each atom, compute and accumulate its contribution to energyval
  	 * for this thread's grid point */
	lp1:for (atomid=0; atomid<atombytes; atomid+=4) {
    		DATATYPE dx = coorx - atominfo[atomid];
    		DATATYPE dy = coory - atominfo[atomid+1];
    		DATATYPE r_1 = 1.0f / sqrtf(dx*dx + dy*dy + atominfo[atomid+2]);
    		energyval[threadIdx.y][threadIdx.x] += atominfo[atomid+3] * r_1;
	}
	energyval1[threadIdx.y][threadIdx.x] = energyval[threadIdx.y][threadIdx.x];
#pragma FCUDA COMPUTE cores=[1] end name=blk_compute unroll=1 mpart=1 array_split=[energyval]

#pragma FCUDA TRANSFER cores=[1] type=burst dir=[1] pointer=[energygrid] begin name=write size=[16] unroll=1 mpart=1 array_split=[energyval]
	energygrid[offset+gridDim.x*blockDim.x*threadIdx.y + threadIdx.x] = energyval1[threadIdx.y][threadIdx.x];
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[1] pointer=[energygrid] end name=write size=[16] unroll=1 mpart=1 array_split=[energyval]


}
