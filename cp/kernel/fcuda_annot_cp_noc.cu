#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include <stdio.h>
#include "cuenergy_noc.h"
#include <fcuda.h>
#include <math.h>
#include <string.h>
__constant__ DATATYPE atominfo[4*MAXATOMS];

/* Size of the benchmark problem.  The GPU can run larger problems in a
 * reasonable time.
 *
 * For VOLSIZEX, VOLSIZEY, size 256 is suitable for a few seconds of
 * GPU computation and size 128 is suitable for a few seconds of
 * CPU computation.
 *
 * For ATOMCOUNT, 100000 is suitable for GPU computation and 10000 is
 * suitable for CPU computation.
 */

/* This is a reference version of the kernel.  It is simpler and slower
 * than the optimzed version. */

#define numatoms MAXATOMS
#define gridspacing 1

#pragma FCUDA GRID x_dim=16 y_dim=16
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=atominfo port_id=0
#pragma FCUDA PORTMERGE remove_port_name=energygrid_input port_id=0
#pragma FCUDA PORTMERGE remove_port_name=energygrid_output port_id=0 
__global__ void cenergy(DATATYPE * atominfo, DATATYPE * energygrid_input, DATATYPE * energygrid_output) {


	int atombytes = numatoms*4;
#pragma FCUDA PORTMERGE remove_port_name=atom
        __shared__ DATATYPE atom[MAXATOMS * 4];
	__shared__ DATATYPE energyval[16][16];

#pragma FCUDA COMPUTE cores=[1] begin name=fetchatom unroll=1 mpart=1 array_split=[energyval]
        //int i = offset + gridDim.x *,blockDim.x * threadIdx.y + threadIdx.x;
        int i;
        i = BLOCKSIZEX * threadIdx.y + threadIdx.x;
        for(; i < MAXATOMS * 4; i += BLOCKSIZEX*BLOCKSIZEY)
	   atom[i] = atominfo[i]; 
        __syncthreads();
#pragma FCUDA COMPUTE cores=[1] end name=fetchatom unroll=1 mpart=1 array_split=[energyval]

//#pragma FCUDA TRANSFER cores=[1] type=stream dir=[0] pointer=[atomdata] local=[atominfo] size=[atombytes] rsize=[4096] begin name=atomfetch
	int offset = gridDim.x*blockDim.x*blockIdx.y*blockDim.y + blockIdx.x*blockDim.x;
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0] pointer=[energygrid_input] begin name=energyfetch size=[16] unroll=1 mpart=1 array_split=[energyval]
	energyval[threadIdx.y][threadIdx.x] = energygrid_input[offset+gridDim.x*blockDim.x*threadIdx.y + threadIdx.x];
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0] pointer=[energygrid_input] end name=energyfetch size=[16] unroll=1 mpart=1 array_split=[energyval]

#pragma FCUDA COMPUTE cores=[1] begin name=blk_compute unroll=1 mpart=1 array_split=[energyval]
	unsigned int xindex  = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	unsigned int yindex  = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	DATATYPE coorx = gridspacing * xindex;
	DATATYPE coory = gridspacing * yindex;

	int atomid;
  	/* For each atom, compute and accumulate its contribution to energyval
  	 * for this thread's grid point */
	lp1:for (atomid=0; atomid<atombytes; atomid+=4) {
    		DATATYPE dx = coorx - atom[atomid];
    		DATATYPE dy = coory - atom[atomid+1];
    		//DATATYPE r_1 = 1.0f / sqrtf(dx*dx + dy*dy + atominfo[atomid+2]);
                DATATYPE r_1 = dx*dx + dy*dy + atom[atomid+2];
    		energyval[threadIdx.y][threadIdx.x] += atom[atomid+3] * r_1;
	}
#pragma FCUDA COMPUTE cores=[1] end name=blk_compute unroll=1 mpart=1 array_split=[energyval]

	//__syncthreads();
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[1] pointer=[energygrid_output] begin name=write size=[16] unroll=1 mpart=1 array_split=[energyval]
        energygrid_output[offset+gridDim.x*blockDim.x*threadIdx.y + threadIdx.x] = energyval[threadIdx.y][threadIdx.x];
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[1] pointer=[energygrid_output] end name=write size=[16] unroll=1 mpart=1 array_split=[energyval]
}
