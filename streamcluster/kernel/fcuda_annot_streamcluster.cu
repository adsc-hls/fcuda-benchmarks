#include "hip/hip_runtime.h"
#include <fcuda.h>
/* this structure represents a point */
/* these will be passed around to avoid copying coordinates */
/*typedef struct {
  float weight;
  float *coord;
  long assign;  // number of point where this one is assigned
  float cost;  // cost of that assignment, weight*distance
  } Point;
 */
#include "define.h"
#include <string.h>
//=======================================
// Euclidean Distance
//=======================================

float d_dist(int p1, int p2, int num, int dim, float *coord_d)
{
  float retval = 0.0;
  int i;
  for (i = 0; i < dim; i++) {
    float tmp = coord_d[(i*num)+p1] - coord_d[(i*num)+p2];
    retval += tmp * tmp;
  }
  return retval;
}

//=======================================
// Kernel - Compute Cost
//=======================================
#pragma FCUDA GRID x_dim=512
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=weight port_id=0
#pragma FCUDA PORTMERGE remove_port_name=assign port_id=1
#pragma FCUDA PORTMERGE remove_port_name=cost port_id=0
#pragma FCUDA PORTMERGE remove_port_name=coord_d port_id=0
#pragma FCUDA PORTMERGE remove_port_name=work_mem_d port_id=0
#pragma FCUDA PORTMERGE remove_port_name=center_table_d port_id=2
#pragma FCUDA PORTMERGE remove_port_name=switch_membership_d port_id=3
__global__ void kernel_compute_cost(int num, int dim, long x,//Point p[4096],
    float *weight,
    long *assign,
    float *cost,
    int K, int stride,
    float *coord_d,
    float *work_mem_d,
    int *center_table_d,
    DATATYPE *switch_membership_d)
{
  // block ID and global thread ID
  const int bid  = blockIdx.x + gridDim.x * blockIdx.y;
  //const int tid = blockDim.x * bid + threadIdx.x;
  //__shared__ Point p_shared[512];
  __shared__ float weight_shared[512];
  __shared__ long assign_shared[512];
  __shared__ float cost_shared[512];
  //#pragma FCUDA COMPUTE cores=[1] begin name=fetch unroll=1 mpart=1 array_split=[p_shared]
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0|0|0] pointer=[weight|assign|cost] size=[512|512|512] begin name=fetch unroll=1 mpart=1 array_split=[weight_shared|assign_shared|cost_shared]

  weight_shared[threadIdx.x] = weight[blockDim.x * bid + threadIdx.x];
  assign_shared[threadIdx.x] = assign[blockDim.x * bid + threadIdx.x];
  cost_shared[threadIdx.x] = cost[blockDim.x * bid + threadIdx.x];
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0|0|0] pointer=[weight|assign|cost] size=[512|512|512] end name=fetch unroll=1 mpart=1 array_split=[weight_shared|assign_shared|cost_shared]

  //#pragma FCUDA COMPUTE cores=[1] end name=fetch unroll=1 mpart=1 array_split=[p_shared]

#pragma FCUDA COMPUTE cores=[1] begin name=compute unroll=1 mpart=1 array_split=[weight_shared|assign_shared|cost_shared]
  const int tid = blockDim.x * bid + threadIdx.x;
  if (tid < num) {
    //float *lower = &work_mem_d[tid*stride];
    // cost between this point and point[x]: euclidean distance multiplied by weight
    //float x_cost = d_dist(tid, x, num, dim, coord_d) * p_shared[threadIdx.x].weight;
    float x_cost = d_dist(tid, x, num, dim, coord_d) * weight_shared[threadIdx.x];
    // if computed cost is less then original (it saves), mark it as to reassign
    //if (x_cost < p_shared[threadIdx.x].cost) {
    if (x_cost < cost_shared[threadIdx.x]) {
      switch_membership_d[tid] = 1;
      //lower[K] += x_cost - p[tid].cost;
      work_mem_d[tid * stride + K] += x_cost - cost_shared[threadIdx.x];
    }
    // if computed cost is larger, save the difference
    else {
      //int temp = center_table_d[p[tid].assign];
      //lower[center_table_d[p[tid].assign]] += p[tid].cost - x_cost;
      work_mem_d[tid * stride + center_table_d[assign_shared[threadIdx.x]]] += cost_shared[threadIdx.x] - x_cost;
    }
  }
#pragma FCUDA COMPUTE cores=[1] end name=compute unroll=1 mpart=1 array_split=[weight_shared|assign_shared|cost_shared]

  }


