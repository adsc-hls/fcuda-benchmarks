#include "hip/hip_runtime.h"
/*

   -------------------------------------------------
   | a_0 | a_0 | a_0 | a_0 | a_0 | a_0 | a_0 | a_0 |
   -------------------------------------------------

   -------------------------------------------------
   | a_1 | a_1 | a_1 | a_1 | d_1 | d_1 | d_1 | d_1 |
   -------------------------------------------------

   -------------------------------------------------
   | a_2 | a_2 | d_2 | d_2 | d_1 | d_1 | d_1 | d_1 |
   -------------------------------------------------

   -------------------------------------------------
   | a_3 | d_3 | d_2 | d_2 | d_1 | d_1 | d_1 | d_1 |
   -------------------------------------------------

 * Device Code.
 */
#ifndef _DWTHAAR1D_KERNEL_H_
#define _DWTHAAR1D_KERNEL_H_

#include <fcuda.h>
//#include "dwtHaar1D.h"
#include <string.h>

#define INV_SQRT_2 10
#define LOG_NUM_BANKS 4
#define NUM_BANKS 16
typedef int DATATYPE;

////////////////////////////////////////////////////////////////////////////////
//! Compute partial wavelet decomposition on the GPU using Haar basis
//! For each thread block the full decomposition is computed but these results
//! have to be combined
//! Use one thread to perform the full decomposition
//! @param id  input data
//! @param od  output data
//! @param approx_final  place to store the final approximation coefficient for
//!                      the subsignal
//! @param dlevels  number of decomposition levels for this transform
//! @param slength_step_half   half signal length for current decomposition
//!                            level (offset for storing detail coefficients in
//!                            global memory
//! @param bdim  block dimension
////////////////////////////////////////////////////////////////////////////////

#pragma FCUDA GRID x_dim=512
#pragma FCUDA COREINFO num_cores=2 pipeline=yes
#pragma FCUDA PORTMERGE remove_port_name=id port_id=0
#pragma FCUDA PORTMERGE remove_port_name=od port_id=0
#pragma FCUDA PORTMERGE remove_port_name=approx_final port_id=0
  __global__ void
dwtHaar1D(DATATYPE *id, DATATYPE *od, DATATYPE *approx_final,
    const unsigned int dlevels,
    const unsigned int slength_step_half,
    const int bdim)
{
  __shared__ DATATYPE shared[1024];
  __shared__ DATATYPE shared2[1024];

  __shared__ DATATYPE dataDiff1[512];
  __shared__ DATATYPE dataDiff2[512];

  const int gdim = gridDim.x;
  const int bid = blockIdx.x;
  const int tid = threadIdx.x;

  DATATYPE data0;
  DATATYPE data1;
  unsigned int offset_neighbor;
  unsigned int num_threads, num_threads2;
  unsigned int stride;
  unsigned int g_wpos;

  const int tid_global = (bid * bdim) ;
  unsigned int idata = (bid * (2 * bdim));

#pragma FCUDA COMPUTE cores=[1] begin name=fetch unroll=1 array_split=[dataDiff1] mpart=1
//#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0] pointer=[id] size=[1024] begin name=fetchid unroll=1 mpart=1 array_split=[dataDiff1]
  int i;
  for (i = 0; i < 1024; i+=512)
  shared[tid + i] = id[idata + tid + i];
//#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0] pointer=[id] size=[1024] end name=fetchid unroll=1 mpart=1  array_split=[dataDiff1]
#pragma FCUDA COMPUTE cores=[1] end name=fetch unroll=1 array_split=[dataDiff1] mpart=1

  __syncthreads();

#pragma FCUDA COMPUTE cores=[1] begin name=dcomp0 unroll=1 array_split=[dataDiff1] mpart=1
  data0 = shared[2*tid];
  data1 = shared[(2*tid) + 1];
  __syncthreads();

  dataDiff1[tid] = (data0 - data1) * INV_SQRT_2;
  unsigned int atid = tid + (tid >> LOG_NUM_BANKS);
  shared2[atid] = (data0 + data1) * INV_SQRT_2;
  __syncthreads();

  offset_neighbor = 1;
  num_threads = bdim >> 1;
  stride = 1;
  unsigned int dlv;
  for(dlv = 1; dlv < dlevels; ++dlv)
  {
    unsigned int idata0 = tid << stride;
    if( tid < num_threads)
    {
      unsigned int idata1 = idata0 + offset_neighbor;
      dataDiff2[num_threads+tid] = (shared2[idata0 + (idata0 >> LOG_NUM_BANKS)] - shared2[idata1 + (idata1 >> LOG_NUM_BANKS)]) * INV_SQRT_2;
      shared2[idata0 + (idata0 >> LOG_NUM_BANKS)] = (shared2[idata0 + (idata0 >> LOG_NUM_BANKS)] + shared2[idata1 + (idata1 >> LOG_NUM_BANKS)]) * INV_SQRT_2;
    }
    num_threads = num_threads >> 1;
    offset_neighbor <<= 1;
    stride = stride + 1;
    __syncthreads();
  }
  __syncthreads();
#pragma FCUDA COMPUTE cores=[1] end name=dcomp0 unroll=1 array_split=[dataDiff1] mpart=1

#pragma FCUDA COMPUTE cores=[1] begin name=dcomp1 unroll=1 array_split=[dataDiff1] mpart=1
  if (tid == 0)
    approx_final[bid + tid] = shared2[tid];
#pragma FCUDA COMPUTE cores=[1] end name=dcomp1 unroll=1 array_split=[dataDiff1] mpart=1


#pragma FCUDA TRANSFER cores=[1] type=burst dir=[1] pointer=[od] size=[bdim] begin name=writeod unroll=1 mpart=1 array_split=[dataDiff1]
  od[tid_global + slength_step_half + tid] = dataDiff1[tid];
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[1] pointer=[od] size=[bdim] end name=writeod unroll=1 mpart=1 array_split=[dataDiff1]

  for(num_threads2 = bdim >> 1; num_threads2>0; num_threads2 >>= 1) {
    g_wpos = (num_threads2 * gdim) + (bid * num_threads2);

#pragma FCUDA TRANSFER cores=[1] type=burst dir=[1] pointer=[od] size=[num_threads2] begin name=writeod2 unroll=1 mpart=1 array_split=[dataDiff1] inline=no
    od[g_wpos + tid] = dataDiff2[tid + num_threads2];
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[1] pointer=[od] size=[num_threads2] end name=writeod2 unroll=1 mpart=1 array_split=[dataDiff1] inline=no
  }
}

#endif
