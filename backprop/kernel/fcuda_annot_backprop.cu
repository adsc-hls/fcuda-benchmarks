#include "hip/hip_runtime.h"
#include <fcuda.h>
#include "backprop.h"
#include <math.h>
#include <string.h>

int custom_pow_base2(int i) {
  int k;
  int result = 1;
  for (k = 0; k < i; k++)
    result = result * 2;
  return result;
}

#pragma FCUDA GRID x_dim=16 y_dim=16
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=input_cuda port_id=0 offset=0
#pragma FCUDA PORTMERGE remove_port_name=input_hidden_cuda port_id=0
#pragma FCUDA PORTMERGE remove_port_name=hidden_partial_sum port_id=0
  __global__ void
bpnn_layerforward_CUDA(float *input_cuda,
    float *input_hidden_cuda,
    float *hidden_partial_sum,
    int in,
    int hid)
{
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  //int index =  ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;
  int index =  ( hid + 1 ) * HEIGHT * by + 1 + ( hid + 1 ) ;

  //int index_in = HEIGHT * by + ty + 1;

  __shared__ float input_node[HEIGHT];
  __shared__ float weight_matrix[HEIGHT][WIDTH];
  __shared__ float weight_matrix1[HEIGHT][WIDTH];
  __shared__ float weight_matrix2[HEIGHT][WIDTH];

#pragma FCUDA COMPUTE begin name=compute_input_node cores=[1] unroll=1 mpart=1 array_split=[input_node|weight_matrix]
  int index_in = HEIGHT * by + ty + 1;
  if ( tx == 0 )
    input_node[ty] = input_cuda[index_in] ;
#pragma FCUDA COMPUTE end name=compute_input_node cores=[1] unroll=1 mpart=1 array_split=[input_node|weight_matrix]

  //__syncthreads();

#pragma FCUDA TRANSFER begin name=fetch_weight_matrix type=burst dir=[0] pointer=[input_hidden_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[input_node|weight_matrix]
  weight_matrix[ty][tx] = input_hidden_cuda[index + (hid + 1) * ty + tx];
#pragma FCUDA TRANSFER end name=fetch_weight_matrix type=burst dir=[0] pointer=[input_hidden_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[input_node|weight_matrix]
  __syncthreads();

#pragma FCUDA COMPUTE begin name=compute_weight_matrix1 cores=[1] unroll=1 mpart=1 array_split=[input_node|weight_matrix]
  weight_matrix1[ty][tx] = weight_matrix[ty][tx] * input_node[ty];
  //#pragma FCUDA COMPUTE end name=compute_weight_matrix1 cores=[1] unroll=1 mpart=1 array_split=[input_node|weight_matrix]

  __syncthreads();
  int i,j;

  for ( i = 1 ; i <= log(HEIGHT) ; i++)
  {
    //int power_two = powf(2, i);
    int power_two = custom_pow_base2(i);
    if( ty % power_two == 0 )
      weight_matrix1[ty][tx] = weight_matrix1[ty][tx] + weight_matrix1[ty + power_two/2][tx];
    __syncthreads();
  }

  weight_matrix2[ty][tx] = weight_matrix1[ty][tx];
  __syncthreads();

#pragma FCUDA COMPUTE end name=compute_weight_matrix1 cores=[1] unroll=1 mpart=1 array_split=[input_node|weight_matrix]



#pragma FCUDA TRANSFER begin name=write_weight_matrix type=burst dir=[1] pointer=[input_hidden_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[input_node|weight_matrix]
  input_hidden_cuda[index + (hid + 1) * ty + tx] = weight_matrix2[ty][tx];
#pragma FCUDA TRANSFER end name=write_weight_matrix type=burst dir=[1] pointer=[input_hidden_cuda] size=[16] cores=[1] unroll=1 mpart=1 array_split=[input_node|weight_matrix]

  /*
     for ( unsigned int i = 2 ; i <= HEIGHT ; i *= 2){

     unsigned int power_two = i - 1;

     if( (ty & power_two) == 0 ) {
     weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + power_two/2][tx];
     }

     }
   */

  __syncthreads();

#pragma FCUDA COMPUTE begin name=compute_hps cores=[1] unroll=1 mpart=1 array_split=[input_node|weight_matrix]
  if ( tx == 0 ) {
    hidden_partial_sum[by * hid + ty] = weight_matrix1[tx][ty];
  }
#pragma FCUDA COMPUTE end name=compute_hps cores=[1] unroll=1 mpart=1 array_split=[input_node|weight_matrix]

}

#pragma FCUDA GRID x_dim=16 y_dim=16
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=delta port_id=0 offset=0
#pragma FCUDA PORTMERGE remove_port_name=ly port_id=0
#pragma FCUDA PORTMERGE remove_port_name=w port_id=0
#pragma FCUDA PORTMERGE remove_port_name=oldw port_id=0
__global__ void bpnn_adjust_weights_cuda(float * delta,
    int hid,
    float * ly,
    int in,
    float * w,
    float * oldw)
{
#pragma FCUDA COMPUTE begin name=compute cores=[1] unroll=1
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int index =  ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;
  int index_y = HEIGHT * by + ty + 1;
  int index_x = tx + 1;
  //eta = 0.3;
  //momentum = 0.3;

  w[index] += ((ETA * delta[index_x] * ly[index_y]) + (MOMENTUM * oldw[index]));
  oldw[index] = ((ETA * delta[index_x] * ly[index_y]) + (MOMENTUM * oldw[index]));


  __syncthreads();

  if (ty == 0 && by ==0){
    w[index_x] += ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
    oldw[index_x] = ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
  }
#pragma FCUDA COMPUTE end name=compute cores=[1] unroll=1
}
