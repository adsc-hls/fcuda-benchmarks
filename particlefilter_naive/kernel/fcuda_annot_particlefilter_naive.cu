#include "hip/hip_runtime.h"
#include <fcuda.h>
#include "device_func.h"

/*****************************
* CUDA Kernel Function to replace FindIndex
* param1: arrayX
* param2: arrayY
* param3: CDF
* param4: u
* param5: xj
* param6: yj
* param7: Nparticles
*****************************/
#pragma FCUDA GRID x_dim=128
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=arrayX port_id=0 offset=0
#pragma FCUDA PORTMERGE remove_port_name=arrayY port_id=0 offset=0
#pragma FCUDA PORTMERGE remove_port_name=CDF port_id=0 offset=0
#pragma FCUDA PORTMERGE remove_port_name=u port_id=0 offset=0
#pragma FCUDA PORTMERGE remove_port_name=xj port_id=0 offset=0
#pragma FCUDA PORTMERGE remove_port_name=yj port_id=0 offset=0
__global__ void kernel(double * arrayX, double * arrayY, double * CDF, double * u, double * xj, double * yj, int Nparticles){

#pragma FCUDA COMPUTE begin name=compute cores=[1] unroll=1 mpart=1 array_split=[]
	int block_id = blockIdx.x;// + gridDim.x * blockIdx.y;
	int i = blockDim.x * block_id + threadIdx.x;

	if(i < Nparticles){

		int index = -1;
		int x;
                int stop = 0;
		for(x = 0; x < Nparticles; x++){
			if(stop == 0 && CDF[x] >= u[i]){
				index = x;
				//break; // Tan: Vivado 2015.4 seems to have an issue with
                                         // break statement here, hence replace this with
                                         // the variable "stop"
                                stop = 1;
			}
		}
		if(index == -1){
			index = Nparticles-1;
		}

		xj[i] = arrayX[index];
		yj[i] = arrayY[index];
	}
#pragma FCUDA COMPUTE end name=compute cores=[1] unroll=1 mpart=1 array_split=[]
}


