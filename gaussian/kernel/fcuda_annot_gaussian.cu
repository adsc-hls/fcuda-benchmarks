#include "hip/hip_runtime.h"
#include <fcuda.h>
/*-------------------------------------------------------
 ** Fan1() -- Calculate multiplier matrix
 ** Pay attention to the index.  Index i give the range
 ** which starts from 0 to range-1.  The real values of
 ** the index should be adjust and related with the value
 ** of t which is defined on the ForwardSub().
 **-------------------------------------------------------
 */
#pragma FCUDA GRID x_dim=512
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=m_cuda port_id=0
#pragma FCUDA PORTMERGE remove_port_name=a_cuda port_id=0
__global__ void Fan1(float *m_cuda, float *a_cuda, int Size, int t)
{
#pragma FCUDA COMPUTE cores=[1] begin name=cmp_fan1 unroll=1
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < Size-1 - t) {
    //*(m_cuda + (Size * (index + t + 1) + t)) = *(a_cuda + (Size*(index + t + 1) + t)) / *(a_cuda + (Size * t + t));
    m_cuda[Size * (index + t + 1) + t] = a_cuda[Size * (index + t + 1) + t] / a_cuda[Size * t + t];
  }
#pragma FCUDA COMPUTE cores=[1] end name=cmp_fan1 unroll=1
}

#pragma FCUDA GRID x_dim=512 y_dim=512
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=m_cuda port_id=0
#pragma FCUDA PORTMERGE remove_port_name=a_cuda port_id=0
#pragma FCUDA PORTMERGE remove_port_name=b_cuda port_id=0
__global__ void Fan2(float *m_cuda, float *a_cuda, float *b_cuda,int Size, int j1, int t)
{
#pragma FCUDA COMPUTE cores=[1] begin name=cmp_fan2 unroll=1
  int xidx = blockIdx.x * blockDim.x + threadIdx.x;
  int yidx = blockIdx.y * blockDim.y + threadIdx.y;

  if(xidx < Size-1-t) {
    //return;
    //}
    if(yidx < Size-t) {
      //return;
      //}
      a_cuda[Size*(xidx+1+t)+(yidx+t)] -= m_cuda[Size*(xidx+1+t)+t] * a_cuda[Size*t+(yidx+t)];
  if(yidx == 0) {
    b_cuda[xidx+1+t] -= m_cuda[Size*(xidx+1+t)+(yidx+t)] * b_cuda[t];
  }
}
}
#pragma FCUDA COMPUTE cores=[1] end name=cmp_fan2 unroll=1
}

