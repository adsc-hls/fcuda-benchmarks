#include "hip/hip_runtime.h"
#include <fcuda.h>
#include <string.h>
typedef float DATATYPE;
#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))
#define BLOCK_SIZE 16
#pragma FCUDA GRID x_dim=16 y_dim=16
#pragma FCUDA COREINFO num_cores=1 pipeline=yes
#pragma FCUDA PORTMERGE remove_port_name=power port_id=0
#pragma FCUDA PORTMERGE remove_port_name=temp_src port_id=0
#pragma FCUDA PORTMERGE remove_port_name=temp_dst port_id=0
__global__ void calculate_temp(int iteration,  			// number of iteration
    DATATYPE *power,   		// power input
    DATATYPE *temp_src, 		// temperature input/output
    DATATYPE *temp_dst,  		// temperature input/output
    int grid_cols,  			// Col of grid
    int grid_rows,  			// Row of grid
    int border_cols,  		// border offset
    int border_rows,  		// border offset
    DATATYPE Cap,      		// Capacitance
    DATATYPE Rx,
    DATATYPE Ry,
    DATATYPE Rz,
    DATATYPE step,
    DATATYPE time_elapsed){

  __shared__ DATATYPE temp_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ DATATYPE power_on_cuda[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ DATATYPE temp_t[BLOCK_SIZE][BLOCK_SIZE]; // saving temporary temperature result

  DATATYPE amb_temp = 80.0;
  DATATYPE step_div_Cap;
  DATATYPE Rx_1,Ry_1,Rz_1;

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx=threadIdx.x;
  int ty=threadIdx.y;

  step_div_Cap=step/Cap;

  Rx_1=1/Rx;
  Ry_1=1/Ry;
  Rz_1=1/Rz;

  // each block finally computes result for a small block
  // after N iterations.
  // it is the non-overlapping small blocks that cover
  // all the input data

  // calculate the small block size
  int small_block_rows = BLOCK_SIZE-iteration*2;//EXPAND_RATE
  int small_block_cols = BLOCK_SIZE-iteration*2;//EXPAND_RATE

  // calculate the boundary for the block according to
  // the boundary of its small block
  int blkY = small_block_rows*by-border_rows;
  int blkX = small_block_cols*bx-border_cols;
  int blkYmax = blkY+BLOCK_SIZE-1;
  int blkXmax = blkX+BLOCK_SIZE-1;

#pragma FCUDA COMPUTE cores=[1] begin name=fetch unroll=1 mpart=1 array_split=[temp_t]
  // calculate the global thread coordination
  int yidx = blkY+ty;
  int xidx = blkX+tx;
  // load data if it is within the valid input range
  int loadYidx=yidx, loadXidx=xidx;
  int index = grid_cols*loadYidx+loadXidx;

  if(IN_RANGE(loadYidx, 0, grid_rows-1) && IN_RANGE(loadXidx, 0, grid_cols-1)) {
    temp_on_cuda[ty][tx] = temp_src[grid_cols*ty + grid_cols*blkY + blkX + tx]; // Load the temperature data from global memory to shared memory
    power_on_cuda[ty][tx] = power[grid_cols*ty + grid_cols*blkY + blkX + tx]; // Load the power data from global memory to shared memory
  }
#pragma FCUDA COMPUTE cores=[1] end name=fetch unroll=1 mpart=1 array_split=[temp_t]
  __syncthreads();

#pragma FCUDA COMPUTE cores=[1] begin name=compute unroll=1 mpart=1 array_split=[temp_t]
  // effective range within this block that falls within
  // the valid range of the input data
  // used to rule out computation outside the boundary.
  int validYmin = (blkY < 0) ? -blkY : 0;
  int validYmax = (blkYmax > grid_rows-1) ? BLOCK_SIZE-1-(blkYmax-grid_rows+1) : BLOCK_SIZE-1;
  int validXmin = (blkX < 0) ? -blkX : 0;
  int validXmax = (blkXmax > grid_cols-1) ? BLOCK_SIZE-1-(blkXmax-grid_cols+1) : BLOCK_SIZE-1;

  /*
     int N = ty-1;
     int S = ty+1;
     int W = tx-1;
     int E = tx+1;
     N = (N < validYmin) ? validYmin : N;
     S = (S > validYmax) ? validYmax : S;
     W = (W < validXmin) ? validXmin : W;
     E = (E > validXmax) ? validXmax : E;
   */

  //bool computed;
  int i, computed;
  for (i=0; i<iteration ; i++){
    int N = ty-1;
    int S = ty+1;
    int W = tx-1;
    int E = tx+1;

    N = (N < validYmin) ? validYmin : N;
    S = (S > validYmax) ? validYmax : S;
    W = (W < validXmin) ? validXmin : W;
    E = (E > validXmax) ? validXmax : E;

    computed = 0;
    if(IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
        IN_RANGE(ty, i+1, BLOCK_SIZE-i-2) &&  \
        IN_RANGE(tx, validXmin, validXmax) && \
        IN_RANGE(ty, validYmin, validYmax)) {
      computed = 1;
      temp_t[ty][tx] =   temp_on_cuda[ty][tx] + step_div_Cap * (power_on_cuda[ty][tx] +
          (temp_on_cuda[S][tx] + temp_on_cuda[N][tx] - 2.0*temp_on_cuda[ty][tx]) * Ry_1 +
          (temp_on_cuda[ty][E] + temp_on_cuda[ty][W] - 2.0*temp_on_cuda[ty][tx]) * Rx_1 +
          (amb_temp - temp_on_cuda[ty][tx]) * Rz_1);
    }
    __syncthreads();

    if(i==iteration-1)
      break;

    if(computed)         //Assign the computation range
      temp_on_cuda[ty][tx]= temp_t[ty][tx];
    __syncthreads();
  }

#pragma FCUDA COMPUTE cores=[1] end name=compute unroll=1 mpart=1 array_split=[temp_t]

  // update the global memory
  // after the last iteration, only threads coordinated within the
  // small block perform the calculation and switch on ``computed''

#pragma FCUDA COMPUTE cores=[1] begin name=write unroll=1 mpart=1 array_split=[temp_t]
  if (computed)
    temp_dst[grid_cols*ty + grid_cols*blkY + blkX + tx] = temp_t[ty][tx];
#pragma FCUDA COMPUTE cores=[1] end name=write unroll=1 mpart=1 array_split=[temp_t]

}


