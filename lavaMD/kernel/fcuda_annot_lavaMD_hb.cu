#include "hip/hip_runtime.h"
#include <fcuda.h>
//#include <main.h>
#include <string.h>
#include <math.h>

#define fp int

#define NUMBER_PAR_PER_BOX 100	// keep this low to allow more blocks that share shared memory to run concurrently,
// code does not work for larger than 110, more speedup can be achieved with larger number and no shared memory used

#define NUMBER_THREADS 128  	// this should be roughly equal to NUMBER_PAR_PER_BOX for best performance

//#define DOT(A,B) ((A.x)*(B.x)+(A.y)*(B.y)+(A.z)*(B.z))	// STABLE
#define DOT(A, i, B, j) ((A[i + 1])*(B[j + 1])+(A[i + 2])*(B[j + 2])+(A[i + 3])*(B[j + 3]))
typedef struct
{
  fp x, y, z;

} THREE_VECTOR;

typedef struct
{
  fp v, x, y, z;

} FOUR_VECTOR;

typedef struct nei_str
{

  // neighbor box
  int x, y, z;
  int number;
  long offset;

} nei_str;

typedef struct box_str
{

  // home box
  int x, y, z;
  int number;
  long offset;

  // neighbor boxes
  int nn;
  nei_str nei[26];

} box_str;

typedef struct par_str
{

  fp alpha;

} par_str;

typedef struct dim_str
{

  // input arguments
  int cur_arg;
  int arch_arg;
  int cores_arg;
  int boxes1d_arg;

  // system memory
  long number_boxes;
  long box_mem;
  long space_elem;
  long space_mem;
  long space_mem2;

} dim_str;


#pragma FCUDA GRID x_dim=128
#pragma FCUDA COREINFO num_cores=1 pipeline=yes
#pragma FCUDA PORTMERGE remove_port_name=d_box_gpu_offset port_id=0
#pragma FCUDA PORTMERGE remove_port_name=d_box_gpu_nn port_id=0
#pragma FCUDA PORTMERGE remove_port_name=d_box_gpu_number port_id=0
#pragma FCUDA PORTMERGE remove_port_name=d_rv_gpu port_id=0
#pragma FCUDA PORTMERGE remove_port_name=d_qv_gpu port_id=0
#pragma FCUDA PORTMERGE remove_port_name=d_fv_gpu port_id=0
__global__ void kernel_gpu_cuda(//par_str d_par_gpu,
    fp alpha,
    //dim_str d_dim_gpu,
    int number_boxes,
    fp *d_box_gpu_offset,
    fp *d_box_gpu_nn,
    fp *d_box_gpu_number,
    //box_str *d_box_gpu,
    fp *d_rv_gpu,
    fp *d_qv_gpu,
    fp *d_fv_gpu)
{
  int bx = blockIdx.x;	// get current horizontal block index (0-n)
  int tx = threadIdx.x;	// get current horizontal thread index (0-n)
  //int wtx = tx;

  //if(bx < d_dim_gpu.number_boxes) {
  if (bx < number_boxes) {
    ///*
    // parameters
    //fp a2 = 2.0 * d_par_gpu.alpha * d_par_gpu.alpha;
    fp a2 = 2.0 * alpha * alpha;

    // home box
    int first_i;
    fp* rA;
    fp* fA;
    __shared__ fp rA_shared[4 * NUMBER_PAR_PER_BOX];
    __shared__ fp fA_shared[4 * NUMBER_PAR_PER_BOX];

    // nei box
    int pointer;
    int k = 0;
    int first_j;
    fp* rB;
    fp* qB;
    int j = 0;
    __shared__ fp rB_shared[4 * NUMBER_PAR_PER_BOX];
    __shared__ fp qB_shared[NUMBER_PAR_PER_BOX];

    // common
    fp r2;
    fp u2;
    fp vij;
    fp fs;
    fp fxij;
    fp fyij;
    fp fzij;
    THREE_VECTOR d;
    //*/
    // home box - box parameters
    first_i = d_box_gpu_offset[bx]; //d_box_gpu[bx].offset;

    // home box - distance, force, charge and type parameters
    //rA = &d_rv_gpu[first_i];
    //fA = &d_fv_gpu[first_i];

    // home box - shared memory
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0|0] begin name=fetch1 pointer=[d_rv_gpu|d_fv_gpu] size=[512|512] unroll=1 mpart=1 array_split=[rA_shared]
    //while (wtx < NUMBER_PAR_PER_BOX) {
    rA_shared[tx] = d_rv_gpu[4 * first_i + tx]; //rA[wtx];
    fA_shared[tx] = d_fv_gpu[4 * first_i + tx];
    //wtx = wtx + NUMBER_THREADS;
    //}
    //wtx = tx;
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0|0] end name=fetch1 pointer=[d_rv_gpu|d_fv_gpu] size=[512|512] unroll=1 mpart=1 array_split=[rA_shared]

    // synchronize threads  - not needed, but just to be safe
    //__syncthreads();

    // loop over neiing boxes of home box
    for (k = 0; k < 1 + d_box_gpu_nn[bx] /*d_box_gpu[bx].nn*/; k++) {

      if(k==0)
        pointer = bx;					// set first box to be processed to home box
      else
        //pointer = d_box_gpu[bx].nei[k-1].number;		// remaining boxes are nei boxes
        pointer = d_box_gpu_number[bx * 26 + k - 1];
      // nei box - box parameters
      first_j = d_box_gpu_offset[pointer]; //d_box_gpu[pointer].offset;

      // nei box - distance, (force), charge and (type) parameters
      //rB = &d_rv_gpu[first_j];
      //qB = &d_qv_gpu[first_j];

      // nei box - shared memory
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0|0] begin name=fetch2 pointer=[d_rv_gpu|d_qv_gpu] size=[512|128] unroll=1 mpart=1 array_split=[rA_shared]
      //while(wtx < NUMBER_PAR_PER_BOX) {
      rB_shared[tx] = d_rv_gpu[4 * first_j + tx];
      qB_shared[tx] = d_qv_gpu[first_j + tx];
      //wtx = wtx + NUMBER_THREADS;
      //}
      //wtx = tx;
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0|0] end name=fetch2 pointer=[d_rv_gpu|d_qv_gpu] size=[512|128] unroll=1 mpart=1 array_split=[rA_shared]
      // synchronize threads because in next section each thread accesses data brought in by different threads here
      //__syncthreads();

      // loop for the number of particles in the home box

#pragma FCUDA COMPUTE cores=[1] begin name=compute unroll=1 mpart=1 array_split=[rA_shared] //shape=[100]
      int wtx = tx;
      if (wtx < NUMBER_PAR_PER_BOX) {
        //while (wtx < NUMBER_PAR_PER_BOX) {

        // loop for the number of particles in the current nei box
        for (j=0; j< 4 *NUMBER_PAR_PER_BOX; j+=4){
          r2 = (fp)rA_shared[4 * wtx] + (fp)rB_shared[j] - DOT((fp)rA_shared, 4 * wtx, (fp)rB_shared, j); //DOT((fp)rA_shared[4 * wtx],(fp)rB_shared[j]);
          u2 = a2*r2;
          vij = -u2; //exp(-u2);
          fs = 2*vij;

          d.x = (fp)rA_shared[4 * wtx + 1]  - (fp)rB_shared[j + 1];
          fxij = fs*d.x;
          d.y = (fp)rA_shared[4 * wtx + 2]  - (fp)rB_shared[j + 2];
          fyij = fs*d.y;
          d.z = (fp)rA_shared[4 * wtx + 3]  - (fp)rB_shared[j + 3];
          fzij = fs*d.z;

          //d_fv_gpu[first_i + wtx].v += (fp)((fp)qB_shared[j]*vij);
          fA_shared[4 * wtx] += (fp)((fp)qB_shared[j / 4]*vij);
          //d_fv_gpu[first_i + wtx].x += (fp)((fp)qB_shared[j]*fxij);
          fA_shared[4 * wtx + 1] += (fp)((fp)qB_shared[j / 4]*fxij);
          //d_fv_gpu[first_i + wtx].y += (fp)((fp)qB_shared[j]*fyij);
          fA_shared[4 * wtx + 2] += (fp)((fp)qB_shared[j / 4]*fyij);
          //d_fv_gpu[first_i + wtx].z += (fp)((fp)qB_shared[j]*fzij);
          fA_shared[4 * wtx + 3] += (fp)((fp)qB_shared[j / 4]*fzij);
        }

        // increment work thread index
        //wtx = wtx + NUMBER_THREADS;

      }
      // reset work index
      //wtx = tx;

      // synchronize after finishing force contributions from current nei box not to cause conflicts when starting next box
      __syncthreads();
#pragma FCUDA COMPUTE cores=[1] end name=compute unroll=1 mpart=1 array_split=[rA_shared] //shape=[100]
      }

#pragma FCUDA TRANSFER cores=[1] dir=[1] type=burst begin name=write pointer=[d_fv_gpu] size=[512] unroll=1 mpart=1 array_split=[rA_shared]
      d_fv_gpu[4 * first_i + tx] = fA_shared[tx];
#pragma FCUDA TRANSFER cores=[1] dir=[1] type=burst end name=write pointer=[d_fv_gpu] size=[512] unroll=1 mpart=1 array_split=[rA_shared]
    }
}
