#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */



#ifndef FWT_KERNEL_CUH
#define FWT_KERNEL_CUH
#ifndef fwt_kernel_cuh
#define fwt_kernel_cuh

#include <fcuda.h>
#include <string.h>
//typedef float float;

///////////////////////////////////////////////////////////////////////////////
// Elementary(for vectors less than elementary size) in-shared memory
// combined radix-2 + radix-4 Fast Walsh Transform
///////////////////////////////////////////////////////////////////////////////
#define ELEMENTARY_LOG2SIZE 11
#define ELEMENTARY_SIZE 2048
#define THREAD_N 256

#pragma FCUDA GRID x_dim=512
#pragma FCUDA COREINFO num_cores=2 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=d_Input port_id=0 offset=0
#pragma FCUDA PORTMERGE remove_port_name=d_Output port_id=0 offset=8388608
//note: this kernel does not allow array partitioning as it will lead to incorrect result
__global__ void fwtBatch1Kernel(float *d_Output, float *d_Input, int log2N)
{
    	const int    N = 1 << log2N;
    	const int base = blockIdx.x << log2N;
    	__shared__ float s_data[ELEMENTARY_SIZE];
	__shared__ float s_data1[ELEMENTARY_SIZE];
#pragma FCUDA TRANSFER name=fetch type=burst dir=[0] pointer=[d_Input] cores=[1] begin size=[2048] unroll=1
   	s_data[0] = d_Input[base];
#pragma FCUDA TRANSFER name=fetch type=burst dir=[0] pointer=[d_Input] cores=[1] end size=[2048] unroll=1

#pragma FCUDA COMPUTE cores=[1] begin name=vec_blk unroll=1
    	//Main radix-4 stages
    	const int pos = threadIdx.x;
    	int stride;
    	for(stride = N >> 2; stride > 0; stride >>= 2) {
        	int lo = pos & (stride - 1);
        	int i0 = ((pos - lo) << 2) + lo;
        	int i1 = i0 + stride;
        	int i2 = i1 + stride;
        	int i3 = i2 + stride;
		//__syncthreads(); // FCUDA: avoid creating ThreadLoop here to make the execution correctly.
        	float D0 = s_data[i0];
        	float D1 = s_data[i1];
       	 	float D2 = s_data[i2];
        	float D3 = s_data[i3];

        	float T;
        	T = D0;
		D0 = D0 + D2;
		D2 = T - D2;
        	T = D1;
		D1 = D1 + D3;
		D3 = T - D3;
        	T = D0;
		s_data[i0] = D0 + D1;
		s_data[i1] = T - D1;
        	T = D2;
		s_data[i2] = D2 + D3;
		s_data[i3] = T - D3;
        	__syncthreads();
    	}

    	//Do single radix-2 stage for odd power of two
	if(log2N & 1) {
    		__syncthreads();
		int pos2;
    		for(pos2 = threadIdx.x+0; pos2 < N / 2; pos2 += blockDim.x) {
    			int i4 = pos2 << 1;
        		int i5 = i4 + 1;

        		float D4 = s_data[i4];
        		float D5 = s_data[i5];
        		s_data[i4] = D4 + D5;
        		s_data[i5] = D4 - D5;
			s_data1[i4] = s_data[i4];
			s_data1[i5] = s_data[i5];
       		}
    	}
	//__syncthreads();
#pragma FCUDA COMPUTE cores=[1] end name=vec_blk unroll=1

#pragma FCUDA TRANSFER name=write type=burst dir=[1] pointer=[d_Output] cores=[1] begin size=[2048] unroll=1
	d_Output[base] = s_data1[0];
#pragma FCUDA TRANSFER name=write type=burst dir=[1] pointer=[d_Output] cores=[1] end size=[2048] unroll=1

}

#pragma FCUDA GRID x_dim=256
#pragma FCUDA COREINFO num_cores=2 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=d_Input port_id=0 offset=0
#pragma FCUDA PORTMERGE remove_port_name=d_Output port_id=0 offset=8388608
__global__ void fwtBatch2Kernel(float *d_Output, float *d_Input, int stride)
{
    	const int pos = blockIdx.x * blockDim.x;
    	__shared__ float bram_d0[THREAD_N];
    	__shared__ float bram_d1[THREAD_N];
    	__shared__ float bram_d2[THREAD_N];
    	__shared__ float bram_d3[THREAD_N];

	__shared__ float bram_d01[THREAD_N];
	__shared__ float bram_d11[THREAD_N];
	__shared__ float bram_d21[THREAD_N];
	__shared__ float bram_d31[THREAD_N];

    	int lo = pos & (stride - 1);
    	int i0 = ((pos - lo) << 2) + lo;
    	int i1 = i0 + stride;
    	int i2 = i1 + stride;
    	int i3 = i2 + stride;

#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0|0|0|0] pointer=[d_Input|d_Input|d_Input|d_Input] size=[256|256|256|256] begin name=fetch unroll=1 mpart=1 array_split=[bram_d0|bram_d1|bram_d2|bram_d3]
    	bram_d0[threadIdx.x] = d_Input[i0 + threadIdx.x];
    	bram_d1[threadIdx.x] = d_Input[i1 + threadIdx.x];
    	bram_d2[threadIdx.x] = d_Input[i2 + threadIdx.x];
    	bram_d3[threadIdx.x] = d_Input[i3 + threadIdx.x];
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[0|0|0|0] pointer=[d_Input|d_Input|d_Input|d_Input] size=[256|256|256|256] end name=fetch unroll=1 mpart=1 array_split=[bram_d0|bram_d1|bram_d2|bram_d3]


#pragma FCUDA COMPUTE cores=[1] begin name=vec_blk array_split=[bram_d0|bram_d1|bram_d2|bram_d3] unroll=1 mpart=1

    	float D0 = bram_d0[threadIdx.x];
    	float D1 = bram_d1[threadIdx.x];
    	float D2 = bram_d2[threadIdx.x];
    	float D3 = bram_d3[threadIdx.x];

    	float T;
    	T = D0;
	D0 = D0 + D2;
	D2 = T - D2;
    	T = D1;
	D1 = D1 + D3;
	D3 = T - D3;
    	T = D0;

    	bram_d0[threadIdx.x] = D0 + D1;
	bram_d01[threadIdx.x] = bram_d0[threadIdx.x];

    	bram_d1[threadIdx.x] = T - D1;
	bram_d11[threadIdx.x] = bram_d1[threadIdx.x];

    	T = D2;
    	bram_d2[threadIdx.x] = D2 + D3;
	bram_d21[threadIdx.x] = bram_d2[threadIdx.x];

    	bram_d3[threadIdx.x] = T - D3;
	bram_d31[threadIdx.x] = bram_d3[threadIdx.x];

#pragma FCUDA COMPUTE cores=[1] end name=vec_blk array_split=[bram_d0|bram_d1|bram_d2|bram_d3] unroll=1 mpart=1


#pragma FCUDA TRANSFER cores=[1] type=burst dir=[1|1|1|1] pointer=[d_Output|d_Output|d_Output|d_Output] size=[256|256|256|256] begin name=write unroll=1 mpart=1 array_split=[bram_d0|bram_d1|bram_d2|bram_d3]
    	d_Output[i0 + threadIdx.x] = bram_d01[threadIdx.x];
    	d_Output[i1 + threadIdx.x] = bram_d11[threadIdx.x];
    	d_Output[i2 + threadIdx.x] = bram_d21[threadIdx.x];
    	d_Output[i3 + threadIdx.x] = bram_d31[threadIdx.x];
#pragma FCUDA TRANSFER cores=[1] type=burst dir=[1|1|1|1] pointer=[d_Output|d_Output|d_Output|d_Output] size=[256|256|256|256] end name=write unroll=1 mpart=1 array_split=[bram_d0|bram_d1|bram_d2|bram_d3]

}

#endif
#endif
