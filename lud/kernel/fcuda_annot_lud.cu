#include "hip/hip_runtime.h"
#include <fcuda.h>
#include "lud.h"
#pragma FCUDA GRID x_dim=16
#pragma FCUDA COREINFO num_cores=4 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=m port_id=0
  __global__ void
lud_diagonal(double *m, int matrix_dim, int offset)
{
  int i, j, array_offset;
  __shared__ double shadow[BLOCK_SIZE][BLOCK_SIZE];
  array_offset = offset*matrix_dim+offset;
  for(i=0; i < BLOCK_SIZE; i++){
#pragma FCUDA TRANSFER begin name=fetch type=burst dir=[0] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    shadow[i][threadIdx.x]=m[array_offset+threadIdx.x];
#pragma FCUDA TRANSFER end name=fetch type=burst dir=[0] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    array_offset += matrix_dim;
  }
  __syncthreads();

#pragma FCUDA COMPUTE begin name=compute cores=[1] unroll=1 mpart=1 array_split=[]
  for(i=0; i < BLOCK_SIZE-1; i++) {

    if (threadIdx.x>i){
      for(j=0; j < i; j++)
        shadow[threadIdx.x][i] -= shadow[threadIdx.x][j]*shadow[j][i];
      shadow[threadIdx.x][i] /= shadow[i][i];
    }

    __syncthreads();
    if (threadIdx.x>i){

      for(j=0; j < i+1; j++)
        shadow[i+1][threadIdx.x] -= shadow[i+1][j]*shadow[j][threadIdx.x];
    }
    __syncthreads();
  }

  /*
     The first row is not modified, it
     is no need to write it back to the
     global memory

   */
#pragma FCUDA COMPUTE end name=compute cores=[1] unroll=1 mpart=1 array_split=[]

  array_offset = (offset+1)*matrix_dim+offset;
  for(i=1; i < BLOCK_SIZE; i++){
#pragma FCUDA TRANSFER begin name=write type=burst dir=[1] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    m[array_offset+threadIdx.x]=shadow[i][threadIdx.x];
#pragma FCUDA TRANSFER end name=write type=burst dir=[1] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    array_offset += matrix_dim;
  }
}

#pragma FCUDA GRID x_dim=32
#pragma FCUDA COREINFO num_cores=2 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=m port_id=0
  __global__ void
lud_perimeter(double *m, int matrix_dim, int offset)
{
  __shared__ double dia[BLOCK_SIZE1][BLOCK_SIZE1];
  __shared__ double peri_row[BLOCK_SIZE1][BLOCK_SIZE1];
  __shared__ double peri_col[BLOCK_SIZE1][BLOCK_SIZE1];

  int i,j, array_offset;
  int idx, idx1;

  //if (threadIdx.x < BLOCK_SIZE) {
  idx = threadIdx.x;

  array_offset = offset*matrix_dim+offset;
  for (i=0; i < BLOCK_SIZE/2; i++){
#pragma FCUDA TRANSFER begin name=fetch1 type=burst dir=[0] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    dia[i][idx]=m[array_offset+idx];
#pragma FCUDA TRANSFER end name=fetch1 type=burst dir=[0] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    array_offset += matrix_dim;
  }

  array_offset = offset*matrix_dim+offset;
  for (i=0; i < BLOCK_SIZE; i++) {
#pragma FCUDA TRANSFER begin name=fetch2 type=burst dir=[0] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    peri_row[i][idx]=m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx];
#pragma FCUDA TRANSFER end name=fetch2 type=burst dir=[0] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    array_offset += matrix_dim;
  }

  //} else {
  //idx = threadIdx.x-BLOCK_SIZE;

  array_offset = (offset+BLOCK_SIZE/2)*matrix_dim+offset;
  for (i=BLOCK_SIZE/2; i < BLOCK_SIZE; i++){
#pragma FCUDA TRANSFER begin name=fetch3 type=burst dir=[0] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    dia[i][idx]=m[array_offset+idx];
#pragma FCUDA TRANSFER end name=fetch3 type=burst dir=[0] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    array_offset += matrix_dim;
  }

  array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
  for (i=0; i < BLOCK_SIZE; i++) {
#pragma FCUDA TRANSFER begin name=fetch4 type=burst dir=[0] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    peri_col[i][idx] = m[array_offset+idx];
#pragma FCUDA TRANSFER end name=fetch4 type=burst dir=[0] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    array_offset += matrix_dim;
  }

  //}
  __syncthreads();

  /* this version works ok on hardware, but not gpgpusim
   **************************************************************
   if (threadIdx.x < BLOCK_SIZE) { //peri-row
   idx=threadIdx.x;
   for(i=1; i < BLOCK_SIZE; i++){
   for (j=0; j < i; j++)
   peri_row[i][idx]-=dia[i][j]*peri_row[j][idx];
   }


   array_offset = (offset+1)*matrix_dim+offset;
   for(i=1; i < BLOCK_SIZE; i++){
   m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx] = peri_row[i][idx];
   array_offset += matrix_dim;
   }
   } else { //peri-col
   idx=threadIdx.x - BLOCK_SIZE;
   for(i=0; i < BLOCK_SIZE; i++){
   for(j=0; j < i; j++)
   peri_col[idx][i]-=peri_col[idx][j]*dia[j][i];
   peri_col[idx][i] /= dia[i][i];
   }

   __syncthreads();

   array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
   for(i=0; i < BLOCK_SIZE; i++){
   m[array_offset+idx] =  peri_col[i][idx];
   array_offset += matrix_dim;
   }
   }
   ***************************************************************/
#pragma FCUDA COMPUTE begin name=compute cores=[1] unroll=1 mpart=1 array_split=[]
  if (threadIdx.x < BLOCK_SIZE) { //peri-row
    idx1=threadIdx.x;
    for(i=1; i < BLOCK_SIZE; i++){
      for (j=0; j < i; j++)
        peri_row[i][idx1]-=dia[i][j]*peri_row[j][idx1];
    }
  } else { //peri-col
    idx1=threadIdx.x - BLOCK_SIZE;
    for(i=0; i < BLOCK_SIZE; i++){
      for(j=0; j < i; j++)
        peri_col[idx1][i]-=peri_col[idx1][j]*dia[j][i];
      peri_col[idx1][i] /= dia[i][i];
    }
  }
#pragma FCUDA COMPUTE end name=compute cores=[1] unroll=1 mpart=1 array_split=[]
  __syncthreads();

  //if (threadIdx.x < BLOCK_SIZE) { //peri-row
  //idx=threadIdx.x;
  array_offset = (offset+1)*matrix_dim+offset;
  for(i=1; i < BLOCK_SIZE; i++){
#pragma FCUDA TRANSFER begin name=write1 type=burst dir=[1] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx] = peri_row[i][idx];
#pragma FCUDA TRANSFER end name=write1 type=burst dir=[1] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    array_offset += matrix_dim;
  }
  //} else { //peri-col
  //idx=threadIdx.x - BLOCK_SIZE;
  array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
  for(i=0; i < BLOCK_SIZE; i++){
#pragma FCUDA TRANSFER begin name=write2 type=burst dir=[1] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    m[array_offset+idx] =  peri_col[i][idx];
#pragma FCUDA TRANSFER end name=write2 type=burst dir=[1] pointer=[m] size=[16] cores=[1] unroll=1 mpart=1 array_split=[]
    array_offset += matrix_dim;
  }
  //}

}

#pragma FCUDA GRID x_dim=16 y_dim=16
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=m port_id=0
  __global__ void
lud_internal(double *m, int matrix_dim, int offset)
{
  __shared__ double peri_row[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double peri_col[BLOCK_SIZE][BLOCK_SIZE];

  int i;
  double sum;
  int global_row_id = offset + (blockIdx.y+1)*BLOCK_SIZE;
  int global_col_id = offset + (blockIdx.x+1)*BLOCK_SIZE;
#pragma FCUDA TRANSFER begin name=fetch type=burst dir=[0|0] pointer=[m|m] size=[16|16] cores=[1] unroll=1 mpart=1 array_split=[peri_col]
  peri_row[threadIdx.y][threadIdx.x] = m[offset * matrix_dim + global_col_id + matrix_dim * threadIdx.y + threadIdx.x]; //m[(offset+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x];
  peri_col[threadIdx.y][threadIdx.x] = m[global_row_id * matrix_dim + offset + matrix_dim * threadIdx.y + threadIdx.x]; //m[(global_row_id+threadIdx.y)*matrix_dim+offset+threadIdx.x];
#pragma FCUDA TRANSFER end name=fetch type=burst dir=[0|0] pointer=[m|m] size=[16|16] cores=[1] unroll=1 mpart=1 array_split=[peri_col]

  __syncthreads();

#pragma FCUDA COMPUTE begin name=compute cores=[1] unroll=1 mpart=1 array_split=[peri_col]
  sum = 0;
  for (i=0; i < BLOCK_SIZE; i++)
    sum += peri_col[threadIdx.y][i] * peri_row[i][threadIdx.x];
  m[(global_row_id+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x] -= sum;
#pragma FCUDA COMPUTE end name=compute cores=[1] unroll=1 mpart=1 array_split=[peri_col]

}
