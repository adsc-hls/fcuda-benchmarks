#include "hip/hip_runtime.h"
/* 1d convolution
 * Device code.
 */

#include <fcuda.h>
#include <stdio.h>


#define X_DIM 16
#define Y_DIM 16
// 6 in either direction
#define CONV_DIM 6

#define X_TILE_DIM 16 + CONV_DIM+CONV_DIM
#define Y_TILE_DIM 16

int kernel[13] = {1,2,3,4,5,6,7,6,5,4,3,2,1};
#define KERNEL_NORM  (2*(1 + 2 + 3 + 4 + 5 + 6) + 7)

// vals = {1,2,3,4,5,6,7,6,5,4,3,2,1};
// [0->5] [ 6 -> 21 ] [22->27]

#pragma FCUDA GRID x_dim=28 y_dim=16
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=A port_id=0 //offset=0
#pragma FCUDA PORTMERGE remove_port_name=B port_id=0 //offset=GRID_X*GRID_Y
__global__ void
conv1d (int * A, int * B) {

#pragma FCUDA COMPUTE begin name=compute
  //#pragma FCUDA PORTMERGE remove_port_name=local_tile_with_halo
  __shared__ int local_tile_with_halo[Y_TILE_DIM][X_TILE_DIM];


  int tx,ty,bx,by,gx,gy;
  tx = threadIdx.x;
  ty = threadIdx.y;

  bx = blockIdx.x;
  by = blockIdx.y;
  
  gx=gridDim.x;
  gy=gridDim.y;

  // input location within tile: what should this tile read?
  int x_tile_idx_in ; 
  int x_tile_blk_in = bx;

  // need to set input memory location based on halos
  if (tx >= X_DIM + CONV_DIM)  {
    x_tile_idx_in = tx  - CONV_DIM - X_DIM;
    x_tile_blk_in = bx + 1 >= gx ? 0 : bx + 1;
  } else if (tx < CONV_DIM){
    x_tile_idx_in = tx  - CONV_DIM + X_DIM;
    x_tile_blk_in = bx < 1 ? gx - 1 : bx -1;
  } else {
    x_tile_idx_in = tx - CONV_DIM;
    x_tile_blk_in = bx;
  }

  /*
  if ( x_tile_blk_in < 0) {
    x_tile_blk_in = gx - 1;
  } 
  */

  /*
  if ( x_tile_blk_in < 0) {
    x_tile_blk_in = gx - 1;
  } 
  */
  /*
  else if ( x_tile_blk_in >= gx ) {
    x_tile_blk_in = 0;
  }
  */
  

  int y_tile_idx_in = ty;
  int y_tile_blk_in = by;

  // input memory location
  int x_mem_idx_in = x_tile_idx_in + x_tile_blk_in * X_DIM;
  int y_mem_idx_in = y_tile_idx_in + y_tile_blk_in * Y_DIM;

  int input_addr = y_mem_idx_in * ( gx * X_DIM ) + x_mem_idx_in;


  // output location
  int x_tile_idx_out = tx - CONV_DIM;
  int y_tile_idx_out = ty;

  // output location 
  int x_mem_idx_out = x_tile_idx_out + X_DIM * bx;
  int y_mem_idx_out = y_tile_idx_out + Y_DIM * by;

  // output location address
  int output_addr = y_mem_idx_out  * (gx * X_DIM) + x_mem_idx_out;



  local_tile_with_halo[ty][tx]  = A[input_addr];

  __syncthreads();

  // [ 0 1 2 3 4] 
  // [-2-1 0 1 2 3 4 5 6 7 8]


  int i, j;

  if (x_tile_idx_out >= 0 && x_tile_idx_out < X_DIM && y_tile_idx_out >= 0 && y_tile_idx_out < Y_DIM ) {
  int result = 0;

    for (j = 0; j < CONV_DIM * 2 + 1; j++) {
      result += kernel[j] * local_tile_with_halo[ty][x_tile_idx_out + j];
    }
    B[output_addr] = result / KERNEL_NORM;
  }
#pragma FCUDA COMPUTE end name=compute
}
