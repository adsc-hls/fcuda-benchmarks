#include "hip/hip_runtime.h"
#include <fcuda.h>
#include "bfs.h"
typedef int DATATYPE;
/*
   struct Node {
   int starting;
   int no_of_edges;
   };
   typedef struct Node Node;
 */
/*********************************************************************************
  Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

  Copyright (c) 2008 International Institute of Information Technology - Hyderabad.
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for
  educational purpose is hereby granted without fee, provided that the above copyright
  notice and this permission notice appear in all copies of this software and that you do
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR
  OTHERWISE.

  The CUDA Kernel for Applying BFS on a loaded Graph. Created By Pawan Harish
 **********************************************************************************/
#pragma FCUDA GRID x_dim=512
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=g_graph_nodes port_id=0 data_pack=yes
#pragma FCUDA PORTMERGE remove_port_name=g_graph_edges port_id=1
#pragma FCUDA PORTMERGE remove_port_name=g_cost port_id=1
#pragma FCUDA PORTMERGE remove_port_name=g_graph_mask port_id=2
#pragma FCUDA PORTMERGE remove_port_name=g_updating_graph_mask port_id=2
#pragma FCUDA PORTMERGE remove_port_name=g_graph_visited port_id=2
__global__ void Kernel(Node* g_graph_nodes, int* g_graph_edges, DATATYPE* g_graph_mask, DATATYPE* g_updating_graph_mask, DATATYPE *g_graph_visited, int* g_cost, int no_of_nodes)
{
#pragma FCUDA COMPUTE begin name=computek1 cores=[1] unroll=1
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  int i;
  if( tid<no_of_nodes && g_graph_mask[tid])
  {
    g_graph_mask[tid]=0;//false;
    //int starting = g_graph_nodes[tid].starting;
    //int no_of_edges = g_graph_nodes[tid].no_of_edges;
    //i = g_graph_nodes[tid].starting;
    //while (i < g_graph_nodes[tid].no_of_edges + g_graph_nodes[tid].starting)
    for (i = g_graph_nodes[tid].starting; i < g_graph_nodes[tid].no_of_edges + g_graph_nodes[tid].starting; i++)
    {
      int id = g_graph_edges[i];
      if(!g_graph_visited[id])
      {
        g_cost[id]=g_cost[tid]+1;
        g_updating_graph_mask[id]=1;//true;
      }
      //i++;
    }
  }
#pragma FCUDA COMPUTE end name=computek1 cores=[1] unroll=1
}


#pragma FCUDA GRID x_dim=512
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=g_graph_mask port_id=2
#pragma FCUDA PORTMERGE remove_port_name=g_updating_graph_mask port_id=2
#pragma FCUDA PORTMERGE remove_port_name=g_graph_visited port_id=2
#pragma FCUDA PORTMERGE remove_port_name=g_over port_id=2
  __global__ void
Kernel2( DATATYPE* g_graph_mask, DATATYPE *g_updating_graph_mask, DATATYPE* g_graph_visited, DATATYPE *g_over, int no_of_nodes)
{
#pragma FCUDA COMPUTE begin name=computek1 cores=[1] unroll=1
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if( tid<no_of_nodes && g_updating_graph_mask[tid])
  {

    g_graph_mask[tid]=1;//true;
    g_graph_visited[tid]=1;//true;
    *g_over=1;//true;
    g_updating_graph_mask[tid]=0;//false;
  }
#pragma FCUDA COMPUTE end name=computek1 cores=[1] unroll=1
}
