#include "hip/hip_runtime.h"
#include <fcuda.h>
#include <math.h>
#include "nn.h"
/**
 * Kernel
 * Executed on GPU
 * Calculates the Euclidean distance from each record in the database to the target position
 */
#pragma FCUDA GRID x_dim=256 y_dim=256
#pragma FCUDA COREINFO num_cores=1 pipeline=no
#pragma FCUDA PORTMERGE remove_port_name=d_locations port_id=0 data_pack=yes
#pragma FCUDA PORTMERGE remove_port_name=d_distances port_id=1
__global__ void euclid(LatLong *d_locations, float *d_distances, int numRecords,float lat, float lng)
{
#pragma FCUDA COMPUTE cores=[1] name=compute begin unroll=1 mpart=1 array_split=[]
  int globalId = blockDim.x * ( gridDim.x * blockIdx.y + blockIdx.x ) + threadIdx.x;
  LatLong *latLong = &d_locations[globalId];
  if (globalId < numRecords) {
    float *dist=&d_distances[globalId];
    *dist = (float)sqrt((lat-latLong->lat)*(lat-latLong->lat)+(lng-latLong->lng)*(lng-latLong->lng));
  }
#pragma FCUDA COMPUTE cores=[1] name=compute end unroll=1 mpart=1 array_split=[]
}
